#include "hip/hip_runtime.h"
[LinkSymbol] 133 updates in 0.01 seconds
[AnnotParser] begin
PreAnnotation: #pragma startinclude #include <fcuda.h>

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma startinclude #include "main.h"

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma startinclude #include <string.h>

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma startinclude #include <math.h>

Parent: TranslationUnit 

PreAnnotation: #pragma endinclude

Parent: TranslationUnit 

PreAnnotation: #pragma FCUDA GRID x_dim=128

Token #
Token pragma
Token FCUDA
Token GRID
Token x_dim=128
#pragma fcuda grid x_dim=128 
attached=1

PreAnnotation: #pragma FCUDA COREINFO num_cores=1 pipeline=yes

Token #
Token pragma
Token FCUDA
Token COREINFO
Token num_cores=1
Token pipeline=yes
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_offset port_id=0

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_box_gpu_offset
Token port_id=0
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_nn port_id=1

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_box_gpu_nn
Token port_id=1
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_box_gpu_number port_id=1

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_box_gpu_number
Token port_id=1
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_rv_gpu port_id=2

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_rv_gpu
Token port_id=2
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_qv_gpu port_id=2

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_qv_gpu
Token port_id=2
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
attached=1

PreAnnotation: #pragma FCUDA PORTMERGE remove_port_name=d_fv_gpu port_id=2

Token #
Token pragma
Token FCUDA
Token PORTMERGE
Token remove_port_name=d_fv_gpu
Token port_id=2
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
attached=1

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] name=compute begin unroll=1 mpart=1 array_split=[] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] name=compute begin unroll=1 mpart=1 array_split=[]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token name=compute
Token begin
Token unroll=1
Token mpart=1
Token array_split=[]
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
attached=0

Parent: DeclarationStatement 

DeclarationStatement: #pragma FCUDA COMPUTE cores=[1] name=compute end unroll=1 mpart=1 array_split=[] ;

PreAnnotation: #pragma FCUDA COMPUTE cores=[1] name=compute end unroll=1 mpart=1 array_split=[]

Token #
Token pragma
Token FCUDA
Token COMPUTE
Token cores=[1]
Token name=compute
Token end
Token unroll=1
Token mpart=1
Token array_split=[]
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
attached=0

Parent: DeclarationStatement 

[AnnotParser] end in 0.04 seconds

*** Before Any Passes  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx = blockIdx.x;
int tx = threadIdx.x;
if ((bx<number_boxes))
{
double a2 = ((2.0*alpha)*alpha);
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k = 0;
int first_j;
double * rB;
double * qB;
int j = 0;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[LinkSymbol] 133 updates in 0.00 seconds
[AnnotParser] begin
[AnnotParser] end in 0.00 seconds

*** After AnnotationParser  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx = blockIdx.x;
int tx = threadIdx.x;
if ((bx<number_boxes))
{
double a2 = ((2.0*alpha)*alpha);
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k = 0;
int first_j;
double * rB;
double * qB;
int j = 0;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[SingleDeclarator] begin
[SingleDeclarator] end in 0.00 seconds
[LinkSymbol] 133 updates in 0.00 seconds

*** After SingleDeclarator  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx = blockIdx.x;
int tx = threadIdx.x;
if ((bx<number_boxes))
{
double a2 = ((2.0*alpha)*alpha);
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k = 0;
int first_j;
double * rB;
double * qB;
int j = 0;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================

*** After InlineDeviceFunctions  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx = blockIdx.x;
int tx = threadIdx.x;
if ((bx<number_boxes))
{
double a2 = ((2.0*alpha)*alpha);
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k = 0;
int first_j;
double * rB;
double * qB;
int j = 0;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[SeparateInitializers] begin
[SeparateInitializers] examining procedure kernel_gpu_cuda
[SeparateInitializers] end in 0.03 seconds
[LinkSymbol] 133 updates in 0.00 seconds

*** After SeparateInitializers  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx;
bx=blockIdx.x;
int tx;
tx=threadIdx.x;
if ((bx<number_boxes))
{
double a2;
a2=((2.0*alpha)*alpha);
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
k=0;
int first_j;
double * rB;
double * qB;
int j;
j=0;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[AnsiDeclarations] begin
[AnsiDeclarations] end in 0.00 seconds
[LinkSymbol] 133 updates in 0.00 seconds

*** After AnsiDeclarations  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx;
int tx;
bx=blockIdx.x;
tx=threadIdx.x;
if ((bx<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[StreamInsertion-FCUDA] begin
[StreamInsertion-FCUDA] examining procedure kernel_gpu_cuda
[StreamInsertion-FCUDA] end in 0.02 seconds
[LinkSymbol] 133 updates in 0.00 seconds

*** After StreamInsertion  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int bx;
int tx;
bx=blockIdx.x;
tx=threadIdx.x;
if ((bx<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[bx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[tx]=d_rv_gpu[((4*first_i)+tx)];
fA_shared[tx]=d_fv_gpu[((4*first_i)+tx)];
for (k=0; k<(1+d_box_gpu_nn[bx]); k ++ )
{
int wtx;
if ((k==0))
{
pointer=bx;
}
else
{
pointer=d_box_gpu_number[(((bx*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[tx]=d_rv_gpu[((4*first_j)+tx)];
qB_shared[tx]=d_qv_gpu[(first_j+tx)];
wtx=tx;
if ((wtx<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*wtx)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*wtx)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*wtx)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*wtx)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*wtx)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*wtx)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*wtx)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*wtx)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*wtx)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*wtx)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*wtx)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+tx)]=fA_shared[tx];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[RemoveThrDepLoops-FCUDA] begin
[RemoveThrDepLoops-FCUDA] examining procedure kernel_gpu_cuda
mVar2Var:
{d_box_gpu_nn=[], d_qv_gpu=[], fxij=[u2, threadIdx, alpha, fs, a2, vij, r2, j], fA=[], rB_shared=[first_j, threadIdx, pointer, k, blockIdx], d_rv_gpu=[], qB_shared=[first_j, threadIdx, pointer, k, blockIdx], first_j=[pointer, k, blockIdx], first_i=[blockIdx], rA_shared=[first_i, threadIdx, blockIdx], fA_shared=[first_i, fyij, u2, fzij, threadIdx, alpha, fs, a2, fxij, vij, r2, j, blockIdx], threadIdx=[], alpha=[], d_box_gpu_number=[], fs=[u2, threadIdx, alpha, a2, vij, r2, j], a2=[alpha], d_fv_gpu=[first_i, threadIdx, blockIdx], vij=[u2, threadIdx, alpha, a2, r2, j], d=[threadIdx, j], u2=[threadIdx, alpha, a2, r2, j], fzij=[u2, threadIdx, alpha, fs, a2, vij, r2, j], d_box_gpu_offset=[], j=[j], k=[k], rB=[], fyij=[u2, threadIdx, alpha, fs, a2, vij, r2, j], rA=[], qB=[], pointer=[k, blockIdx], r2=[threadIdx, j], number_boxes=[], blockIdx=[]}
[RemoveThrDepLoops-FCUDA] end in 0.03 seconds
[LinkSymbol] 154 updates in 0.00 seconds

*** After RemoveThrDepLoops  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[MakeArraysInCompute-FCUDA] begin
[MakeArraysInCompute-FCUDA] examining procedure kernel_gpu_cuda
Statement: {
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}
Statement: if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
Statement: {
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
Statement: double a2;
Statement: int first_i;
Statement: double * rA;
Statement: double * fA;
Statement: __shared__ double rA_shared[(4*100)];
Statement: __shared__ double fA_shared[(4*100)];
Statement: int pointer;
Statement: int k;
Statement: int first_j;
Statement: double * rB;
Statement: double * qB;
Statement: int j;
Statement: __shared__ double rB_shared[(4*100)];
Statement: __shared__ double qB_shared[100];
Statement: double r2;
Statement: double u2;
Statement: double vij;
Statement: double fs;
Statement: double fxij;
Statement: double fyij;
Statement: double fzij;
Statement: THREE_VECTOR d;
Statement: a2=((2.0*alpha)*alpha);
Statement: k=0;
Statement: j=0;
Statement: first_i=d_box_gpu_offset[blockIdx.x];
Statement: #pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
Statement: rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
Inside compute: rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
Statement: fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
Inside compute: fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
Statement: for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
Inside compute: for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
Statement: d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
Inside compute: d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
Statement: #pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
IDEXPR u2
[MakeArrays]: u2 is scalar:true used:false
IDEXPR fzij
[MakeArrays]: fzij is scalar:true used:false
IDEXPR fxij
[MakeArrays]: fxij is scalar:true used:false
WARNING: currently do not handle RHS elements of AccessExpression d in findVarsToConvert()
IDEXPR d
[MakeArrays]: d is scalar:true used:false
IDEXPR j
[MakeArrays]: j is scalar:true used:false
IDEXPR k
[MakeArrays]: k is scalar:true used:false
IDEXPR first_j
[MakeArrays]: first_j is scalar:true used:false
IDEXPR fyij
[MakeArrays]: fyij is scalar:true used:false
WARNING: currently do not handle RHS elements of AccessExpression d in findVarsToConvert()
IDEXPR d
[MakeArrays]: d is scalar:true used:false
WARNING: currently do not handle RHS elements of AccessExpression d in findVarsToConvert()
IDEXPR d
[MakeArrays]: d is scalar:true used:false
IDEXPR fs
[MakeArrays]: fs is scalar:true used:false
IDEXPR pointer
[MakeArrays]: pointer is scalar:true used:false
IDEXPR vij
[MakeArrays]: vij is scalar:true used:false
IDEXPR r2
[MakeArrays]: r2 is scalar:true used:false
Statement: k=0;
Statement: {
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
Statement: if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
Statement: first_j=d_box_gpu_offset[pointer];
Statement: rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
Statement: qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
Statement: if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
Statement: __syncthreads();
Statement: {
pointer=blockIdx.x;
}
Statement: {
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
Statement: {
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
Statement: pointer=blockIdx.x;
Statement: pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
Statement: for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
Statement: j=0;
Statement: {
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
Statement: r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
Statement: u2=(a2*r2);
Statement: vij=exp(( - u2));
Statement: fs=(2*vij);
Statement: d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
Statement: fxij=(fs*d.x);
Statement: d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
Statement: fyij=(fs*d.y);
Statement: d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
Statement: fzij=(fs*d.z);
Statement: fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
Statement: fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
Statement: fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
Statement: fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
[MakeArraysInCompute-FCUDA] end in 0.02 seconds
[LinkSymbol] 154 updates in 0.00 seconds

*** After MakeArraysInCompute  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
__shared__ double rA_shared[(4*100)];
__shared__ double fA_shared[(4*100)];
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[SplitFcudaTasks-FCUDA] begin
[SplitFcudaTasks-FCUDA] examining procedure kernel_gpu_cuda
BRAM:rA_shared  specs: [[(4*100)]] size:1
BRAM:fA_shared  specs: [[(4*100)]] size:1
BRAM:rB_shared  specs: [[(4*100)]] size:1
BRAM:qB_shared  specs: [[100]] size:1

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
Creating new FcudaCoreData for core: kernel_gpu_cuda_compute()

 ... Preprocessing pragma: 
	#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
fcudaCores (splitTasks-start):
[kernel_gpu_cuda_compute()]
coreNames: 
[kernel_gpu_cuda_compute()]
Checking Annotation Statement: [#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true ]
FCUDA compute begin=true level=2
Task stmt: rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
of type: class cetus.hir.ExpressionStatement
Task stmt: fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
of type: class cetus.hir.ExpressionStatement
Task stmt: for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
of type: class cetus.hir.ForLoop
Task stmt: d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
of type: class cetus.hir.ExpressionStatement
Task stmt: #pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
of type: class cetus.hir.AnnotationStatement
Checking Annotation Statement: [#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false ]
FCUDA compute begin=false level=2
Task use set: [a2, blockIdx.x, d.x, d.y, d.z, d_box_gpu_nn[blockIdx.x], d_box_gpu_number[(((blockIdx.x*26)+k)-1)], d_box_gpu_offset[pointer], d_fv_gpu[((4*first_i)+threadIdx.x)], d_qv_gpu[(first_j+threadIdx.x)], d_rv_gpu[((4*first_i)+threadIdx.x)], d_rv_gpu[((4*first_j)+threadIdx.x)], fA_shared[((4*threadIdx.x)+1)], fA_shared[((4*threadIdx.x)+2)], fA_shared[((4*threadIdx.x)+3)], fA_shared[(4*threadIdx.x)], fA_shared[threadIdx.x], first_i, first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared[(j/4)], r2, rA_shared[((4*threadIdx.x)+1)], rA_shared[((4*threadIdx.x)+2)], rA_shared[((4*threadIdx.x)+3)], rA_shared[(4*threadIdx.x)], rB_shared[(j+1)], rB_shared[(j+2)], rB_shared[(j+3)], rB_shared[j], threadIdx.x, u2, vij]
Task def set: [d.x, d.y, d.z, d_fv_gpu[((4*first_i)+threadIdx.x)], fA_shared[((4*threadIdx.x)+1)], fA_shared[((4*threadIdx.x)+2)], fA_shared[((4*threadIdx.x)+3)], fA_shared[(4*threadIdx.x)], fA_shared[threadIdx.x], first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared[threadIdx.x], r2, rA_shared[threadIdx.x], rB_shared[threadIdx.x], u2, vij]
Task maydef set: []
tmpExpr: a2
  of class: class cetus.hir.Identifier
decl: double a2
defStmt: none
tmpExpr: blockIdx.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression blockIdx.x in buildTaskVars()
tmpExpr: d.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression d.x in buildTaskVars()
decl: THREE_VECTOR d
defStmt: none
tmpExpr: d.y
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression d.y in buildTaskVars()
decl: THREE_VECTOR d
defStmt: none
tmpExpr: d.z
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression d.z in buildTaskVars()
decl: THREE_VECTOR d
defStmt: none
tmpExpr: d_box_gpu_nn[blockIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: int * d_box_gpu_nn
defStmt: none
tmpExpr: d_box_gpu_number[(((blockIdx.x*26)+k)-1)]
  of class: class cetus.hir.ArrayAccess
decl: int * d_box_gpu_number
defStmt: none
tmpExpr: d_box_gpu_offset[pointer]
  of class: class cetus.hir.ArrayAccess
decl: long * d_box_gpu_offset
defStmt: none
tmpExpr: d_fv_gpu[((4*first_i)+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: double * d_fv_gpu
defStmt: none
tmpExpr: d_qv_gpu[(first_j+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: double * d_qv_gpu
defStmt: none
tmpExpr: d_rv_gpu[((4*first_i)+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: double * d_rv_gpu
defStmt: none
tmpExpr: d_rv_gpu[((4*first_j)+threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: double * d_rv_gpu
defStmt: none
tmpExpr: fA_shared[((4*threadIdx.x)+1)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double fA_shared[(4*100)]
defStmt: none
tmpExpr: fA_shared[((4*threadIdx.x)+2)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double fA_shared[(4*100)]
defStmt: none
tmpExpr: fA_shared[((4*threadIdx.x)+3)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double fA_shared[(4*100)]
defStmt: none
tmpExpr: fA_shared[(4*threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double fA_shared[(4*100)]
defStmt: none
tmpExpr: fA_shared[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double fA_shared[(4*100)]
defStmt: none
tmpExpr: first_i
  of class: class cetus.hir.Identifier
decl: int first_i
defStmt: none
tmpExpr: first_j
  of class: class cetus.hir.Identifier
decl: int first_j
defStmt: none
tmpExpr: fs
  of class: class cetus.hir.Identifier
decl: double fs
defStmt: none
tmpExpr: fxij
  of class: class cetus.hir.Identifier
decl: double fxij
defStmt: none
tmpExpr: fyij
  of class: class cetus.hir.Identifier
decl: double fyij
defStmt: none
tmpExpr: fzij
  of class: class cetus.hir.Identifier
decl: double fzij
defStmt: none
tmpExpr: j
  of class: class cetus.hir.Identifier
decl: int j
defStmt: none
tmpExpr: k
  of class: class cetus.hir.Identifier
decl: int k
defStmt: none
tmpExpr: pointer
  of class: class cetus.hir.Identifier
decl: int pointer
defStmt: none
tmpExpr: qB_shared[(j/4)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double qB_shared[100]
defStmt: none
tmpExpr: qB_shared[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double qB_shared[100]
defStmt: none
tmpExpr: r2
  of class: class cetus.hir.Identifier
decl: double r2
defStmt: none
tmpExpr: rA_shared[((4*threadIdx.x)+1)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rA_shared[(4*100)]
defStmt: none
tmpExpr: rA_shared[((4*threadIdx.x)+2)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rA_shared[(4*100)]
defStmt: none
tmpExpr: rA_shared[((4*threadIdx.x)+3)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rA_shared[(4*100)]
defStmt: none
tmpExpr: rA_shared[(4*threadIdx.x)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rA_shared[(4*100)]
defStmt: none
tmpExpr: rA_shared[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rA_shared[(4*100)]
defStmt: none
tmpExpr: rB_shared[(j+1)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rB_shared[(4*100)]
defStmt: none
tmpExpr: rB_shared[(j+2)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rB_shared[(4*100)]
defStmt: none
tmpExpr: rB_shared[(j+3)]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rB_shared[(4*100)]
defStmt: none
tmpExpr: rB_shared[j]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rB_shared[(4*100)]
defStmt: none
tmpExpr: rB_shared[threadIdx.x]
  of class: class cetus.hir.ArrayAccess
decl: __shared__ double rB_shared[(4*100)]
defStmt: none
tmpExpr: threadIdx.x
  of class: class cetus.hir.AccessExpression
WARNING: currently do not handle RHS elements of AccessExpression threadIdx.x in buildTaskVars()
tmpExpr: u2
  of class: class cetus.hir.Identifier
decl: double u2
defStmt: none
tmpExpr: vij
  of class: class cetus.hir.Identifier
decl: double vij
defStmt: none
taskArgs: [enableSignal_compute, blockDim, gridDim, blockIdx, a2, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij]
taskDecls: [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, THREE_VECTOR d, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int first_j, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
defStmts: []
[SplitFcudaTasks-FCUDA] end in 0.03 seconds
[LinkSymbol] 181 updates in 0.00 seconds

*** After SplitFcudaTasks  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, THREE_VECTOR d, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int first_j, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij)
{
if (enableSignal_compute)
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double fA_shared[(4*100)];
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
double * rA;
double * fA;
int pointer;
int k;
int first_j;
double * rB;
double * qB;
int j;
double r2;
double u2;
double vij;
double fs;
double fxij;
double fyij;
double fzij;
THREE_VECTOR d;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[CleanKernelDecls-FCUDA] begin
[CleanKernelDecls-FCUDA] examining procedure kernel_gpu_cuda
cur_level:0
Defs+Uses:[blockIdx, blockIdx.x, blockIdx.y, enableSignal_compute, gridDim, gridDim.x, gridDim.y]
cur_level:1
Defs+Uses:[a2, alpha]
Defs+Uses:[k]
Defs+Uses:[j]
Defs+Uses:[blockIdx, blockIdx.x, d_box_gpu_offset, d_box_gpu_offset[blockIdx.x], first_i]
Defs+Uses:[]
Defs+Uses:[a2, blockDim, blockIdx, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, enableSignal_compute, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, gridDim, j, k, kernel_gpu_cuda_compute, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij]
Defs+Uses:[]
cur_level:1
var2freqMap{a2=2, d=1, fA=0, first_i=2, first_j=1, fs=1, fxij=1, fyij=1, fzij=1, j=2, k=2, pointer=1, qB=0, r2=1, rA=0, rB=0, u2=1, vij=1}
funcCallParams[a2, blockDim, blockIdx, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, enableSignal_compute, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, gridDim, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:d contains d
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, THREE_VECTOR d, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int first_j, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int first_j, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:first_j contains first_j
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int first_j, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, fs, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:fs contains fs
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fs, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, fxij, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:fxij contains fxij
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fxij, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, fyij, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:fyij contains fyij
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fyij, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, fzij, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:fzij contains fzij
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, double fzij, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:pointer contains pointer
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, int pointer, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, j, k, qB_shared, r2, rA_shared, rB_shared, u2, vij)
-arg:r2 contains r2
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], double r2, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, j, k, qB_shared, rA_shared, rB_shared, u2, vij)
-arg:u2 contains u2
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double u2, double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, j, k, qB_shared, rA_shared, rB_shared, vij)
-arg:vij contains vij
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)], double vij]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
cur_level:0
var2freqMap{enableSignal_compute=2, fA_shared=1, qB_shared=1, rA_shared=1, rB_shared=1}
funcCallParams[a2, blockDim, blockIdx, d, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, enableSignal_compute, fA_shared, first_i, first_j, fs, fxij, fyij, fzij, gridDim, j, k, pointer, qB_shared, r2, rA_shared, rB_shared, u2, vij]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, fA_shared, first_i, j, k, qB_shared, rA_shared, rB_shared)
-arg:fA_shared contains fA_shared
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, __shared__ double fA_shared[(4*100)], int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k, qB_shared, rA_shared, rB_shared)
-arg:qB_shared contains qB_shared
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double qB_shared[100], __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k, rA_shared, rB_shared)
-arg:rA_shared contains rA_shared
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double rA_shared[(4*100)], __shared__ double rB_shared[(4*100)]]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double rB_shared[(4*100)]]
fcall:kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k, rB_shared)
-arg:rB_shared contains rB_shared
- and are equal
- declList b4 = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k, __shared__ double rB_shared[(4*100)]]
- declList after = [int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k]
[CleanKernelDecls-FCUDA] end in 0.02 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After CleanKernelDecls  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[SerializeThreads-MCUDA] begin
[SerializeThreads-MCUDA] examining procedure kernel_gpu_cuda
[SerializeThreads-MCUDA] end in 0.01 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After SerializeThreads  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
for (k=0; k<(1+d_box_gpu_nn[blockIdx.x]); k ++ )
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
__syncthreads();
}
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[EnforceSyncs-MCUDA] begin
[EnforceSyncs-MCUDA] examining procedure kernel_gpu_cuda
[EnforceSyncs-MCUDA] end in 0.02 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After EnforceSyncs  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[PrivatizeScalarsInThreadLoops - FCUDA] begin
[PrivatizeScalarsInThreadLoops - FCUDA] examining procedure kernel_gpu_cuda
THREADLOOP: {
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
THREADLOOP: {
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
THREADLOOP: {
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
THREADLOOP: {
k ++ ;
}
DEFSET: [r2, u2, vij, fs, <d.x>, fxij, <d.y>, fyij, <d.z>, fzij]
Find UseSet of: {
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
USESET: [* d_rv_gpu, * d_fv_gpu, first_i, <threadIdx.x>]
DEFSET WITHIN LOOP: [rA_shared[(4*100)], fA_shared[(4*100)], k]
REMAINSET: []
Find UseSet of: {
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
USESET: [first_i, fA_shared[(4*100)], <threadIdx.x>]
DEFSET WITHIN LOOP: [* d_fv_gpu]
REMAINSET: []
Find UseSet of: {
k ++ ;
}
USESET: [k]
DEFSET WITHIN LOOP: [k]
REMAINSET: []
Scalars to be privatized: []
[PrivatizeScalarsInThreadLoops - FCUDA] end in 0.03 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After PrivatizeScalarsInThreadLoop  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[UnrollThreadLoops-MCUDA] begin
[UnrollThreadLoops-MCUDA] examining procedure kernel_gpu_cuda

[Unrolling] : kernel_gpu_cuda_compute
[Proc]: #pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}


[unrollFactor] 1
mUnrolledIDs: 
{}
[UnrollThreadLoops-MCUDA] end in 0.00 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After UnrollThreadLoops  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[PartitionArrays-MCUDA] begin
[PartitionArrays-MCUDA] examining procedure kernel_gpu_cuda
[numDims]1
[Memory partition] : kernel_gpu_cuda_compute

[Proc]: #pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}


[mempartFactor]1
[Memory partition] : kernel_gpu_cuda

HAA 1 {
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}
[PartitionArrays-MCUDA] end in 0.00 seconds
[LinkSymbol] 167 updates in 0.00 seconds

*** After PartitionArrays  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
double a2;
int first_i;
int k;
int j;
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=true unroll=1 begin=false 
}
}



===========================================
[IfSplitPass-FCUDA] begin
[IfSplitPass-FCUDA] examining procedure kernel_gpu_cuda
fcudaCores:
[kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k)]
coreNames: 
[kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k)]
Handling control flow for kernel_gpu_cuda_compute(enableSignal_compute, blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k)
mCurrEnableSignal: enableSignal_compute
In if stmt, lead list {
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}

In if stmt, trail list {

}
[IfSplitPass-FCUDA] end in 0.00 seconds
[LinkSymbol] 170 updates in 0.00 seconds

*** After IfSplitPass  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu)
{
int enableSignal_compute;
dim3 blockIdx;
double a2;
int first_i;
int k;
int j;
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}
kernel_gpu_cuda_compute((enableSignal_compute&&(blockIdx.x<number_boxes)), blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
}



===========================================
[WrapBlockIdxLoop-FCUDA] begin
[WrapBlockIdxLoop-FCUDA] examining procedure kernel_gpu_cuda
[WrapBlockIdxLoop-FCUDA] end in 0.00 seconds
[LinkSymbol] 170 updates in 0.00 seconds

*** After WrapBlockIdxLoop  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int enableSignal_compute;
dim3 blockIdx;
double a2;
int first_i;
int k;
int j;
blockIdx.y=0;
blockIdx.x=core_id;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}
kernel_gpu_cuda_compute((enableSignal_compute&&(blockIdx.x<number_boxes)), blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
[CleanThreadLoops-MCUDA] begin
[CleanThreadLoops-MCUDA] examining procedure kernel_gpu_cuda
[CleanThreadLoops-MCUDA] end in 0.00 seconds
[LinkSymbol] 170 updates in 0.00 seconds

*** After CleanThreadLoops  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int enableSignal_compute;
dim3 blockIdx;
double a2;
int first_i;
int k;
int j;
blockIdx.y=0;
blockIdx.x=core_id;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}
kernel_gpu_cuda_compute((enableSignal_compute&&(blockIdx.x<number_boxes)), blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
[KernelStateTransform-MCUDA] begin
[KernelStateTransform-MCUDA] examining procedure kernel_gpu_cuda
>>> outside uses:
[a2, alpha, blockIdx.x, blockIdx.y, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_box_gpu_offset[blockIdx.x], d_fv_gpu, d_qv_gpu, d_rv_gpu, enableSignal_compute, first_i, gridDim.x, gridDim.y, j, k, number_boxes]
>>> handling: a2
>>> handling: alpha
>>> handling: blockIdx
>>> handling: blockIdx
>>> handling: * d_box_gpu_nn
>>> handling: * d_box_gpu_number
>>> handling: * d_box_gpu_offset
>>> handling: * d_box_gpu_offset
>>> handling: * d_fv_gpu
>>> handling: * d_qv_gpu
>>> handling: * d_rv_gpu
>>> handling: enableSignal_compute
>>> handling: first_i
>>> handling: gridDim
>>> handling: gridDim
>>> handling: j
>>> handling: k
>>> handling: number_boxes
transforming Decls
[KernelStateTransform-MCUDA] end in 0.01 seconds
[LinkSymbol] 170 updates in 0.00 seconds

*** After KernelStateTransform  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
__syncthreads();
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
__syncthreads();
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
__syncthreads();
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int enableSignal_compute;
dim3 blockIdx;
double a2;
int first_i;
int k;
int j;
blockIdx.y=0;
blockIdx.x=core_id;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}
kernel_gpu_cuda_compute((enableSignal_compute&&(blockIdx.x<number_boxes)), blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
[CleanSyncFunc-MCUDA] begin
[CleanSyncFunc-MCUDA] examining procedure kernel_gpu_cuda
[CleanSyncFunc-MCUDA] end in 0.00 seconds
[LinkSymbol] 169 updates in 0.00 seconds

*** After CleanSyncFunc  ***
#include <fcuda.h>
#include "main.h"
#include <string.h>
#include <math.h>
const int BLOCKDIM_X_kernel_gpu_cuda = 128;
#pragma fcuda compute array_split=[] mpart=1 name=compute cores=[1] end=false unroll=1 begin=true 
void kernel_gpu_cuda_compute(int enableSignal_compute, dim3 blockDim, dim3 gridDim, dim3 blockIdx, double a2, int * d_box_gpu_nn, int * d_box_gpu_number, long * d_box_gpu_offset, double * d_fv_gpu, double * d_qv_gpu, double * d_rv_gpu, int first_i, int j, int k)
{
dim3 __shared__ threadIdx;
THREE_VECTOR d;
int first_j;
double fs;
double fxij;
double fyij;
double fzij;
int pointer;
double r2;
double u2;
double vij;
__shared__ double fA_shared[(4*100)];
__shared__ double qB_shared[100];
__shared__ double rA_shared[(4*100)];
__shared__ double rB_shared[(4*100)];
if (enableSignal_compute)
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
rA_shared[threadIdx.x]=d_rv_gpu[((4*first_i)+threadIdx.x)];
fA_shared[threadIdx.x]=d_fv_gpu[((4*first_i)+threadIdx.x)];
k=0;
}
while (k<(1+d_box_gpu_nn[blockIdx.x]))
{
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
if ((k==0))
{
pointer=blockIdx.x;
}
else
{
pointer=d_box_gpu_number[(((blockIdx.x*26)+k)-1)];
}
first_j=d_box_gpu_offset[pointer];
rB_shared[threadIdx.x]=d_rv_gpu[((4*first_j)+threadIdx.x)];
qB_shared[threadIdx.x]=d_qv_gpu[(first_j+threadIdx.x)];
if ((threadIdx.x<100))
{
for (j=0; j<(4*100); j+=4)
{
r2=((((double)rA_shared[(4*threadIdx.x)])+((double)rB_shared[j]))-(((((double)rA_shared[((4*threadIdx.x)+1)])*((double)rB_shared[(j+1)]))+(((double)rA_shared[((4*threadIdx.x)+2)])*((double)rB_shared[(j+2)])))+(((double)rA_shared[((4*threadIdx.x)+3)])*((double)rB_shared[(j+3)]))));
u2=(a2*r2);
vij=exp(( - u2));
fs=(2*vij);
d.x=(((double)rA_shared[((4*threadIdx.x)+1)])-((double)rB_shared[(j+1)]));
fxij=(fs*d.x);
d.y=(((double)rA_shared[((4*threadIdx.x)+2)])-((double)rB_shared[(j+2)]));
fyij=(fs*d.y);
d.z=(((double)rA_shared[((4*threadIdx.x)+3)])-((double)rB_shared[(j+3)]));
fzij=(fs*d.z);
fA_shared[(4*threadIdx.x)]+=((double)(((double)qB_shared[(j/4)])*vij));
fA_shared[((4*threadIdx.x)+1)]+=((double)(((double)qB_shared[(j/4)])*fxij));
fA_shared[((4*threadIdx.x)+2)]+=((double)(((double)qB_shared[(j/4)])*fyij));
fA_shared[((4*threadIdx.x)+3)]+=((double)(((double)qB_shared[(j/4)])*fzij));
}
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
k ++ ;
}
}
for (threadIdx.x=0;threadIdx.x<blockDim.x ; threadIdx.x=threadIdx.x+1) 
{
d_fv_gpu[((4*first_i)+threadIdx.x)]=fA_shared[threadIdx.x];
}
}
}

#pragma fcuda grid x_dim=128 
#pragma fcuda coreinfo num_cores=1 pipeline=yes 
#pragma fcuda portmerge port_id=0 remove_port_name=d_box_gpu_offset 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_nn 
#pragma fcuda portmerge port_id=1 remove_port_name=d_box_gpu_number 
#pragma fcuda portmerge port_id=2 remove_port_name=d_rv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_qv_gpu 
#pragma fcuda portmerge port_id=2 remove_port_name=d_fv_gpu 
__global__ void kernel_gpu_cuda(double alpha, long number_boxes, long * d_box_gpu_offset, int * d_box_gpu_nn, int * d_box_gpu_number, double * d_rv_gpu, double * d_qv_gpu, double * d_fv_gpu, dim3 gridDim, dim3 blockDim, int num_cores, int core_id)
{
int enableSignal_compute;
dim3 blockIdx;
double a2;
int first_i;
int k;
int j;
blockIdx.y=0;
blockIdx.x=core_id;
while (1)
{
while (blockIdx.x>=gridDim.x)
{
blockIdx.x=(blockIdx.x-gridDim.x);
blockIdx.y=(blockIdx.y+1);
}
if ((blockIdx.y>=gridDim.y))
{
break;
}
enableSignal_compute=((blockIdx.x<gridDim.x)&&(blockIdx.y<gridDim.y));
if ((blockIdx.x<number_boxes))
{
a2=((2.0*alpha)*alpha);
k=0;
j=0;
first_i=d_box_gpu_offset[blockIdx.x];
}
kernel_gpu_cuda_compute((enableSignal_compute&&(blockIdx.x<number_boxes)), blockDim, gridDim, blockIdx, a2, d_box_gpu_nn, d_box_gpu_number, d_box_gpu_offset, d_fv_gpu, d_qv_gpu, d_rv_gpu, first_i, j, k);
blockIdx.x=(blockIdx.x+num_cores);
}
}



===========================================
