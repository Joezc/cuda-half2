#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
//	plasmaKernel_gpu_2
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200

__global__ void kernel_gpu_cuda(par_str d_par_gpu,
								dim_str d_dim_gpu,
								box_str* d_box_gpu,
								FOUR_VECTOR_HALF* d_rv_gpu,
								half* d_qv_gpu,
								FOUR_VECTOR_HALF* d_fv_gpu)
{

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	THREAD PARAMETERS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	int bx = blockIdx.x;																// get current horizontal block index (0-n)
	int tx = threadIdx.x;															// get current horizontal thread index (0-n)
	// int ax = bx*NUMBER_THREADS+tx;
	// int wbx = bx;
	int wtx = tx;
	__half alpha_temp = __float2half(d_par_gpu.alpha);
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	DO FOR THE NUMBER OF BOXES
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	if(bx<d_dim_gpu.number_boxes){
	// while(wbx<box_indexes_counter){

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Extract input parameters
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// parameters
		__half a2 = __float2half(2.0*d_par_gpu.alpha*d_par_gpu.alpha);
		//__half a2 = __hmul(__hmul(alpha_temp, alpha_temp), __float2half(2.0));
		// home box
		int first_i;
		FOUR_VECTOR_HALF* rA;
		FOUR_VECTOR_HALF* fA;
		__shared__ FOUR_VECTOR_HALF rA_shared[100];

		// nei box
		int pointer;
		int k = 0;
		int first_j;
		FOUR_VECTOR_HALF* rB;
		half* qB;
		int j = 0;
		__shared__ FOUR_VECTOR_HALF rB_shared[100];
		__shared__ half qB_shared[100];

		// common
		//~ fp r2;
		//~ fp u2;
		//~ fp vij;
		//~ fp fs;
		//~ fp fxij;
		//~ fp fyij;
		//~ fp fzij;
		
		half r2;
		__half u2;
		__half vij;
		__half fs;
		half fxij;
		half fyij;
		__half fzij;
		
		THREE_VECTOR_HALF d;

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Home box
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Setup parameters
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - box parameters
		first_i = d_box_gpu[bx].offset;
		rA = &d_rv_gpu[first_i];
		fA = &d_fv_gpu[first_i];
		
		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Copy to shared memory
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - shared memory
		while(wtx<NUMBER_PAR_PER_BOX){
			rA_shared[wtx] = rA[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// loop over neiing boxes of home box
		//~ k=0;{
		for (k=0; k<(1+d_box_gpu[bx].nn); k++){

			//----------------------------------------50
			//	nei box - get pointer to the right box
			//----------------------------------------50

			if(k==0){
				pointer = bx;													// set first box to be processed to home box
			}
			else{
				pointer = d_box_gpu[bx].nei[k-1].number;							// remaining boxes are nei boxes
			}

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset;

			// nei box - distance, (force), charge and (type) parameters
			rB = &d_rv_gpu[first_j];
			qB = &d_qv_gpu[first_j];

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - shared memory
			while(wtx<NUMBER_PAR_PER_BOX){
				rB_shared[wtx] = rB[wtx];
				qB_shared[wtx] = qB[wtx];
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation
			//----------------------------------------------------------------------------------------------------------------------------------140
			FOUR_VECTOR_HALF fA_tmp;
			
			// loop for the number of particles in the home box
			// for (int i=0; i<nTotal_i; i++){
			//~ {
			while(wtx<NUMBER_PAR_PER_BOX){
				fA_tmp.v  = fA[wtx].v;
				fA_tmp.x  = fA[wtx].x;
				fA_tmp.y  = fA[wtx].y;
				fA_tmp.z  = fA[wtx].z;
				
				// loop for the number of particles in the current nei box
				//~ j=0;{
				for (j=0; j<NUMBER_PAR_PER_BOX; j++){

					// r2 = rA[wtx].v + rB[j].v - DOT(rA[wtx],rB[j]); 
					// u2 = a2*r2;
					// vij= exp(-u2);
					// fs = 2.*vij;

					// d.x = rA[wtx].x  - rB[j].x;
					// fxij=fs*d.x;
					// d.y = rA[wtx].y  - rB[j].y;
					// fyij=fs*d.y;
					// d.z = rA[wtx].z  - rB[j].z;
					// fzij=fs*d.z;

					// fA[wtx].v +=  qB[j]*vij;
					// fA[wtx].x +=  qB[j]*fxij;
					// fA[wtx].y +=  qB[j]*fyij;
					// fA[wtx].z +=  qB[j]*fzij;



					r2 = rA_shared[wtx].v + rB_shared[j].v - DOT(rA_shared[wtx],rB_shared[j]); 
					u2 = a2*r2;
					vij= exp(-u2);
					fs = 2*vij;

					d.x = rA_shared[wtx].x  - rB_shared[j].x;
					fxij=fs*d.x;
					d.y = rA_shared[wtx].y  - rB_shared[j].y;
					fyij=fs*d.y;
					d.z = rA_shared[wtx].z  - rB_shared[j].z;
					fzij=fs*d.z;

					//~ fA[wtx].v += (qB_shared[j]*vij);
					//~ fA[wtx].x += (qB_shared[j]*fxij);
					//~ fA[wtx].y += (qB_shared[j]*fyij);
					//~ fA[wtx].z += (qB_shared[j]*fzij);

					fA_tmp.v += (qB_shared[j]*vij);
					fA_tmp.x += (qB_shared[j]*fxij);
					fA_tmp.y += (qB_shared[j]*fyij);
					fA_tmp.z += (qB_shared[j]*fzij);
					//~ fA[wtx].v += rB_shared[j].v;
					//~ fA[wtx].x += rB_shared[j].x;
					//~ fA[wtx].y += rB_shared[j].y;
					//~ fA[wtx].z += rB_shared[j].z;


				}
				fA[wtx].v = fA_tmp.v ;
				fA[wtx].x = fA_tmp.x ;
				fA[wtx].y = fA_tmp.y ;
				fA[wtx].z = fA_tmp.z ;
				

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;

			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation END
			//----------------------------------------------------------------------------------------------------------------------------------140

		}

		// // increment work block index
		// wbx = wbx + NUMBER_BLOCKS;

		// // synchronize - because next iteration will overwrite current shared memory
		// __syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop END
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

	}

}
