//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

//======================================================================================================================================================150
//	MAIN FUNCTION HEADER
//======================================================================================================================================================150

#include "./../main.h"								// (in the main program folder)	needed to recognized input parameters

//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150

#include "./../util/device/device.h"				// (in library path specified to compiler)	needed by for device functions
#include "./../util/timer/timer.h"					// (in library path specified to compiler)	needed by timer

//======================================================================================================================================================150
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION HEADER
//======================================================================================================================================================150

#include "./kernel_gpu_cuda_wrapper.h"				// (in the current directory)

#include <hip/hip_fp16.h>
#include "../newhalf.hpp"
#include "../half_operator_overload.cuh"
#include "../half2_operator_overload.cuh"

typedef struct
{
	half x, y, z;

} THREE_VECTOR_HALF;

typedef struct
{
	half v, x, y, z;

} FOUR_VECTOR_HALF;


typedef struct
{
	half_float::half x, y, z;

} THREE_VECTOR_HALF_HOST;

typedef struct
{
	half_float::half v, x, y, z;

} FOUR_VECTOR_HALF_HOST;

typedef struct
{
	half2 x, y, z;//x.high, x.low, ....

} THREE_VECTOR_HALF2;
typedef struct
{
	half2 v, x, y, z;

} FOUR_VECTOR_HALF2;

typedef struct
{
	uint32_t x, y, z;

} THREE_VECTOR_HALF2_HOST;

typedef struct
{
	uint32_t v, x, y, z;

} FOUR_VECTOR_HALF2_HOST;

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

#include "./kernel_gpu_cuda.cu"						// (in the current directory)	GPU kernel, cannot include with header file because of complications with passing of constant memory variables

//========================================================================================================================================================================================================200
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION
//========================================================================================================================================================================================================200

void 
kernel_gpu_cuda_wrapper(par_str par_cpu,
						dim_str dim_cpu,
						box_str* box_cpu,
						FOUR_VECTOR* rv_cpu,
						fp* qv_cpu,
						FOUR_VECTOR* fv_cpu)
{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;

	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	VARIABLES
	//====================================================================================================100

	box_str* d_box_gpu;
	
	
	FOUR_VECTOR_HALF2* d_rv_gpu;
	half2* d_qv_gpu;
	FOUR_VECTOR_HALF2* d_fv_gpu;

	dim3 threads;
	dim3 blocks;

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	blocks.x = dim_cpu.number_boxes;
	blocks.y = 1;
	threads.x = NUMBER_THREADS;											// define the number of threads in the block
	threads.y = 1;

	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

	hipMalloc(	(void **)&d_box_gpu, 
				dim_cpu.box_mem);

	//==================================================50
	//	rv
	//==================================================50

	hipMalloc(	(void **)&d_rv_gpu, 
				dim_cpu.space_mem/2);

	//==================================================50
	//	qv
	//==================================================50

	hipMalloc(	(void **)&d_qv_gpu, 
				dim_cpu.space_mem2/2);

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50

	hipMalloc(	(void **)&d_fv_gpu, 
				dim_cpu.space_mem/2);

	time2 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	boxes
	//==================================================50

//convert before copying dim_cpu.space_elem 
						FOUR_VECTOR_HALF2_HOST* rv_cpu_half = (FOUR_VECTOR_HALF2_HOST*) malloc(dim_cpu.space_mem/2);
						half_float::half* qv_cpu_half = (half_float::half*) malloc(dim_cpu.space_mem2/2);
						FOUR_VECTOR_HALF2_HOST* fv_cpu_half = (FOUR_VECTOR_HALF2_HOST*) malloc(dim_cpu.space_mem/2);
						
	//~ int i;
	//~ for(i=0; i<dim_cpu.space_elem; i=i+1){
		//~ rv_cpu_half[i].v = half_float::half(rv_cpu[i].v);
		//~ rv_cpu_half[i].x = half_float::half(rv_cpu[i].x);			// get a number in the range 0.1 - 1.0
		//~ rv_cpu_half[i].y = half_float::half(rv_cpu[i].y);				// get a number in the range 0.1 - 1.0
		//~ rv_cpu_half[i].z = half_float::half(rv_cpu[i].z);			// get a number in the range 0.1 - 1.0
		
		//~ fv_cpu_half[i].v = half_float::half(fv_cpu[i].v);
		//~ fv_cpu_half[i].x = half_float::half(fv_cpu[i].x);			// get a number in the range 0.1 - 1.0
		//~ fv_cpu_half[i].y = half_float::half(fv_cpu[i].y);				// get a number in the range 0.1 - 1.0
		//~ fv_cpu_half[i].z = half_float::half(fv_cpu[i].z);			// get a number in the range 0.1 - 1.0
	//~ }

	int i;
	for(i=0; i<dim_cpu.space_elem/2; i=i+1){ //need to convert
		//rv_cpu_half[i].v = rv_cpu[i].v;
		//uint32_t haf2_valv = std::floats2half2(low_val,high_val);
		rv_cpu_half[i].v  = floats2half2(rv_cpu[i*2+1].v,rv_cpu[i*2].v);
		rv_cpu_half[i].x = floats2half2(rv_cpu[i*2+1].x,rv_cpu[i*2].x);
		rv_cpu_half[i].y = floats2half2(rv_cpu[i*2+1].y,rv_cpu[i*2].y);
		rv_cpu_half[i].z = floats2half2(rv_cpu[i*2+1].z,rv_cpu[i*2].z);
		
		fv_cpu_half[i].v  = floats2half2(fv_cpu[i*2+1].v,fv_cpu[i*2].v);
		fv_cpu_half[i].x = floats2half2(fv_cpu[i*2+1].x,fv_cpu[i*2].x);
		fv_cpu_half[i].y = floats2half2(fv_cpu[i*2+1].y,fv_cpu[i*2].y);
		fv_cpu_half[i].z = floats2half2(fv_cpu[i*2+1].z,fv_cpu[i*2].z);
	}
	
	for(i=0; i<dim_cpu.space_elem; i=i+1){
		qv_cpu_half[i] = half_float::half(qv_cpu[i]);			
	}

	hipMemcpy(	d_box_gpu, 
				box_cpu,
				dim_cpu.box_mem, 
				hipMemcpyHostToDevice);

	//==================================================50
	//	rv
	//==================================================50

	hipMemcpy(	d_rv_gpu,
				rv_cpu_half,
				dim_cpu.space_mem/2,
				hipMemcpyHostToDevice);

	//==================================================50
	//	qv
	//==================================================50

	hipMemcpy(	d_qv_gpu,
				qv_cpu_half,
				dim_cpu.space_mem2/2,
				hipMemcpyHostToDevice);

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY
	//====================================================================================================100

	//==================================================50
	//	fv
	//==================================================50

	hipMemcpy(	d_fv_gpu, 
				fv_cpu_half, 
				dim_cpu.space_mem/2, 
				hipMemcpyHostToDevice);

	time3 = get_time();

	//======================================================================================================================================================150
	//	KERNEL
	//======================================================================================================================================================150

	// launch kernel - all boxes
	kernel_gpu_cuda<<<blocks, threads>>>(	par_cpu,
											dim_cpu,
											d_box_gpu,
											d_rv_gpu,
											d_qv_gpu,
											d_fv_gpu);

	checkCUDAError("Start");
	hipDeviceSynchronize();

	time4 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)
	//======================================================================================================================================================150

	hipMemcpy(	fv_cpu_half, 
				d_fv_gpu, 
				dim_cpu.space_mem/2, 
				hipMemcpyDeviceToHost);

//convert back
	for(i=0; i<dim_cpu.space_elem/2; i=i+1){ //need to convert
		//rv_cpu_half[i].v = rv_cpu[i].v;
		//uint32_t haf2_valv = std::floats2half2(low_val,high_val);
		fv_cpu[i*2].v = half2high2float(fv_cpu_half[i].v);
		fv_cpu[i*2+1].v =  half2low2float(fv_cpu_half[i].v);
		
		fv_cpu[i*2].x = half2high2float(fv_cpu_half[i].x);
		fv_cpu[i*2+1].x =  half2low2float(fv_cpu_half[i].x);
		
		fv_cpu[i*2].y = half2high2float(fv_cpu_half[i].y);
		fv_cpu[i*2+1].y =  half2low2float(fv_cpu_half[i].y);
		
		fv_cpu[i*2].z = half2high2float(fv_cpu_half[i].z);
		fv_cpu[i*2+1].z =  half2low2float(fv_cpu_half[i].z);

	}
	
	//~ for(i=0; i<dim_cpu.space_elem/4; i=i+1){ 
	//~ printf( "%d %f %f %f %f\n", i, fv_cpu[i].v, fv_cpu[i].x, fv_cpu[i].y, fv_cpu[i].z);
	//~ }

	time5 = get_time();







	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150

	hipFree(d_rv_gpu);
	hipFree(d_qv_gpu);
	hipFree(d_fv_gpu);
	hipFree(d_box_gpu);

	time6 = get_time();
printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);


	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150
	//~ printf ("test teST test");
	/*
	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / 1000000);
	*/
}
