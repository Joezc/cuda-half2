#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200
//	plasmaKernel_gpu_2
//----------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------------200

__global__ void kernel_gpu_cuda(par_str d_par_gpu,
								dim_str d_dim_gpu,
								box_str* d_box_gpu,
								FOUR_VECTOR_HALF2* d_rv_gpu,
								half2* d_qv_gpu,
								FOUR_VECTOR_HALF2* d_fv_gpu)
{

	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	THREAD PARAMETERS
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	int bx = blockIdx.x;																// get current horizontal block index (0-n)
	int tx = threadIdx.x;															// get current horizontal thread index (0-n)
	// int ax = bx*NUMBER_THREADS+tx;
	// int wbx = bx;
	int wtx = tx;
	__half2 alpha_temp = __float2half2_rn(d_par_gpu.alpha);
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180
	//	DO FOR THE NUMBER OF BOXES
	//--------------------------------------------------------------------------------------------------------------------------------------------------------------------------180

	if(bx<d_dim_gpu.number_boxes){
	// while(wbx<box_indexes_counter){

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Extract input parameters
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// parameters
		__half2 a2 = __float2half2_rn(2.0*d_par_gpu.alpha*d_par_gpu.alpha);
		//__half a2 = __hmul(__hmul(alpha_temp, alpha_temp), __float2half(2.0));
		// home box
		int first_i;
		FOUR_VECTOR_HALF2* rA;
		FOUR_VECTOR_HALF2* fA;
		__shared__ FOUR_VECTOR_HALF2 rA_shared[100/2];

		// nei box
		int pointer;
		int k = 0;
		int first_j;
		FOUR_VECTOR_HALF2* rB;
		half2* qB;
		int j = 0;
		__shared__ FOUR_VECTOR_HALF rB_shared[100];
		__shared__ half qB_shared[100];

		// common
		//~ fp r2;
		//~ fp u2;
		//~ fp vij;
		//~ fp fs;
		//~ fp fxij;
		//~ fp fyij;
		//~ fp fzij;

		half2 r2;
		__half2 u2;
		__half2 vij;
		__half2 fs;
		half2 fxij;
		half2 fyij;
		__half2 fzij;

		THREE_VECTOR_HALF2 d;

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	Home box
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Setup parameters
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - box parameters
		first_i = d_box_gpu[bx].offset/2;
		rA = &d_rv_gpu[first_i];
		fA = &d_fv_gpu[first_i];

		//----------------------------------------------------------------------------------------------------------------------------------140
		//	Copy to shared memory
		//----------------------------------------------------------------------------------------------------------------------------------140

		// home box - shared memory
		while(wtx<NUMBER_PAR_PER_BOX/2){
			rA_shared[wtx] = rA[wtx];
			wtx = wtx + NUMBER_THREADS;
		}
		wtx = tx;

		// synchronize threads  - not needed, but just to be safe
		__syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

		// loop over neiing boxes of home box
		//~ k=0;{
		for (k=0; k<(1+d_box_gpu[bx].nn); k++){

			//----------------------------------------50
			//	nei box - get pointer to the right box
			//----------------------------------------50

			if(k==0){
				pointer = bx;													// set first box to be processed to home box
			}
			else{
				pointer = d_box_gpu[bx].nei[k-1].number;							// remaining boxes are nei boxes
			}

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - box parameters
			first_j = d_box_gpu[pointer].offset/2;

			// nei box - distance, (force), charge and (type) parameters
			rB = &d_rv_gpu[first_j];
			qB = &d_qv_gpu[first_j];

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Setup parameters
			//----------------------------------------------------------------------------------------------------------------------------------140

			// nei box - shared memory
			while(wtx<NUMBER_PAR_PER_BOX/2){
				//can use half2high2float and half2low2float, the same result.
				rB_shared[2*wtx].v = ((__half*)&(rB[wtx].v))[0]; // sthing changed, float4 vec
				rB_shared[2*wtx+1].v = ((__half*)&(rB[wtx].v))[1]; // sthing changed, float4 vec

				rB_shared[2*wtx].x = ((__half*)&(rB[wtx].x))[0]; // sthing changed, float4 vec
				rB_shared[2*wtx+1].x = ((__half*)&(rB[wtx].x))[1]; // sthing changed, float4 vec
				rB_shared[2*wtx].y = ((__half*)&(rB[wtx].y))[0]; // sthing changed, float4 vec
				rB_shared[2*wtx+1].y = ((__half*)&(rB[wtx].y))[1]; // sthing changed, float4 vec
				rB_shared[2*wtx].z = ((__half*)&(rB[wtx].z))[0]; // sthing changed, float4 vec
				rB_shared[2*wtx+1].z = ((__half*)&(rB[wtx].z))[1]; // sthing changed, float4 vec

				((half2*)qB_shared)[wtx] = qB[wtx]; // no change, 1d float vector
				wtx = wtx + NUMBER_THREADS;
			}
			wtx = tx;

			// synchronize threads because in next section each thread accesses data brought in by different threads here
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation
			//----------------------------------------------------------------------------------------------------------------------------------140

			// loop for the number of particles in the home box
			// for (int i=0; i<nTotal_i; i++){
			//~ {
			FOUR_VECTOR_HALF2 fA_tmp;
			while(wtx<NUMBER_PAR_PER_BOX/2){

				fA_tmp.v = fA[wtx].v;
				fA_tmp.x = fA[wtx].x;
				fA_tmp.y = fA[wtx].y;
				fA_tmp.z = fA[wtx].z;

				// loop for the number of particles in the current nei box
				//~ j=0;{
				for (j=0; j<NUMBER_PAR_PER_BOX; j++){

					// r2 = rA[wtx].v + rB[j].v - DOT(rA[wtx],rB[j]);
					// u2 = a2*r2;
					// vij= exp(-u2);
					// fs = 2.*vij;

					// d.x = rA[wtx].x  - rB[j].x;
					// fxij=fs*d.x;
					// d.y = rA[wtx].y  - rB[j].y;
					// fyij=fs*d.y;
					// d.z = rA[wtx].z  - rB[j].z;
					// fzij=fs*d.z;

					// fA[wtx].v +=  qB[j]*vij;
					// fA[wtx].x +=  qB[j]*fxij;
					// fA[wtx].y +=  qB[j]*fyij;
					// fA[wtx].z +=  qB[j]*fzij;



					r2 = rA_shared[wtx].v + rB_shared[j].v - DOT(rA_shared[wtx],rB_shared[j]);
					u2 = a2*r2;
					vij= exp(-u2);
					fs = 2*vij;

					d.x = rA_shared[wtx].x  - rB_shared[j].x;
					fxij=fs*d.x;
					d.y = rA_shared[wtx].y  - rB_shared[j].y;
					fyij=fs*d.y;
					d.z = rA_shared[wtx].z  - rB_shared[j].z;
					fzij=fs*d.z;

					//~ fA[wtx].v += (qB_shared[j]*vij);
					//~ fA[wtx].x += (qB_shared[j]*fxij);
					//~ fA[wtx].y += (qB_shared[j]*fyij);
					//~ fA[wtx].z += (qB_shared[j]*fzij);
					fA_tmp.v += (qB_shared[j]*vij);
					fA_tmp.x += (qB_shared[j]*fxij);
					fA_tmp.y += (qB_shared[j]*fyij);
					fA_tmp.z += (qB_shared[j]*fzij);
					//~ fA[wtx].v += __half2half2(rB_shared[j].v);
					//~ fA[wtx].x += __half2half2(rB_shared[j].x);
					//~ fA[wtx].y += __half2half2(rB_shared[j].y);
					//~ fA[wtx].z += __half2half2(rB_shared[j].z);
				}
				fA[wtx].v = fA_tmp.v;
				fA[wtx].x = fA_tmp.x;
				fA[wtx].y = fA_tmp.y;
				fA[wtx].z = fA_tmp.z;

				// increment work thread index
				wtx = wtx + NUMBER_THREADS;

			}

			// reset work index
			wtx = tx;

			// synchronize after finishing force contributions from current nei box not to cause conflicts when starting next box
			__syncthreads();

			//----------------------------------------------------------------------------------------------------------------------------------140
			//	Calculation END
			//----------------------------------------------------------------------------------------------------------------------------------140

		}

		// // increment work block index
		// wbx = wbx + NUMBER_BLOCKS;

		// // synchronize - because next iteration will overwrite current shared memory
		// __syncthreads();

		//------------------------------------------------------------------------------------------------------------------------------------------------------160
		//	nei box loop END
		//------------------------------------------------------------------------------------------------------------------------------------------------------160

	}

}
