#include "hip/hip_runtime.h"
//=====================================================================
//	MAIN FUNCTION
//=====================================================================
#undef fp
#define fp half
#include <math.h>

//~ #define max(x,y) ( (x) < (y) ? (y) : (x) )
//~ #define min(x,y) ( (x) < (y) ? (x) : (y) )

#define ATTEMPTS 12
#define MIN_SCALE_FACTOR 0.125
#define MAX_SCALE_FACTOR 4.0

#include <hip/hip_fp16.h>
#include "half_operator_overload.cuh"


__device__ void kernel_fin_2(	int timeinst,
													fp* d_initvalu,
													fp* d_finavalu,
													int offset_ecc,
													int offset_Dyad,
													int offset_SL,
													int offset_Cyt,
													fp* d_params,
													fp* d_com){

//=====================================================================
//	VARIABLES
//=====================================================================

	// input parameters
	fp BtotDyad;
	fp CaMKIItotDyad;

	// compute variables
	fp Vmyo;																			// [L]
	fp Vdyad;																			// [L]
	fp VSL;																				// [L]
	// fp kDyadSL;																			// [L/msec]
	fp kSLmyo;																			// [L/msec]
	fp k0Boff;																			// [s^-1]
	fp k0Bon;																			// [uM^-1 s^-1] kon = koff/Kd
	fp k2Boff;																			// [s^-1]
	fp k2Bon;																			// [uM^-1 s^-1]
	// fp k4Boff;																			// [s^-1]
	fp k4Bon;																			// [uM^-1 s^-1]
	fp CaMtotDyad;
	fp Bdyad;																			// [uM dyad]
	fp J_cam_dyadSL;																	// [uM/msec dyad]
	fp J_ca2cam_dyadSL;																	// [uM/msec dyad]
	fp J_ca4cam_dyadSL;																	// [uM/msec dyad]
	fp J_cam_SLmyo;																		// [umol/msec]
	fp J_ca2cam_SLmyo;																	// [umol/msec]
	fp J_ca4cam_SLmyo;																	// [umol/msec]

//=====================================================================
//	COMPUTATION
//=====================================================================

	// input parameters
	BtotDyad = d_params[1];
	CaMKIItotDyad = d_params[2];

	// ADJUST ECC incorporate Ca buffering from CaM, convert JCaCyt from uM/msec to mM/msec
	d_finavalu[offset_ecc+35] = d_finavalu[offset_ecc+35] + 1e-3*d_com[0];
	d_finavalu[offset_ecc+36] = d_finavalu[offset_ecc+36] + 1e-3*d_com[1];
	d_finavalu[offset_ecc+37] = d_finavalu[offset_ecc+37] + 1e-3*d_com[2];

	// incorporate CaM diffusion between compartments
	Vmyo = __float2half(2.1454e-11);																// [L]
	Vdyad = __float2half(1.7790e-14);																// [L]
	VSL = __float2half(6.6013e-13);																// [L]
	// kDyadSL = 3.6363e-16;															// [L/msec]
	kSLmyo = __float2half(8.587e-15);																// [L/msec]
	k0Boff = __float2half(0.0014);																// [s^-1]
	k0Bon = k0Boff/0.2;																// [uM^-1 s^-1] kon = koff/Kd
	k2Boff = k0Boff/100;															// [s^-1]
	k2Bon = k0Bon;																	// [uM^-1 s^-1]
	// k4Boff = k2Boff;																// [s^-1]
	k4Bon = k0Bon;																	// [uM^-1 s^-1]
	CaMtotDyad = d_initvalu[offset_Dyad+0]
			   + d_initvalu[offset_Dyad+1]
			   + d_initvalu[offset_Dyad+2]
			   + d_initvalu[offset_Dyad+3]
			   + d_initvalu[offset_Dyad+4]
			   + d_initvalu[offset_Dyad+5]
			   + CaMKIItotDyad * (	  d_initvalu[offset_Dyad+6]
												  + d_initvalu[offset_Dyad+7]
												  + d_initvalu[offset_Dyad+8]
												  + d_initvalu[offset_Dyad+9])
			   + d_initvalu[offset_Dyad+12]
			   + d_initvalu[offset_Dyad+13]
			   + d_initvalu[offset_Dyad+14];
	Bdyad = BtotDyad - CaMtotDyad;																				// [uM dyad]
	J_cam_dyadSL = 1e-3 * (  k0Boff*d_initvalu[offset_Dyad+0] - k0Bon*Bdyad*d_initvalu[offset_SL+0]);			// [uM/msec dyad]
	J_ca2cam_dyadSL = 1e-3 * (  k2Boff*d_initvalu[offset_Dyad+1] - k2Bon*Bdyad*d_initvalu[offset_SL+1]);		// [uM/msec dyad]
	J_ca4cam_dyadSL = 1e-3 * (  k2Boff*d_initvalu[offset_Dyad+2] - k4Bon*Bdyad*d_initvalu[offset_SL+2]);		// [uM/msec dyad]

	J_cam_SLmyo = kSLmyo * (  d_initvalu[offset_SL+0] - d_initvalu[offset_Cyt+0]);								// [umol/msec]
	J_ca2cam_SLmyo = kSLmyo * (  d_initvalu[offset_SL+1] - d_initvalu[offset_Cyt+1]);							// [umol/msec]
	J_ca4cam_SLmyo = kSLmyo * (  d_initvalu[offset_SL+2] - d_initvalu[offset_Cyt+2]);							// [umol/msec]

	// ADJUST CAM Dyad
	d_finavalu[offset_Dyad+0] = d_finavalu[offset_Dyad+0] - J_cam_dyadSL;
	d_finavalu[offset_Dyad+1] = d_finavalu[offset_Dyad+1] - J_ca2cam_dyadSL;
	d_finavalu[offset_Dyad+2] = d_finavalu[offset_Dyad+2] - J_ca4cam_dyadSL;

	// ADJUST CAM Sl
	d_finavalu[offset_SL+0] = d_finavalu[offset_SL+0] + J_cam_dyadSL*Vdyad/VSL - J_cam_SLmyo/VSL;
	d_finavalu[offset_SL+1] = d_finavalu[offset_SL+1] + J_ca2cam_dyadSL*Vdyad/VSL - J_ca2cam_SLmyo/VSL;
	d_finavalu[offset_SL+2] = d_finavalu[offset_SL+2] + J_ca4cam_dyadSL*Vdyad/VSL - J_ca4cam_SLmyo/VSL;

	// ADJUST CAM Cyt
	d_finavalu[offset_Cyt+0] = d_finavalu[offset_Cyt+0] + J_cam_SLmyo/Vmyo;
	d_finavalu[offset_Cyt+1] = d_finavalu[offset_Cyt+1] + J_ca2cam_SLmyo/Vmyo;
	d_finavalu[offset_Cyt+2] = d_finavalu[offset_Cyt+2] + J_ca4cam_SLmyo/Vmyo;

}
//=====================================================================
//	MAIN FUNCTION
//=====================================================================
__device__ void kernel_ecc_2(	fp timeinst,
													fp* d_initvalu,
													fp* d_finavalu,
													int valu_offset,
													fp* d_params){

	//=====================================================================
	//	VARIABLES
	//=====================================================================

	// input parameters
	fp cycleLength;

	// variable references				// GET VARIABLES FROM MEMORY AND SAVE LOCALLY !!!!!!!!!!!!!!!!!!
	int offset_1;
	int offset_2;
	int offset_3;
	int offset_4;
	int offset_5;
	int offset_6;
	int offset_7;
	int offset_8;
	int offset_9;
	int offset_10;
	int offset_11;
	int offset_12;
	int offset_13;
	int offset_14;
	int offset_15;
	int offset_16;
	int offset_17;
	int offset_18;
	int offset_19;
	int offset_20;
	int offset_21;
	int offset_22;
	int offset_23;
	int offset_24;
	int offset_25;
	int offset_26;
	int offset_27;
	int offset_28;
	int offset_29;
	int offset_30;
	int offset_31;
	int offset_32;
	int offset_33;
	int offset_34;
	int offset_35;
	int offset_36;
	int offset_37;
	int offset_38;
	int offset_39;
	int offset_40;
	int offset_41;
	int offset_42;
	int offset_43;
	int offset_44;
	int offset_45;
	int offset_46;

	// stored input array
	fp d_initvalu_1;
	fp d_initvalu_2;
	fp d_initvalu_3;
	fp d_initvalu_4;
	fp d_initvalu_5;
	fp d_initvalu_6;
	fp d_initvalu_7;
	fp d_initvalu_8;
	fp d_initvalu_9;
	fp d_initvalu_10;
	fp d_initvalu_11;
	fp d_initvalu_12;
	fp d_initvalu_13;
	fp d_initvalu_14;
	fp d_initvalu_15;
	fp d_initvalu_16;
	fp d_initvalu_17;
	fp d_initvalu_18;
	fp d_initvalu_19;
	fp d_initvalu_20;
	fp d_initvalu_21;
	// fp d_initvalu_22;
	fp d_initvalu_23;
	fp d_initvalu_24;
	fp d_initvalu_25;
	fp d_initvalu_26;
	fp d_initvalu_27;
	fp d_initvalu_28;
	fp d_initvalu_29;
	fp d_initvalu_30;
	fp d_initvalu_31;
	fp d_initvalu_32;
	fp d_initvalu_33;
	fp d_initvalu_34;
	fp d_initvalu_35;
	fp d_initvalu_36;
	fp d_initvalu_37;
	fp d_initvalu_38;
	fp d_initvalu_39;
	fp d_initvalu_40;
	// fp d_initvalu_41;
	// fp d_initvalu_42;
	// fp d_initvalu_43;
	// fp d_initvalu_44;
	// fp d_initvalu_45;
	// fp d_initvalu_46;

	// matlab constants undefined in c
	fp pi;

	// Constants
	fp R;																			// [J/kmol*K]
	fp Frdy;																		// [C/mol]
	fp Temp;																		// [K] 310
	fp FoRT;																		//
	fp Cmem;																		// [F] membrane capacitance
	fp Qpow;

	// Cell geometry
	fp cellLength;																	// cell length [um]
	fp cellRadius;																	// cell radius [um]
	// fp junctionLength;																// junc length [um]
	// fp junctionRadius;																// junc radius [um]
	// fp distSLcyto;																	// dist. SL to cytosol [um]
	// fp distJuncSL;																	// dist. junc to SL [um]
	// fp DcaJuncSL;																	// Dca junc to SL [cm^2/sec]
	// fp DcaSLcyto;																	// Dca SL to cyto [cm^2/sec]
	// fp DnaJuncSL;																	// Dna junc to SL [cm^2/sec]
	// fp DnaSLcyto;																	// Dna SL to cyto [cm^2/sec]
	fp Vcell;																		// [L]
	fp Vmyo;
	fp Vsr;
	fp Vsl;
	fp Vjunc;
	// fp SAjunc;																		// [um^2]
	// fp SAsl;																		// [um^2]
	fp J_ca_juncsl;																	// [L/msec]
	fp J_ca_slmyo;																	// [L/msec]
	fp J_na_juncsl;																	// [L/msec]
	fp J_na_slmyo;																	// [L/msec]

	// Fractional currents in compartments
	fp Fjunc;
	fp Fsl;
	fp Fjunc_CaL;
	fp Fsl_CaL;

	// Fixed ion concentrations
	fp Cli;																			// Intracellular Cl  [mM]
	fp Clo;																			// Extracellular Cl  [mM]
	fp Ko;																			// Extracellular K   [mM]
	fp Nao;																			// Extracellular Na  [mM]
	fp Cao;																			// Extracellular Ca  [mM]
	fp Mgi;																			// Intracellular Mg  [mM]

	// Nernst Potentials
	fp ena_junc;																	// [mV]
	fp ena_sl;																		// [mV]
	fp ek;																			// [mV]
	fp eca_junc;																	// [mV]
	fp eca_sl;																		// [mV]
	fp ecl;																			// [mV]

	// Na transport parameters
	fp GNa;																			// [mS/uF]
	fp GNaB;																		// [mS/uF]
	fp IbarNaK;																		// [uA/uF]
	fp KmNaip;																		// [mM]
	fp KmKo;																		// [mM]
	// fp Q10NaK;
	// fp Q10KmNai;

	// K current parameters
	fp pNaK;
	fp GtoSlow;																		// [mS/uF]
	fp GtoFast;																		// [mS/uF]
	fp gkp;

	// Cl current parameters
	fp GClCa;																		// [mS/uF]
	fp GClB;																		// [mS/uF]
	fp KdClCa;																		// [mM]																// [mM]

	// I_Ca parameters
	fp pNa;																			// [cm/sec]
	fp pCa;																			// [cm/sec]
	fp pK;																			// [cm/sec]
	// fp KmCa;																		// [mM]
	fp Q10CaL;

	// Ca transport parameters
	fp IbarNCX;																		// [uA/uF]
	fp KmCai;																		// [mM]
	fp KmCao;																		// [mM]
	fp KmNai;																		// [mM]
	fp KmNao;																		// [mM]
	fp ksat;																			// [none]
	fp nu;																			// [none]
	fp Kdact;																		// [mM]
	fp Q10NCX;																		// [none]
	fp IbarSLCaP;																	// [uA/uF]
	fp KmPCa;																		// [mM]
	fp GCaB;																		// [uA/uF]
	fp Q10SLCaP;																	// [none]																	// [none]

	// SR flux parameters
	fp Q10SRCaP;																	// [none]
	fp Vmax_SRCaP;																	// [mM/msec] (mmol/L cytosol/msec)
	fp Kmf;																			// [mM]
	fp Kmr;																			// [mM]L cytosol
	fp hillSRCaP;																	// [mM]
	fp ks;																			// [1/ms]
	fp koCa;																		// [mM^-2 1/ms]
	fp kom;																			// [1/ms]
	fp kiCa;																		// [1/mM/ms]
	fp kim;																			// [1/ms]
	fp ec50SR;																		// [mM]

	// Buffering parameters
	fp Bmax_Naj;																	// [mM]
	fp Bmax_Nasl;																	// [mM]
	fp koff_na;																		// [1/ms]
	fp kon_na;																		// [1/mM/ms]
	fp Bmax_TnClow;																	// [mM], TnC low affinity
	fp koff_tncl;																	// [1/ms]
	fp kon_tncl;																	// [1/mM/ms]
	fp Bmax_TnChigh;																// [mM], TnC high affinity
	fp koff_tnchca;																	// [1/ms]
	fp kon_tnchca;																	// [1/mM/ms]
	fp koff_tnchmg;																	// [1/ms]
	fp kon_tnchmg;																	// [1/mM/ms]
	// fp Bmax_CaM;																	// [mM], CaM buffering
	// fp koff_cam;																	// [1/ms]
	// fp kon_cam;																		// [1/mM/ms]
	fp Bmax_myosin;																	// [mM], Myosin buffering
	fp koff_myoca;																	// [1/ms]
	fp kon_myoca;																	// [1/mM/ms]
	fp koff_myomg;																	// [1/ms]
	fp kon_myomg;																	// [1/mM/ms]
	fp Bmax_SR;																		// [mM]
	fp koff_sr;																		// [1/ms]
	fp kon_sr;																		// [1/mM/ms]
	fp Bmax_SLlowsl;																// [mM], SL buffering
	fp Bmax_SLlowj;																	// [mM]
	fp koff_sll;																	// [1/ms]
	fp kon_sll;																		// [1/mM/ms]
	fp Bmax_SLhighsl;																// [mM]
	fp Bmax_SLhighj;																// [mM]
	fp koff_slh;																	// [1/ms]
	fp kon_slh;																		// [1/mM/ms]
	fp Bmax_Csqn;																	// 140e-3*Vmyo/Vsr; [mM]
	fp koff_csqn;																	// [1/ms]
	fp kon_csqn;																	// [1/mM/ms]

	// I_Na: Fast Na Current
	fp am;
	fp bm;
	fp ah;
	fp bh;
	fp aj;
	fp bj;
	fp I_Na_junc;
	fp I_Na_sl;
	// fp I_Na;

	// I_nabk: Na Background Current
	fp I_nabk_junc;
	fp I_nabk_sl;
	// fp I_nabk;

	// I_nak: Na/K Pump Current
	fp sigma;
	fp fnak;
	fp I_nak_junc;
	fp I_nak_sl;
	fp I_nak;

	// I_kr: Rapidly Activating K Current
	fp gkr;
	fp xrss;
	fp tauxr;
	fp rkr;
	fp I_kr;

	// I_ks: Slowly Activating K Current
	fp pcaks_junc;
	fp pcaks_sl;
	fp gks_junc;
	fp gks_sl;
	fp eks;
	fp xsss;
	fp tauxs;
	fp I_ks_junc;
	fp I_ks_sl;
	fp I_ks;

	// I_kp: Plateau K current
	fp kp_kp;
	fp I_kp_junc;
	fp I_kp_sl;
	fp I_kp;

	// I_to: Transient Outward K Current (slow and fast components)
	fp xtoss;
	fp ytoss;
	fp rtoss;
	fp tauxtos;
	fp tauytos;
	fp taurtos;
	fp I_tos;

	//
	fp tauxtof;
	fp tauytof;
	fp I_tof;
	fp I_to;

	// I_ki: Time-Independent K Current
	fp aki;
	fp bki;
	fp kiss;
	fp I_ki;

	// I_ClCa: Ca-activated Cl Current, I_Clbk: background Cl Current
	fp I_ClCa_junc;
	fp I_ClCa_sl;
	fp I_ClCa;
	fp I_Clbk;

	// I_Ca: L-type Calcium Current
	fp dss;
	fp taud;
	fp fss;
	fp tauf;

	//
	fp ibarca_j;
	fp ibarca_sl;
	fp ibark;
	fp ibarna_j;
	fp ibarna_sl;
	fp I_Ca_junc;
	fp I_Ca_sl;
	fp I_Ca;
	fp I_CaK;
	fp I_CaNa_junc;
	fp I_CaNa_sl;
	// fp I_CaNa;
	// fp I_Catot;

	// I_ncx: Na/Ca Exchanger flux
	fp Ka_junc;
	fp Ka_sl;
	fp s1_junc;
	fp s1_sl;
	fp s2_junc;
	fp s3_junc;
	fp s2_sl;
	fp s3_sl;
	fp I_ncx_junc;
	fp I_ncx_sl;
	fp I_ncx;

	// I_pca: Sarcolemmal Ca Pump Current
	fp I_pca_junc;
	fp I_pca_sl;
	fp I_pca;

	// I_cabk: Ca Background Current
	fp I_cabk_junc;
	fp I_cabk_sl;
	fp I_cabk;

	// SR fluxes: Calcium Release, SR Ca pump, SR Ca leak
	fp MaxSR;
	fp MinSR;
	fp kCaSR;
	fp koSRCa;
	fp kiSRCa;
	fp RI;
	fp J_SRCarel;																	// [mM/ms]
	fp J_serca;
	fp J_SRleak;																		//   [mM/ms]

	// Cytosolic Ca Buffers
	fp J_CaB_cytosol;

	// Junctional and SL Ca Buffers
	fp J_CaB_junction;
	fp J_CaB_sl;

	// SR Ca Concentrations
	fp oneovervsr;

	// Sodium Concentrations
	fp I_Na_tot_junc;																// [uA/uF]
	fp I_Na_tot_sl;																	// [uA/uF]
	fp oneovervsl;

	// Potassium Concentration
	fp I_K_tot;

	// Calcium Concentrations
	fp I_Ca_tot_junc;																// [uA/uF]
	fp I_Ca_tot_sl;																	// [uA/uF]
	// fp junc_sl;
	// fp sl_junc;
	// fp sl_myo;
	// fp myo_sl;

	//	Simulation type
	int state;																			// 0-none; 1-pace; 2-vclamp
	fp I_app;
	fp V_hold;
	fp V_test;
	fp V_clamp;
	fp R_clamp;

	//	Membrane Potential
	fp I_Na_tot;																		// [uA/uF]
	fp I_Cl_tot;																		// [uA/uF]
	fp I_Ca_tot;
	fp I_tot;

	//=====================================================================
	//	EXECUTION
	//=====================================================================

	// input parameters
	cycleLength = d_params[15];

	// variable references
	offset_1 = valu_offset;
	offset_2 = valu_offset+1;
	offset_3 = valu_offset+2;
	offset_4 = valu_offset+3;
	offset_5 = valu_offset+4;
	offset_6 = valu_offset+5;
	offset_7 = valu_offset+6;
	offset_8 = valu_offset+7;
	offset_9 = valu_offset+8;
	offset_10 = valu_offset+9;
	offset_11 = valu_offset+10;
	offset_12 = valu_offset+11;
	offset_13 = valu_offset+12;
	offset_14 = valu_offset+13;
	offset_15 = valu_offset+14;
	offset_16 = valu_offset+15;
	offset_17 = valu_offset+16;
	offset_18 = valu_offset+17;
	offset_19 = valu_offset+18;
	offset_20 = valu_offset+19;
	offset_21 = valu_offset+20;
	offset_22 = valu_offset+21;
	offset_23 = valu_offset+22;
	offset_24 = valu_offset+23;
	offset_25 = valu_offset+24;
	offset_26 = valu_offset+25;
	offset_27 = valu_offset+26;
	offset_28 = valu_offset+27;
	offset_29 = valu_offset+28;
	offset_30 = valu_offset+29;
	offset_31 = valu_offset+30;
	offset_32 = valu_offset+31;
	offset_33 = valu_offset+32;
	offset_34 = valu_offset+33;
	offset_35 = valu_offset+34;
	offset_36 = valu_offset+35;
	offset_37 = valu_offset+36;
	offset_38 = valu_offset+37;
	offset_39 = valu_offset+38;
	offset_40 = valu_offset+39;
	offset_41 = valu_offset+40;
	offset_42 = valu_offset+41;
	offset_43 = valu_offset+42;
	offset_44 = valu_offset+43;
	offset_45 = valu_offset+44;
	offset_46 = valu_offset+45;

	// stored input array
	d_initvalu_1 = d_initvalu[offset_1];
	d_initvalu_2 = d_initvalu[offset_2];
	d_initvalu_3 = d_initvalu[offset_3];
	d_initvalu_4 = d_initvalu[offset_4];
	d_initvalu_5 = d_initvalu[offset_5];
	d_initvalu_6 = d_initvalu[offset_6];
	d_initvalu_7 = d_initvalu[offset_7];
	d_initvalu_8 = d_initvalu[offset_8];
	d_initvalu_9 = d_initvalu[offset_9];
	d_initvalu_10 = d_initvalu[offset_10];
	d_initvalu_11 = d_initvalu[offset_11];
	d_initvalu_12 = d_initvalu[offset_12];
	d_initvalu_13 = d_initvalu[offset_13];
	d_initvalu_14 = d_initvalu[offset_14];
	d_initvalu_15 = d_initvalu[offset_15];
	d_initvalu_16 = d_initvalu[offset_16];
	d_initvalu_17 = d_initvalu[offset_17];
	d_initvalu_18 = d_initvalu[offset_18];
	d_initvalu_19 = d_initvalu[offset_19];
	d_initvalu_20 = d_initvalu[offset_20];
	d_initvalu_21 = d_initvalu[offset_21];
	// d_initvalu_22 = d_initvalu[offset_22];
	d_initvalu_23 = d_initvalu[offset_23];
	d_initvalu_24 = d_initvalu[offset_24];
	d_initvalu_25 = d_initvalu[offset_25];
	d_initvalu_26 = d_initvalu[offset_26];
	d_initvalu_27 = d_initvalu[offset_27];
	d_initvalu_28 = d_initvalu[offset_28];
	d_initvalu_29 = d_initvalu[offset_29];
	d_initvalu_30 = d_initvalu[offset_30];
	d_initvalu_31 = d_initvalu[offset_31];
	d_initvalu_32 = d_initvalu[offset_32];
	d_initvalu_33 = d_initvalu[offset_33];
	d_initvalu_34 = d_initvalu[offset_34];
	d_initvalu_35 = d_initvalu[offset_35];
	d_initvalu_36 = d_initvalu[offset_36];
	d_initvalu_37 = d_initvalu[offset_37];
	d_initvalu_38 = d_initvalu[offset_38];
	d_initvalu_39 = d_initvalu[offset_39];
	d_initvalu_40 = d_initvalu[offset_40];
	// d_initvalu_41 = d_initvalu[offset_41];
	// d_initvalu_42 = d_initvalu[offset_42];
	// d_initvalu_43 = d_initvalu[offset_43];
	// d_initvalu_44 = d_initvalu[offset_44];
	// d_initvalu_45 = d_initvalu[offset_45];
	// d_initvalu_46 = d_initvalu[offset_46];

	// matlab constants undefined in c
	pi = __float2half(3.1416);

	// Constants
	R = __float2half(8314);																			// [J/kmol*K]
	Frdy = __float2half(96485);																		// [C/mol]
	Temp = __float2half(310);																			// [K] 310
	FoRT = Frdy/R/Temp;																	//
	Cmem = __float2half(1.3810e-10);																	// [F] membrane capacitance
	Qpow = (Temp-310)/10;

	// Cell geometry
	cellLength = __float2half(100);																	// cell length [um]
	cellRadius = __float2half(10.25);																	// cell radius [um]
	// junctionLength = 160e-3;															// junc length [um]
	// junctionRadius = 15e-3;																// junc radius [um]
	// distSLcyto = 0.45;																	// dist. SL to cytosol [um]
	// distJuncSL = 0.5;																	// dist. junc to SL [um]
	// DcaJuncSL = 1.64e-6;																// Dca junc to SL [cm^2/sec]
	// DcaSLcyto = 1.22e-6;																// Dca SL to cyto [cm^2/sec]
	// DnaJuncSL = 1.09e-5;																// Dna junc to SL [cm^2/sec]
	// DnaSLcyto = 1.79e-5;																// Dna SL to cyto [cm^2/sec]
	//Vcell = pi*pow(cellRadius,2)*cellLength*1e-15;											// [L]
	Vcell = pi*cellRadius*cellRadius*cellLength*1e-15;
	Vmyo = 0.65*Vcell;
	Vsr = 0.035*Vcell;
	Vsl = 0.02*Vcell;
	Vjunc = 0.0539*0.01*Vcell;
	// SAjunc = 20150*pi*2*junctionLength*junctionRadius;									// [um^2]
	// SAsl = pi*2*cellRadius*cellLength;													// [um^2]
	J_ca_juncsl = __float2half(1/1.2134e12);															// [L/msec]
	J_ca_slmyo = __float2half(1/2.68510e11);															// [L/msec]
	J_na_juncsl = __float2half(1/(1.6382e12/3*100));													// [L/msec]
	J_na_slmyo = __float2half(1/(1.8308e10/3*100));													// [L/msec]

	// Fractional currents in compartments
	Fjunc = __float2half(0.11);
	Fsl = 1-Fjunc;
	Fjunc_CaL = __float2half(0.9);
	Fsl_CaL = 1-Fjunc_CaL;

	// Fixed ion concentrations
	Cli = __float2half(15);																			// Intracellular Cl  [mM]
	Clo = __float2half(150);																			// Extracellular Cl  [mM]
	Ko = __float2half(5.4);																			// Extracellular K   [mM]
	Nao = __float2half(140);																			// Extracellular Na  [mM]
	Cao = __float2half(1.8);																			// Extracellular Ca  [mM]
	Mgi = __float2half(1);																			// Intracellular Mg  [mM]

	// Nernst Potentials
	ena_junc = (1/FoRT)*log(Nao/d_initvalu_32);													// [mV]
	ena_sl = (1/FoRT)*log(Nao/d_initvalu_33);													// [mV]
	ek = (1/FoRT)*log(Ko/d_initvalu_35);														// [mV]
	eca_junc = (1/FoRT/2)*log(Cao/d_initvalu_36);												// [mV]
	eca_sl = (1/FoRT/2)*log(Cao/d_initvalu_37);													// [mV]
	ecl = (1/FoRT)*log(Cli/Clo);														// [mV]

	// Na transport parameters
	GNa =  __float2half(16.0);																		// [mS/uF]
	GNaB = __float2half(0.297e-3);																	// [mS/uF]
	IbarNaK = __float2half(1.90719);																	// [uA/uF]
	KmNaip = __float2half(11);																		// [mM]
	KmKo = __float2half(1.5);																			// [mM]
	// Q10NaK = 1.63;
	// Q10KmNai = 1.39;

	// K current parameters
	pNaK = __float2half(0.01833);
	GtoSlow = __float2half(0.06);																		// [mS/uF]
	GtoFast = __float2half(0.02);																		// [mS/uF]
	gkp = __float2half(0.001);

	// Cl current parameters
	GClCa = __float2half(0.109625);																	// [mS/uF]
	GClB = __float2half(9e-3);																		// [mS/uF]
	KdClCa = __float2half(100e-3);																	// [mM]

	// I_Ca parameters
	pNa = __float2half(1.5e-8);																		// [cm/sec]
	pCa = __float2half(5.4e-4);																		// [cm/sec]
	pK = __float2half(2.7e-7);																		// [cm/sec]
	// KmCa = 0.6e-3;																		// [mM]
	Q10CaL = __float2half(1.8);

	// Ca transport parameters
	IbarNCX = __float2half(9.0);																		// [uA/uF]
	KmCai = __float2half(3.59e-3);																	// [mM]
	KmCao = __float2half(1.3);																		// [mM]
	KmNai = __float2half(12.29);																		// [mM]
	KmNao = __float2half(87.5);																		// [mM]
	ksat = __float2half(0.27);																		// [none]
	nu = __float2half(0.35);																			// [none]
	Kdact = __float2half(0.256e-3);																	// [mM]
	Q10NCX = __float2half(1.57);																		// [none]
	IbarSLCaP = __float2half(0.0673);																	// [uA/uF]
	KmPCa = __float2half(0.5e-3);																		// [mM]
	GCaB = __float2half(2.513e-4);																	// [uA/uF]
	Q10SLCaP = __float2half(2.35);																	// [none]

	// SR flux parameters
	Q10SRCaP = __float2half(2.6);																		// [none]
	Vmax_SRCaP = __float2half(2.86e-4);																// [mM/msec] (mmol/L cytosol/msec)
	Kmf = __float2half(0.246e-3);																		// [mM]
	Kmr = __float2half(1.7);																			// [mM]L cytosol
	hillSRCaP = __float2half(1.787);																	// [mM]
	ks = __float2half(25);																			// [1/ms]
	koCa = __float2half(10);																			// [mM^-2 1/ms]
	kom = __float2half(0.06);																			// [1/ms]
	kiCa = __float2half(0.5);																			// [1/mM/ms]
	kim = __float2half(0.005);																		// [1/ms]
	ec50SR =__float2half( 0.45);																		// [mM]

	// Buffering parameters
	Bmax_Naj = __float2half(7.561);																	// [mM]
	Bmax_Nasl = __float2half(1.65);																	// [mM]
	koff_na = __float2half(1e-3);																		// [1/ms]
	kon_na = __float2half(0.1e-3);																	// [1/mM/ms]
	Bmax_TnClow = __float2half(70e-3);																// [mM], TnC low affinity
	koff_tncl = __float2half(19.6e-3);																// [1/ms]
	kon_tncl = __float2half(32.7);																	// [1/mM/ms]
	Bmax_TnChigh = __float2half(140e-3);																// [mM], TnC high affinity
	koff_tnchca = __float2half(0.032e-3);																// [1/ms]
	kon_tnchca = __float2half(2.37);																	// [1/mM/ms]
	koff_tnchmg = __float2half(3.33e-3);																// [1/ms]
	kon_tnchmg = __float2half(3e-3);																	// [1/mM/ms]
	// Bmax_CaM = 24e-3);																	// [mM], CaM buffering
	// koff_cam = 238e-3);																	// [1/ms]
	// kon_cam = 34);																		// [1/mM/ms]
	Bmax_myosin = __float2half(140e-3);																// [mM], Myosin buffering
	koff_myoca = __float2half(0.46e-3);																// [1/ms]
	kon_myoca = __float2half(13.8);																	// [1/mM/ms]
	koff_myomg = __float2half(0.057e-3);																// [1/ms]
	kon_myomg = __float2half(0.0157);																	// [1/mM/ms]
	Bmax_SR = __float2half(19*0.9e-3);																	// [mM]
	koff_sr = __float2half(60e-3);																	// [1/ms]
	kon_sr = __float2half(100);																		// [1/mM/ms]
	Bmax_SLlowsl = 37.38e-3*Vmyo/Vsl;													// [mM], SL buffering
	Bmax_SLlowj = 4.62e-3*Vmyo/Vjunc*0.1;												// [mM]
	koff_sll = __float2half(1300e-3);																	// [1/ms]
	kon_sll = __float2half(100);																		// [1/mM/ms]
	Bmax_SLhighsl = 13.35e-3*Vmyo/Vsl;													// [mM]
	Bmax_SLhighj = 1.65e-3*Vmyo/Vjunc*0.1;												// [mM]
	koff_slh = __float2half(30e-3);																	// [1/ms]
	kon_slh = __float2half(100);																		// [1/mM/ms]
	Bmax_Csqn = __float2half(2.7);																	// 140e-3*Vmyo/Vsr); [mM]
	koff_csqn = __float2half(65);																		// [1/ms]
	kon_csqn = __float2half(100);																		// [1/mM/ms]

	// I_Na: Fast Na Current
	am = 0.32*(d_initvalu_39+47.13)/(1-exp(-0.1*(d_initvalu_39+47.13)));
	bm = 0.08*exp(-d_initvalu_39/11);
	if(d_initvalu_39 >= -40.0){
		ah = __float2half(0); aj = __float2half(0);
		bh = 1/(0.13*(1+exp(-(d_initvalu_39+10.66)/11.1)));
		bj = 0.3*exp(-2.535e-7*d_initvalu_39)/(1+exp(-0.1*(d_initvalu_39+32)));
	}
	else{
		ah = 0.135*exp((80+d_initvalu_39)/-6.8);
		bh = 3.56*exp(0.079*d_initvalu_39)+3.1e5*exp(0.35*d_initvalu_39);
		aj = (-127140*exp(0.2444*d_initvalu_39)-3.474e-5*exp(-0.04391*d_initvalu_39))*(d_initvalu_39+37.78)/(1+exp(0.311*(d_initvalu_39+79.23)));
		bj = 0.1212*exp(-0.01052*d_initvalu_39)/(1+exp(-0.1378*(d_initvalu_39+40.14)));
	}
	d_finavalu[offset_1] = am*(1-d_initvalu_1)-bm*d_initvalu_1;
	d_finavalu[offset_2] = ah*(1-d_initvalu_2)-bh*d_initvalu_2;
	d_finavalu[offset_3] = aj*(1-d_initvalu_3)-bj*d_initvalu_3;
	I_Na_junc = Fjunc*GNa*pow(d_initvalu_1,3)*d_initvalu_2*d_initvalu_3*(d_initvalu_39-ena_junc);
	I_Na_sl = Fsl*GNa*pow(d_initvalu_1,3)*d_initvalu_2*d_initvalu_3*(d_initvalu_39-ena_sl);
	// I_Na = I_Na_junc+I_Na_sl;

	// I_nabk: Na Background Current
	I_nabk_junc = Fjunc*GNaB*(d_initvalu_39-ena_junc);
	I_nabk_sl = Fsl*GNaB*(d_initvalu_39-ena_sl);
	// I_nabk = I_nabk_junc+I_nabk_sl;

	// I_nak: Na/K Pump Current
	sigma = (exp(Nao/67.3)-1)/7;
	fnak = 1/(1+0.1245*exp(-0.1*d_initvalu_39*FoRT)+0.0365*sigma*exp(-d_initvalu_39*FoRT));
	I_nak_junc = Fjunc*IbarNaK*fnak*Ko /(1+pow((KmNaip/d_initvalu_32),4)) /(Ko+KmKo);
	I_nak_sl = Fsl*IbarNaK*fnak*Ko /(1+pow((KmNaip/d_initvalu_33),4)) /(Ko+KmKo);
	I_nak = I_nak_junc+I_nak_sl;

	// I_kr: Rapidly Activating K Current
	gkr = 0.03*sqrt(Ko/5.4);
	xrss = 1/(1+exp(-(d_initvalu_39+50)/7.5));
	tauxr = 1/(0.00138*(d_initvalu_39+7)/(1-exp(-0.123*(d_initvalu_39+7)))+6.1e-4*(d_initvalu_39+10)/(exp(0.145*(d_initvalu_39+10))-1));
	d_finavalu[offset_12] = (xrss-d_initvalu_12)/tauxr;
	rkr = 1/(1+exp((d_initvalu_39+33)/22.4));
	I_kr = gkr*d_initvalu_12*rkr*(d_initvalu_39-ek);

	// I_ks: Slowly Activating K Current
	pcaks_junc = -log10(d_initvalu_36)+3.0;
	pcaks_sl = -log10(d_initvalu_37)+3.0;
	gks_junc = 0.07*(0.057 +0.19/(1+ exp((-7.2+pcaks_junc)/0.6)));
	gks_sl = 0.07*(0.057 +0.19/(1+ exp((-7.2+pcaks_sl)/0.6)));
	eks = (1/FoRT)*log((Ko+pNaK*Nao)/(d_initvalu_35+pNaK*d_initvalu_34));
	xsss = 1/(1+exp(-(d_initvalu_39-1.5)/16.7));
	tauxs = 1/(7.19e-5*(d_initvalu_39+30)/(1-exp(-0.148*(d_initvalu_39+30)))+1.31e-4*(d_initvalu_39+30)/(exp(0.0687*(d_initvalu_39+30))-1));
	d_finavalu[offset_13] = (xsss-d_initvalu_13)/tauxs;
	I_ks_junc = Fjunc*gks_junc*pow(d_initvalu_12,2)*(d_initvalu_39-eks);
	I_ks_sl = Fsl*gks_sl*pow(d_initvalu_13,2)*(d_initvalu_39-eks);
	I_ks = I_ks_junc+I_ks_sl;

	// I_kp: Plateau K current
	kp_kp = 1/(1+exp(7.488-d_initvalu_39/5.98));
	I_kp_junc = Fjunc*gkp*kp_kp*(d_initvalu_39-ek);
	I_kp_sl = Fsl*gkp*kp_kp*(d_initvalu_39-ek);
	I_kp = I_kp_junc+I_kp_sl;

	// I_to: Transient Outward K Current (slow and fast components)
	xtoss = 1/(1+exp(-(d_initvalu_39+3.0)/15));
	ytoss = 1/(1+exp((d_initvalu_39+33.5)/10));
	rtoss = 1/(1+exp((d_initvalu_39+33.5)/10));
	tauxtos = 9/(1+exp((d_initvalu_39+3.0)/15))+0.5;
	tauytos = 3e3/(1+exp((d_initvalu_39+60.0)/10))+30;
	taurtos = 2800/(1+exp((d_initvalu_39+60.0)/10))+220;
	d_finavalu[offset_8] = (xtoss-d_initvalu_8)/tauxtos;
	d_finavalu[offset_9] = (ytoss-d_initvalu_9)/tauytos;
	d_finavalu[offset_40]= (rtoss-d_initvalu_40)/taurtos;
	I_tos = GtoSlow*d_initvalu_8*(d_initvalu_9+0.5*d_initvalu_40)*(d_initvalu_39-ek);									// [uA/uF]

	//
	tauxtof = 3.5*exp(-d_initvalu_39*d_initvalu_39/30/30)+1.5;
	tauytof = 20.0/(1+exp((d_initvalu_39+33.5)/10))+20.0;
	d_finavalu[offset_10] = (xtoss-d_initvalu_10)/tauxtof;
	d_finavalu[offset_11] = (ytoss-d_initvalu_11)/tauytof;
	I_tof = GtoFast*d_initvalu_10*d_initvalu_11*(d_initvalu_39-ek);
	I_to = I_tos + I_tof;

	// I_ki: Time-Independent K Current
	aki = 1.02/(1+exp(0.2385*(d_initvalu_39-ek-59.215)));
	bki =(0.49124*exp(0.08032*(d_initvalu_39+5.476-ek)) + exp(0.06175*(d_initvalu_39-ek-594.31))) /(1 + exp(-0.5143*(d_initvalu_39-ek+4.753)));
	kiss = aki/(aki+bki);
	I_ki = 0.9*sqrt(Ko/5.4)*kiss*(d_initvalu_39-ek);

	// I_ClCa: Ca-activated Cl Current, I_Clbk: background Cl Current
	I_ClCa_junc = Fjunc*GClCa/(1+KdClCa/d_initvalu_36)*(d_initvalu_39-ecl);
	I_ClCa_sl = Fsl*GClCa/(1+KdClCa/d_initvalu_37)*(d_initvalu_39-ecl);
	I_ClCa = I_ClCa_junc+I_ClCa_sl;
	I_Clbk = GClB*(d_initvalu_39-ecl);

	// I_Ca: L-type Calcium Current
	dss = 1/(1+exp(-(d_initvalu_39+14.5)/6.0));
	taud = dss*(1-exp(-(d_initvalu_39+14.5)/6.0))/(0.035*(d_initvalu_39+14.5));
	fss = 1/(1+exp((d_initvalu_39+35.06)/3.6))+0.6/(1+exp((50-d_initvalu_39)/20));
	tauf = 1/(0.0197*exp(-pow(0.0337*(d_initvalu_39+14.5),2))+0.02);
	d_finavalu[offset_4] = (dss-d_initvalu_4)/taud;
	d_finavalu[offset_5] = (fss-d_initvalu_5)/tauf;
	d_finavalu[offset_6] = 1.7*d_initvalu_36*(1-d_initvalu_6)-11.9e-3*d_initvalu_6;											// fCa_junc
	d_finavalu[offset_7] = 1.7*d_initvalu_37*(1-d_initvalu_7)-11.9e-3*d_initvalu_7;											// fCa_sl

	//
	ibarca_j = pCa*4*(d_initvalu_39*Frdy*FoRT) * (0.341*d_initvalu_36*exp(2*d_initvalu_39*FoRT)-0.341*Cao) /(exp(2*d_initvalu_39*FoRT)-1);
	ibarca_sl = pCa*4*(d_initvalu_39*Frdy*FoRT) * (0.341*d_initvalu_37*exp(2*d_initvalu_39*FoRT)-0.341*Cao) /(exp(2*d_initvalu_39*FoRT)-1);
	ibark = pK*(d_initvalu_39*Frdy*FoRT)*(0.75*d_initvalu_35*exp(d_initvalu_39*FoRT)-0.75*Ko) /(exp(d_initvalu_39*FoRT)-1);
	ibarna_j = pNa*(d_initvalu_39*Frdy*FoRT) *(0.75*d_initvalu_32*exp(d_initvalu_39*FoRT)-0.75*Nao)  /(exp(d_initvalu_39*FoRT)-1);
	ibarna_sl = pNa*(d_initvalu_39*Frdy*FoRT) *(0.75*d_initvalu_33*exp(d_initvalu_39*FoRT)-0.75*Nao)  /(exp(d_initvalu_39*FoRT)-1);
	I_Ca_junc = (Fjunc_CaL*ibarca_j*d_initvalu_4*d_initvalu_5*(1-d_initvalu_6)*pow(Q10CaL,Qpow))*0.45;
	I_Ca_sl = (Fsl_CaL*ibarca_sl*d_initvalu_4*d_initvalu_5*(1-d_initvalu_7)*pow(Q10CaL,Qpow))*0.45;
	I_Ca = I_Ca_junc+I_Ca_sl;
	d_finavalu[offset_43]=-I_Ca*Cmem/(Vmyo*2*Frdy)*1e3;
	I_CaK = (ibark*d_initvalu_4*d_initvalu_5*(Fjunc_CaL*(1-d_initvalu_6)+Fsl_CaL*(1-d_initvalu_7))*pow(Q10CaL,Qpow))*0.45;
	I_CaNa_junc = (Fjunc_CaL*ibarna_j*d_initvalu_4*d_initvalu_5*(1-d_initvalu_6)*pow(Q10CaL,Qpow))*0.45;
	I_CaNa_sl = (Fsl_CaL*ibarna_sl*d_initvalu_4*d_initvalu_5*(1-d_initvalu_7)*pow(Q10CaL,Qpow))*0.45;
	// I_CaNa = I_CaNa_junc+I_CaNa_sl;
	// I_Catot = I_Ca+I_CaK+I_CaNa;

	// I_ncx: Na/Ca Exchanger flux
	Ka_junc = 1/(1+pow((Kdact/d_initvalu_36),3));
	Ka_sl = 1/(1+pow((Kdact/d_initvalu_37),3));
	s1_junc = exp(nu*d_initvalu_39*FoRT)*pow(d_initvalu_32,3)*Cao;
	s1_sl = exp(nu*d_initvalu_39*FoRT)*pow(d_initvalu_33,3)*Cao;
	s2_junc = exp((nu-1)*d_initvalu_39*FoRT)*pow(Nao,3)*d_initvalu_36;
	s3_junc = (KmCai*pow(Nao,3)*(1+pow((d_initvalu_32/KmNai),3))+pow(KmNao,3)*d_initvalu_36+ pow(KmNai,3)*Cao*(1+d_initvalu_36/KmCai)+KmCao*pow(d_initvalu_32,3)+pow(d_initvalu_32,3)*Cao+pow(Nao,3)*d_initvalu_36)*(1+ksat*exp((nu-1)*d_initvalu_39*FoRT));
	s2_sl = exp((nu-1)*d_initvalu_39*FoRT)*pow(Nao,3)*d_initvalu_37;
	s3_sl = (KmCai*pow(Nao,3)*(1+pow((d_initvalu_33/KmNai),3)) + pow(KmNao,3)*d_initvalu_37+pow(KmNai,3)*Cao*(1+d_initvalu_37/KmCai)+KmCao*pow(d_initvalu_33,3)+pow(d_initvalu_33,3)*Cao+pow(Nao,3)*d_initvalu_37)*(1+ksat*exp((nu-1)*d_initvalu_39*FoRT));
	I_ncx_junc = Fjunc*IbarNCX*pow(Q10NCX,Qpow)*Ka_junc*(s1_junc-s2_junc)/s3_junc;
	I_ncx_sl = Fsl*IbarNCX*pow(Q10NCX,Qpow)*Ka_sl*(s1_sl-s2_sl)/s3_sl;
	I_ncx = I_ncx_junc+I_ncx_sl;
	d_finavalu[offset_45]=2*I_ncx*Cmem/(Vmyo*2*Frdy)*1e3;

	// I_pca: Sarcolemmal Ca Pump Current
	I_pca_junc = Fjunc*pow(Q10SLCaP,Qpow)*IbarSLCaP*pow(d_initvalu_36,__float2half(1.6))/(pow(KmPCa,__float2half(1.6))+pow(d_initvalu_36,__float2half(1.6)));
	I_pca_sl = Fsl*pow(Q10SLCaP,Qpow)*IbarSLCaP*pow(d_initvalu_37,__float2half(1.6))/(pow(KmPCa,__float2half(1.6))+pow(d_initvalu_37,__float2half(1.6)));
	I_pca = I_pca_junc+I_pca_sl;
	d_finavalu[offset_44]=-I_pca*Cmem/(Vmyo*2*Frdy)*1e3;

	// I_cabk: Ca Background Current
	I_cabk_junc = Fjunc*GCaB*(d_initvalu_39-eca_junc);
	I_cabk_sl = Fsl*GCaB*(d_initvalu_39-eca_sl);
	I_cabk = I_cabk_junc+I_cabk_sl;
	d_finavalu[offset_46]=-I_cabk*Cmem/(Vmyo*2*Frdy)*1e3;

	// SR fluxes: Calcium Release, SR Ca pump, SR Ca leak
	MaxSR = __float2half(15);
	MinSR = __float2half(1);
	kCaSR = MaxSR - (MaxSR-MinSR)/(1+pow(ec50SR/d_initvalu_31,__float2half(2.5)));
	koSRCa = koCa/kCaSR;
	kiSRCa = kiCa*kCaSR;
	RI = 1-d_initvalu_14-d_initvalu_15-d_initvalu_16;
	d_finavalu[offset_14] = (kim*RI-kiSRCa*d_initvalu_36*d_initvalu_14)-(koSRCa*pow(d_initvalu_36,2)*d_initvalu_14-kom*d_initvalu_15);			// R
	d_finavalu[offset_15] = (koSRCa*pow(d_initvalu_36,2)*d_initvalu_14-kom*d_initvalu_15)-(kiSRCa*d_initvalu_36*d_initvalu_15-kim*d_initvalu_16);			// O
	d_finavalu[offset_16] = (kiSRCa*d_initvalu_36*d_initvalu_15-kim*d_initvalu_16)-(kom*d_initvalu_16-koSRCa*pow(d_initvalu_36,2)*RI);			// I
	J_SRCarel = ks*d_initvalu_15*(d_initvalu_31-d_initvalu_36);													// [mM/ms]
	J_serca = pow(Q10SRCaP,Qpow)*Vmax_SRCaP*(pow((d_initvalu_38/Kmf),hillSRCaP)-pow((d_initvalu_31/Kmr),hillSRCaP))
										 /(1+pow((d_initvalu_38/Kmf),hillSRCaP)+pow((d_initvalu_31/Kmr),hillSRCaP));
	J_SRleak = 5.348e-6*(d_initvalu_31-d_initvalu_36);													//   [mM/ms]

	// Sodium and Calcium Buffering
	d_finavalu[offset_17] = kon_na*d_initvalu_32*(Bmax_Naj-d_initvalu_17)-koff_na*d_initvalu_17;								// NaBj      [mM/ms]
	d_finavalu[offset_18] = kon_na*d_initvalu_33*(Bmax_Nasl-d_initvalu_18)-koff_na*d_initvalu_18;							// NaBsl     [mM/ms]

	// Cytosolic Ca Buffers
	d_finavalu[offset_19] = kon_tncl*d_initvalu_38*(Bmax_TnClow-d_initvalu_19)-koff_tncl*d_initvalu_19;						// TnCL      [mM/ms]
	d_finavalu[offset_20] = kon_tnchca*d_initvalu_38*(Bmax_TnChigh-d_initvalu_20-d_initvalu_21)-koff_tnchca*d_initvalu_20;			// TnCHc     [mM/ms]
	d_finavalu[offset_21] = kon_tnchmg*Mgi*(Bmax_TnChigh-d_initvalu_20-d_initvalu_21)-koff_tnchmg*d_initvalu_21;				// TnCHm     [mM/ms]
	d_finavalu[offset_22] = __float2half(0);																		// CaM       [mM/ms]
	d_finavalu[offset_23] = kon_myoca*d_initvalu_38*(Bmax_myosin-d_initvalu_23-d_initvalu_24)-koff_myoca*d_initvalu_23;				// Myosin_ca [mM/ms]
	d_finavalu[offset_24] = kon_myomg*Mgi*(Bmax_myosin-d_initvalu_23-d_initvalu_24)-koff_myomg*d_initvalu_24;				// Myosin_mg [mM/ms]
	d_finavalu[offset_25] = kon_sr*d_initvalu_38*(Bmax_SR-d_initvalu_25)-koff_sr*d_initvalu_25;								// SRB       [mM/ms]
	J_CaB_cytosol = d_finavalu[offset_19] + d_finavalu[offset_20] + d_finavalu[offset_21] + d_finavalu[offset_22] + d_finavalu[offset_23] + d_finavalu[offset_24] + d_finavalu[offset_25];

	// Junctional and SL Ca Buffers
	d_finavalu[offset_26] = kon_sll*d_initvalu_36*(Bmax_SLlowj-d_initvalu_26)-koff_sll*d_initvalu_26;						// SLLj      [mM/ms]
	d_finavalu[offset_27] = kon_sll*d_initvalu_37*(Bmax_SLlowsl-d_initvalu_27)-koff_sll*d_initvalu_27;						// SLLsl     [mM/ms]
	d_finavalu[offset_28] = kon_slh*d_initvalu_36*(Bmax_SLhighj-d_initvalu_28)-koff_slh*d_initvalu_28;						// SLHj      [mM/ms]
	d_finavalu[offset_29] = kon_slh*d_initvalu_37*(Bmax_SLhighsl-d_initvalu_29)-koff_slh*d_initvalu_29;						// SLHsl     [mM/ms]
	J_CaB_junction = d_finavalu[offset_26]+d_finavalu[offset_28];
	J_CaB_sl = d_finavalu[offset_27]+d_finavalu[offset_29];

	// SR Ca Concentrations
	d_finavalu[offset_30] = kon_csqn*d_initvalu_31*(Bmax_Csqn-d_initvalu_30)-koff_csqn*d_initvalu_30;						// Csqn      [mM/ms]
	oneovervsr = 1/Vsr;
	d_finavalu[offset_31] = J_serca*Vmyo*oneovervsr-(J_SRleak*Vmyo*oneovervsr+J_SRCarel)-d_finavalu[offset_30];   // Ca_sr     [mM/ms] %Ratio 3 leak current

	// Sodium Concentrations
	I_Na_tot_junc = I_Na_junc+I_nabk_junc+3*I_ncx_junc+3*I_nak_junc+I_CaNa_junc;		// [uA/uF]
	I_Na_tot_sl = I_Na_sl+I_nabk_sl+3*I_ncx_sl+3*I_nak_sl+I_CaNa_sl;					// [uA/uF]
	d_finavalu[offset_32] = -I_Na_tot_junc*Cmem/(Vjunc*Frdy)+J_na_juncsl/Vjunc*(d_initvalu_33-d_initvalu_32)-d_finavalu[offset_17];
	oneovervsl = 1/Vsl;
	d_finavalu[offset_33] = -I_Na_tot_sl*Cmem*oneovervsl/Frdy+J_na_juncsl*oneovervsl*(d_initvalu_32-d_initvalu_33)+J_na_slmyo*oneovervsl*(d_initvalu_34-d_initvalu_33)-d_finavalu[offset_18];
	d_finavalu[offset_34] = J_na_slmyo/Vmyo*(d_initvalu_33-d_initvalu_34);											// [mM/msec]

	// Potassium Concentration
	I_K_tot = I_to+I_kr+I_ks+I_ki-2*I_nak+I_CaK+I_kp;									// [uA/uF]
	d_finavalu[offset_35] = __float2half(0);															// [mM/msec]

	// Calcium Concentrations
	I_Ca_tot_junc = I_Ca_junc+I_cabk_junc+I_pca_junc-2*I_ncx_junc;						// [uA/uF]
	I_Ca_tot_sl = I_Ca_sl+I_cabk_sl+I_pca_sl-2*I_ncx_sl;								// [uA/uF]
	d_finavalu[offset_36] = -I_Ca_tot_junc*Cmem/(Vjunc*2*Frdy)+J_ca_juncsl/Vjunc*(d_initvalu_37-d_initvalu_36)
	         - J_CaB_junction+(J_SRCarel)*Vsr/Vjunc+J_SRleak*Vmyo/Vjunc;				// Ca_j
	d_finavalu[offset_37] = -I_Ca_tot_sl*Cmem/(Vsl*2*Frdy)+J_ca_juncsl/Vsl*(d_initvalu_36-d_initvalu_37)
	         + J_ca_slmyo/Vsl*(d_initvalu_38-d_initvalu_37)-J_CaB_sl;									// Ca_sl
	d_finavalu[offset_38] = -J_serca-J_CaB_cytosol +J_ca_slmyo/Vmyo*(d_initvalu_37-d_initvalu_38);
	// junc_sl=J_ca_juncsl/Vsl*(d_initvalu_36-d_initvalu_37);
	// sl_junc=J_ca_juncsl/Vjunc*(d_initvalu_37-d_initvalu_36);
	// sl_myo=J_ca_slmyo/Vsl*(d_initvalu_38-d_initvalu_37);
	// myo_sl=J_ca_slmyo/Vmyo*(d_initvalu_37-d_initvalu_38);

	// Simulation type
	state = 1;
	switch(state){
		case 0:
			I_app = __float2half(0);
			break;
		case 1:																			// pace w/ current injection at cycleLength 'cycleLength'
			if(fmod(timeinst,cycleLength) <= 5){
				I_app = __float2half(9.5);
			}
			else{
				I_app = __float2half(0.0);
			}
			break;
		case 2:
			V_hold = __float2half(-55);
			V_test = __float2half(0);
			if(timeinst>0.5 & timeinst<200.5){
				V_clamp = V_test;
			}
			else{
				V_clamp = V_hold;
			}
			R_clamp = __float2half(0.04);
			I_app = (V_clamp-d_initvalu_39)/R_clamp;
			break;
	}

	// Membrane Potential
	I_Na_tot = I_Na_tot_junc + I_Na_tot_sl;												// [uA/uF]
	I_Cl_tot = I_ClCa+I_Clbk;															// [uA/uF]
	I_Ca_tot = I_Ca_tot_junc+I_Ca_tot_sl;
	I_tot = I_Na_tot+I_Cl_tot+I_Ca_tot+I_K_tot;
	d_finavalu[offset_39] = -(I_tot-I_app);

	// Set unused output values to 0 (MATLAB does it by default)
	d_finavalu[offset_41] = __float2half(0);
	d_finavalu[offset_42] = __float2half(0);

}
//=====================================================================
//	MAIN FUNCTION
//=====================================================================
__device__ void kernel_cam_2(	fp timeinst,
													fp* d_initvalu,
													fp *d_finavalu,
													int valu_offset,
													fp* d_params,
													int params_offset,
													fp* d_com,
													int com_offset,
													fp Ca){

	//=====================================================================
	//	VARIABLES
	//=====================================================================

	// inputs
	// fp CaMtot;
	fp Btot;
	fp CaMKIItot;
	fp CaNtot;
	fp PP1tot;
	fp K;
	fp Mg;

	// variable references
	int offset_1;
	int offset_2;
	int offset_3;
	int offset_4;
	int offset_5;
	int offset_6;
	int offset_7;
	int offset_8;
	int offset_9;
	int offset_10;
	int offset_11;
	int offset_12;
	int offset_13;
	int offset_14;
	int offset_15;

	// decoding input array
	fp CaM;
	fp Ca2CaM;
	fp Ca4CaM;
	fp CaMB;
	fp Ca2CaMB;
	fp Ca4CaMB;
	fp Pb2;
	fp Pb;
	fp Pt;
	fp Pt2;
	fp Pa;
	fp Ca4CaN;
	fp CaMCa4CaN;
	fp Ca2CaMCa4CaN;
	fp Ca4CaMCa4CaN;

	// Ca/CaM parameters
	fp Kd02;																		// [uM^2]
	fp Kd24;																		// [uM^2]
	fp k20;																			// [s^-1]
	fp k02;																			// [uM^-2 s^-1]
	fp k42;																			// [s^-1]
	fp k24;																			// [uM^-2 s^-1]

	// CaM buffering (B) parameters
	fp k0Boff;																		// [s^-1]
	fp k0Bon;																		// [uM^-1 s^-1] kon = koff/Kd
	fp k2Boff;																		// [s^-1]
	fp k2Bon;																		// [uM^-1 s^-1]
	fp k4Boff;																		// [s^-1]
	fp k4Bon;																		// [uM^-1 s^-1]

	// using thermodynamic constraints
	fp k20B;																		// [s^-1] thermo constraint on loop 1
	fp k02B;																		// [uM^-2 s^-1]
	fp k42B;																		// [s^-1] thermo constraint on loop 2
	fp k24B;																		// [uM^-2 s^-1]

	// Wi Wa Wt Wp
	fp kbi;																			// [s^-1] (Ca4CaM dissocation from Wb)
	fp kib;																			// [uM^-1 s^-1]
	fp kpp1;																		// [s^-1] (PP1-dep dephosphorylation rates)
	fp Kmpp1;																		// [uM]
	fp kib2;
	fp kb2i;
	fp kb24;
	fp kb42;
	fp kta;																			// [s^-1] (Ca4CaM dissociation from Wt)
	fp kat;																			// [uM^-1 s^-1] (Ca4CaM reassociation with Wa)
	fp kt42;
	fp kt24;
	fp kat2;
	fp kt2a;

	// CaN parameters
	fp kcanCaoff;																	// [s^-1]
	fp kcanCaon;																	// [uM^-1 s^-1]
	fp kcanCaM4on;																	// [uM^-1 s^-1]
	fp kcanCaM4off;																	// [s^-1]
	fp kcanCaM2on;
	fp kcanCaM2off;
	fp kcanCaM0on;
	fp kcanCaM0off;
	fp k02can;
	fp k20can;
	fp k24can;
	fp k42can;

	// CaM Reaction fluxes
	fp rcn02;
	fp rcn24;

	// CaM buffer fluxes
	fp B;
	fp rcn02B;
	fp rcn24B;
	fp rcn0B;
	fp rcn2B;
	fp rcn4B;

	// CaN reaction fluxes
	fp Ca2CaN;
	fp rcnCa4CaN;
	fp rcn02CaN;
	fp rcn24CaN;
	fp rcn0CaN;
	fp rcn2CaN;
	fp rcn4CaN;

	// CaMKII reaction fluxes
	fp Pix;
	fp rcnCKib2;
	fp rcnCKb2b;
	fp rcnCKib;
	fp T;
	fp kbt;
	fp rcnCKbt;
	fp rcnCKtt2;
	fp rcnCKta;
	fp rcnCKt2a;
	fp rcnCKt2b2;
	fp rcnCKai;

	// CaM equations
	fp dCaM;
	fp dCa2CaM;
	fp dCa4CaM;
	fp dCaMB;
	fp dCa2CaMB;
	fp dCa4CaMB;

	// CaMKII equations
	fp dPb2;																					// Pb2
	fp dPb;																					// Pb
	fp dPt;																					// Pt
	fp dPt2;																					// Pt2
	fp dPa;																					// Pa

	// CaN equations
	fp dCa4CaN;																			// Ca4CaN
	fp dCaMCa4CaN;																	// CaMCa4CaN
	fp dCa2CaMCa4CaN;																// Ca2CaMCa4CaN
	fp dCa4CaMCa4CaN;																// Ca4CaMCa4CaN

	//=====================================================================
	//	EXECUTION
	//=====================================================================

	// inputs
	// CaMtot = d_params[params_offset];
	Btot = d_params[params_offset+1];
	CaMKIItot = d_params[params_offset+2];
	CaNtot = d_params[params_offset+3];
	PP1tot = d_params[params_offset+4];
	K = d_params[16];
	Mg = d_params[17];

	// variable references
	offset_1 = valu_offset;
	offset_2 = valu_offset+1;
	offset_3 = valu_offset+2;
	offset_4 = valu_offset+3;
	offset_5 = valu_offset+4;
	offset_6 = valu_offset+5;
	offset_7 = valu_offset+6;
	offset_8 = valu_offset+7;
	offset_9 = valu_offset+8;
	offset_10 = valu_offset+9;
	offset_11 = valu_offset+10;
	offset_12 = valu_offset+11;
	offset_13 = valu_offset+12;
	offset_14 = valu_offset+13;
	offset_15 = valu_offset+14;

	// decoding input array
	CaM				= d_initvalu[offset_1];
	Ca2CaM			= d_initvalu[offset_2];
	Ca4CaM			= d_initvalu[offset_3];
	CaMB			= d_initvalu[offset_4];
	Ca2CaMB			= d_initvalu[offset_5];
	Ca4CaMB			= d_initvalu[offset_6];
	Pb2				= d_initvalu[offset_7];
	Pb				= d_initvalu[offset_8];
	Pt				= d_initvalu[offset_9];
	Pt2				= d_initvalu[offset_10];
	Pa				= d_initvalu[offset_11];
	Ca4CaN			= d_initvalu[offset_12];
	CaMCa4CaN		= d_initvalu[offset_13];
	Ca2CaMCa4CaN	= d_initvalu[offset_14];
	Ca4CaMCa4CaN	= d_initvalu[offset_15];

	// Ca/CaM parameters
	if (Mg <= 1){
		Kd02 = 0.0025*(1+K/0.94-Mg/0.012)*(1+K/8.1+Mg/0.022);							// [uM^2]
		Kd24 = 0.128*(1+K/0.64+Mg/0.0014)*(1+K/13.0-Mg/0.153);							// [uM^2]
	}
	else{
		Kd02 = 0.0025*(1+K/0.94-1/0.012+(Mg-1)/0.060)*(1+K/8.1+1/0.022+(Mg-1)/0.068);   // [uM^2]
		Kd24 = 0.128*(1+K/0.64+1/0.0014+(Mg-1)/0.005)*(1+K/13.0-1/0.153+(Mg-1)/0.150);  // [uM^2]
	}
	k20 = __float2half(10);																			// [s^-1]
	k02 = k20/Kd02;																		// [uM^-2 s^-1]
	k42 = __float2half(500);																			// [s^-1]
	k24 = k42/Kd24;																		// [uM^-2 s^-1]

	// CaM buffering (B) parameters
	k0Boff = __float2half(0.0014);																	// [s^-1]
	k0Bon = k0Boff/0.2;																	// [uM^-1 s^-1] kon = koff/Kd
	k2Boff = k0Boff/100;																// [s^-1]
	k2Bon = k0Bon;																		// [uM^-1 s^-1]
	k4Boff = k2Boff;																	// [s^-1]
	k4Bon = k0Bon;																		// [uM^-1 s^-1]

	// using thermodynamic constraints
	k20B = k20/100;																		// [s^-1] thermo constraint on loop 1
	k02B = k02;																			// [uM^-2 s^-1]
	k42B = k42;																			// [s^-1] thermo constraint on loop 2
	k24B = k24;																			// [uM^-2 s^-1]

	// Wi Wa Wt Wp
	kbi = __float2half(2.2);																			// [s^-1] (Ca4CaM dissocation from Wb)
	kib = kbi/33.5e-3;																	// [uM^-1 s^-1]
	kpp1 = __float2half(1.72);																		// [s^-1] (PP1-dep dephosphorylation rates)
	Kmpp1 = __float2half(11.5);																		// [uM]
	kib2 = kib;
	kb2i = kib2*5;
	kb24 = k24;
	kb42 = k42*33.5e-3/5;
	kta = kbi/1000;																		// [s^-1] (Ca4CaM dissociation from Wt)
	kat = kib;																			// [uM^-1 s^-1] (Ca4CaM reassociation with Wa)
	kt42 = k42*33.5e-6/5;
	kt24 = k24;
	kat2 = kib;
	kt2a = kib*5;

	// CaN parameters
	kcanCaoff = __float2half(1);																		// [s^-1]
	kcanCaon = kcanCaoff/0.5;															// [uM^-1 s^-1]
	kcanCaM4on = __float2half(46);																	// [uM^-1 s^-1]
	kcanCaM4off = __float2half(0.0013);																// [s^-1]
	kcanCaM2on = kcanCaM4on;
	kcanCaM2off = 2508*kcanCaM4off;
	kcanCaM0on = kcanCaM4on;
	kcanCaM0off = 165*kcanCaM2off;
	k02can = k02;
	k20can = k20/165;
	k24can = k24;
	k42can = k20/2508;

	// CaM Reaction fluxes
	rcn02 = k02*pow(Ca,2)*CaM - k20*Ca2CaM;
	rcn24 = k24*pow(Ca,2)*Ca2CaM - k42*Ca4CaM;

	// CaM buffer fluxes
	B = Btot - CaMB - Ca2CaMB - Ca4CaMB;
	rcn02B = k02B*pow(Ca,2)*CaMB - k20B*Ca2CaMB;
	rcn24B = k24B*pow(Ca,2)*Ca2CaMB - k42B*Ca4CaMB;
	rcn0B = k0Bon*CaM*B - k0Boff*CaMB;
	rcn2B = k2Bon*Ca2CaM*B - k2Boff*Ca2CaMB;
	rcn4B = k4Bon*Ca4CaM*B - k4Boff*Ca4CaMB;

	// CaN reaction fluxes
	Ca2CaN = CaNtot - Ca4CaN - CaMCa4CaN - Ca2CaMCa4CaN - Ca4CaMCa4CaN;
	rcnCa4CaN = kcanCaon*pow(Ca,2)*Ca2CaN - kcanCaoff*Ca4CaN;
	rcn02CaN = k02can*pow(Ca,2)*CaMCa4CaN - k20can*Ca2CaMCa4CaN;
	rcn24CaN = k24can*pow(Ca,2)*Ca2CaMCa4CaN - k42can*Ca4CaMCa4CaN;
	rcn0CaN = kcanCaM0on*CaM*Ca4CaN - kcanCaM0off*CaMCa4CaN;
	rcn2CaN = kcanCaM2on*Ca2CaM*Ca4CaN - kcanCaM2off*Ca2CaMCa4CaN;
	rcn4CaN = kcanCaM4on*Ca4CaM*Ca4CaN - kcanCaM4off*Ca4CaMCa4CaN;

	// CaMKII reaction fluxes
	Pix = 1 - Pb2 - Pb - Pt - Pt2 - Pa;
	rcnCKib2 = kib2*Ca2CaM*Pix - kb2i*Pb2;
	rcnCKb2b = kb24*pow(Ca,2)*Pb2 - kb42*Pb;
	rcnCKib = kib*Ca4CaM*Pix - kbi*Pb;
	T = Pb + Pt + Pt2 + Pa;
	kbt = 0.055*T + 0.0074*pow(T,2) + 0.015*pow(T,3);
	rcnCKbt = kbt*Pb - kpp1*PP1tot*Pt/(Kmpp1+CaMKIItot*Pt);
	rcnCKtt2 = kt42*Pt - kt24*pow(Ca,2)*Pt2;
	rcnCKta = kta*Pt - kat*Ca4CaM*Pa;
	rcnCKt2a = kt2a*Pt2 - kat2*Ca2CaM*Pa;
	rcnCKt2b2 = kpp1*PP1tot*Pt2/(Kmpp1+CaMKIItot*Pt2);
	rcnCKai = kpp1*PP1tot*Pa/(Kmpp1+CaMKIItot*Pa);

	// CaM equations
	dCaM = 1e-3*(-rcn02 - rcn0B - rcn0CaN);
	dCa2CaM = 1e-3*(rcn02 - rcn24 - rcn2B - rcn2CaN + CaMKIItot*(-rcnCKib2 + rcnCKt2a) );
	dCa4CaM = 1e-3*(rcn24 - rcn4B - rcn4CaN + CaMKIItot*(-rcnCKib+rcnCKta) );
	dCaMB = 1e-3*(rcn0B-rcn02B);
	dCa2CaMB = 1e-3*(rcn02B + rcn2B - rcn24B);
	dCa4CaMB = 1e-3*(rcn24B + rcn4B);

	// CaMKII equations
	dPb2 = 1e-3*(rcnCKib2 - rcnCKb2b + rcnCKt2b2);										// Pb2
	dPb = 1e-3*(rcnCKib + rcnCKb2b - rcnCKbt);											// Pb
	dPt = 1e-3*(rcnCKbt-rcnCKta-rcnCKtt2);												// Pt
	dPt2 = 1e-3*(rcnCKtt2-rcnCKt2a-rcnCKt2b2);											// Pt2
	dPa = 1e-3*(rcnCKta+rcnCKt2a-rcnCKai);												// Pa

	// CaN equations
	dCa4CaN = 1e-3*(rcnCa4CaN - rcn0CaN - rcn2CaN - rcn4CaN);							// Ca4CaN
	dCaMCa4CaN = 1e-3*(rcn0CaN - rcn02CaN);												// CaMCa4CaN
	dCa2CaMCa4CaN = 1e-3*(rcn2CaN+rcn02CaN-rcn24CaN);									// Ca2CaMCa4CaN
	dCa4CaMCa4CaN = 1e-3*(rcn4CaN+rcn24CaN);											// Ca4CaMCa4CaN

	// encode output array
	d_finavalu[offset_1] = dCaM;
	d_finavalu[offset_2] = dCa2CaM;
	d_finavalu[offset_3] = dCa4CaM;
	d_finavalu[offset_4] = dCaMB;
	d_finavalu[offset_5] = dCa2CaMB;
	d_finavalu[offset_6] = dCa4CaMB;
	d_finavalu[offset_7] = dPb2;
	d_finavalu[offset_8] = dPb;
	d_finavalu[offset_9] = dPt;
	d_finavalu[offset_10] = dPt2;
	d_finavalu[offset_11] = dPa;
	d_finavalu[offset_12] = dCa4CaN;
	d_finavalu[offset_13] = dCaMCa4CaN;
	d_finavalu[offset_14] = dCa2CaMCa4CaN;
	d_finavalu[offset_15] = dCa4CaMCa4CaN;

	// write to global variables for adjusting Ca buffering in EC coupling model
	d_finavalu[com_offset] = 1e-3*(2*CaMKIItot*(rcnCKtt2-rcnCKb2b) - 2*(rcn02+rcn24+rcn02B+rcn24B+rcnCa4CaN+rcn02CaN+rcn24CaN)); // [uM/msec]
	//d_finavalu[JCa] = 1; // [uM/msec]

}
//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================
//		KERNEL
//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================

__device__ void kernel_2(	int timeinst,
											fp* initvalu,
											fp* params,
											fp* finavalu,
											fp* com){

	//======================================================================================================================================================
	// 	VARIABLES
	//======================================================================================================================================================

	// pointers
	int valu_offset_ecc;															// inivalu and finavalu offset
	int valu_offset_Dyad;														// Dyad value offset
	int valu_offset_SL;																// SL value offset
	int valu_offset_Cyt;															// Cyt value offset

	int params_offset_Dyad;													// Dyad parameters offset
	int params_offset_SL;														// SL parameters offset
	int params_offset_Cyt;														// Cyt parameters offset

	int com_offset_Dyad;															// kernel1-kernel2 Dyad communication offset
	int com_offset_SL;																// kernel1-kernel2 SL communication offset
	int com_offset_Cyt;															// kernel-kernel Cyt communication offset

	// module parameters
	fp CaDyad;																			// from ECC model, *** Converting from [mM] to [uM] ***
	fp CaSL;																				// from ECC model, *** Converting from [mM] to [uM] ***
	fp CaCyt;																			// from ECC model, *** Converting from [mM] to [uM] ***

	// counter
	int i;

	//======================================================================================================================================================
	// 	COMPUTATION
	//======================================================================================================================================================

	valu_offset_ecc = 0;
	valu_offset_Dyad = 46;
	valu_offset_SL = 61;
	valu_offset_Cyt = 76;

	params_offset_Dyad = 0;
	params_offset_SL = 5;
	params_offset_Cyt = 10;

	com_offset_Dyad = 0;
	com_offset_SL = 1;
	com_offset_Cyt = 2;

	//==================================================
	//		ECC
	//==================================================
	fp timeinst_half = __float2half(timeinst);
	// ecc function
	kernel_ecc_2(timeinst_half,
							initvalu,
							finavalu,
							valu_offset_ecc,
							params);

	//==================================================
	//		3xCAM
	//==================================================

	// specific
	CaDyad = initvalu[35]*1e3;							// from ECC model, *** Converting from [mM] to [uM] ***
	// cam function for Dyad
	kernel_cam_2(	timeinst_half,
								initvalu,
								finavalu,
								valu_offset_Dyad,
								params,
								params_offset_Dyad,
								com,
								com_offset_Dyad,
								CaDyad);

	// specific
	CaSL = initvalu[36]*1e3;								// from ECC model, *** Converting from [mM] to [uM] ***
	// cam function for Dyad
	kernel_cam_2(	timeinst_half,
								initvalu,
								finavalu,
								valu_offset_SL,
								params,
								params_offset_SL,
								com,
								com_offset_SL,
								CaSL);

	// specific
	CaCyt = initvalu[37]*1e3;							// from ECC model, *** Converting from [mM] to [uM] ***
	// cam function for Dyad
	kernel_cam_2(	timeinst_half,
								initvalu,
								finavalu,
								valu_offset_Cyt,
								params,
								params_offset_Cyt,
								com,
								com_offset_Cyt,
								CaCyt);

	//====================================================================================================
	//		SEGMENT HAPPENING 2ND IN TIME: FINAL
	//====================================================================================================

	kernel_fin_2(	timeinst,
							initvalu,
							finavalu,
							valu_offset_ecc,
							valu_offset_Dyad,
							valu_offset_SL,
							valu_offset_Cyt,
							params,
							com);

	//====================================================================================================
	//		make sure function does not return NANs and INFs
	//====================================================================================================

	for(i=0; i<EQUATIONS; i++){
		if (isnan(finavalu[i]) == 1){
			finavalu[i] = __float2half(0.0001);												// for NAN set rate of change to 0.0001
		}
		else if (isinf(finavalu[i]) == 1){
			finavalu[i] = __float2half(0.0001);												// for INF set rate of change to 0.0001
		}
	}

//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================
//		END OF KERNEL
//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================

}
//====================================================================================================100
//		UPDATE
//====================================================================================================100

// This file is the modified version of embedded fehlberg 7.8 solver obrained from (http://mymathlib.webtrellis.net/index.html)
// Lukasz G. Szafaryn 15 DEC 09

//====================================================================================================100
//		DESCRIPTION
//====================================================================================================100

//                                                                            //
//  Description:                                                              //
//     The Runge-Kutta-Fehlberg method is an adaptive procedure for approxi-  //
//     mating the solution of the differential equation y'(x) = f(x,y) with   //
//     initial condition y(x0) = c.  This implementation evaluates f(x,y)     //
//     thirteen times per step using embedded seventh order and eight order   //
//     Runge-Kutta estimates to estimate the not only the solution but also   //
//     the error.                                                             //
//     The next step size is then calculated using the preassigned tolerance  //
//     and error estimate.                                                    //
//     For step i+1,                                                          //
//        y[i+1] = y[i] +  h * (41/840 * k1 + 34/105 * finavalu_temp[5] + 9/35 * finavalu_temp[6]         //
//                        + 9/35 * finavalu_temp[7] + 9/280 * finavalu_temp[8] + 9/280 finavalu_temp[9] + 41/840 finavalu_temp[10] ) //
//     where                                                                  //
//     k1 = f( x[i],y[i] ),                                                   //
//     finavalu_temp[1] = f( x[i]+2h/27, y[i] + 2h*k1/27),                                  //
//     finavalu_temp[2] = f( x[i]+h/9, y[i]+h/36*( k1 + 3 finavalu_temp[1]) ),                            //
//     finavalu_temp[3] = f( x[i]+h/6, y[i]+h/24*( k1 + 3 finavalu_temp[2]) ),                            //
//     finavalu_temp[4] = f( x[i]+5h/12, y[i]+h/48*(20 k1 - 75 finavalu_temp[2] + 75 finavalu_temp[3])),                //
//     finavalu_temp[5] = f( x[i]+h/2, y[i]+h/20*( k1 + 5 finavalu_temp[3] + 4 finavalu_temp[4] ) ),                    //
//     finavalu_temp[6] = f( x[i]+5h/6, y[i]+h/108*( -25 k1 + 125 finavalu_temp[3] - 260 finavalu_temp[4] + 250 finavalu_temp[5] ) ), //
//     finavalu_temp[7] = f( x[i]+h/6, y[i]+h*( 31/300 k1 + 61/225 finavalu_temp[4] - 2/9 finavalu_temp[5]              //
//                                                            + 13/900 finavalu_temp[6]) )  //
//     finavalu_temp[8] = f( x[i]+2h/3, y[i]+h*( 2 k1 - 53/6 finavalu_temp[3] + 704/45 finavalu_temp[4] - 107/9 finavalu_temp[5]      //
//                                                      + 67/90 finavalu_temp[6] + 3 finavalu_temp[7]) ), //
//     finavalu_temp[9] = f( x[i]+h/3, y[i]+h*( -91/108 k1 + 23/108 finavalu_temp[3] - 976/135 finavalu_temp[4]        //
//                             + 311/54 finavalu_temp[5] - 19/60 finavalu_temp[6] + 17/6 finavalu_temp[7] - 1/12 finavalu_temp[8]) ), //
//     finavalu_temp[10] = f( x[i]+h, y[i]+h*( 2383/4100 k1 - 341/164 finavalu_temp[3] + 4496/1025 finavalu_temp[4]     //
//          - 301/82 finavalu_temp[5] + 2133/4100 finavalu_temp[6] + 45/82 finavalu_temp[7] + 45/164 finavalu_temp[8] + 18/41 finavalu_temp[9]) )  //
//     finavalu_temp[11] = f( x[i], y[i]+h*( 3/205 k1 - 6/41 finavalu_temp[5] - 3/205 finavalu_temp[6] - 3/41 finavalu_temp[7]        //
//                                                   + 3/41 finavalu_temp[8] + 6/41 finavalu_temp[9]) )  //
//     finavalu_temp[12] = f( x[i]+h, y[i]+h*( -1777/4100 k1 - 341/164 finavalu_temp[3] + 4496/1025 finavalu_temp[4]    //
//                      - 289/82 finavalu_temp[5] + 2193/4100 finavalu_temp[6] + 51/82 finavalu_temp[7] + 33/164 finavalu_temp[8] +   //
//                                                        12/41 finavalu_temp[9] + finavalu_temp[11]) )  //
//     x[i+1] = x[i] + h.                                                     //
//                                                                            //
//     The error is estimated to be                                           //
//        err = -41/840 * h * ( k1 + finavalu_temp[10] - finavalu_temp[11] - finavalu_temp[12])                         //
//     The step size h is then scaled by the scale factor                     //
//         scale = 0.8 * | epsilon * y[i] / [err * (xmax - x[0])] | ^ 1/7     //
//     The scale factor is further constrained 0.125 < scale < 4.0.           //
//     The new step size is h := scale * h.                                   //
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
//  static fp Runge_Kutta(fp (*f)(fp,fp), fp *y,          //
//                                                       fp x0, fp h) //
//                                                                            //
//  Description:                                                              //
//     This routine uses Fehlberg's embedded 7th and 8th order methods to     //
//     approximate the solution of the differential equation y'=f(x,y) with   //
//     the initial condition y = y[0] at x = x0.  The value at x + h is       //
//     returned in y[1].  The function returns err / h ( the absolute error   //
//     per step size ).                                                       //
//                                                                            //
//  Arguments:                                                                //
//     fp *f  Pointer to the function which returns the slope at (x,y) of //
//                integral curve of the differential equation y' = f(x,y)     //
//                which passes through the point (x0,y[0]).                   //
//     fp y[] On input y[0] is the initial value of y at x, on output     //
//                y[1] is the solution at x + h.                              //
//     fp x   Initial value of x.                                         //
//     fp h   Step size                                                   //
//                                                                            //
//  Return Values:                                                            //
//     This routine returns the err / h.  The solution of y(x) at x + h is    //
//     returned in y[1].                                                      //
//                                                                            //
////////////////////////////////////////////////////////////////////////////////

//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================
//		PARTICULAR SOLVER FUNCTION
//===============================================================================================================================================================================================================
//===============================================================================================================================================================================================================

__device__ void embedded_fehlberg_7_8_2(	fp h,

																			fp timeinst,
																			fp* initvalu,
																			fp* finavalu,
																			fp* parameter,

																			fp* error,
																			fp* initvalu_temp,
																			fp* finavalu_temp,
																			fp* com) {

	//======================================================================================================================================================
	//	VARIABLES
	//======================================================================================================================================================

	const fp c_1_11 = __float2half(41.0 / 840.0);
	const fp c6 = __float2half(34.0 / 105.0);
	const fp c_7_8= __float2half( 9.0 / 35.0);
	const fp c_9_10 = __float2half( 9.0 / 280.0);

	const fp a2 = __float2half( 2.0 / 27.0);
	const fp a3 = __float2half( 1.0 / 9.0);
	const fp a4 = __float2half( 1.0 / 6.0);
	const fp a5 = __float2half( 5.0 / 12.0);
	const fp a6 = __float2half( 1.0 / 2.0);
	const fp a7 = __float2half( 5.0 / 6.0);
	const fp a8 = __float2half( 1.0 / 6.0);
	const fp a9 = __float2half( 2.0 / 3.0);
	const fp a10 = __float2half( 1.0 / 3.0);

	const fp b31 = __float2half( 1.0 / 36.0);
	const fp b32 = __float2half( 3.0 / 36.0);
	const fp b41 = __float2half( 1.0 / 24.0);
	const fp b43 = __float2half( 3.0 / 24.0);
	const fp b51 = __float2half( 20.0 / 48.0);
	const fp b53 = __float2half( -75.0 / 48.0);
	const fp b54 = __float2half( 75.0 / 48.0);
	const fp b61 = __float2half( 1.0 / 20.0);
	const fp b64 = __float2half( 5.0 / 20.0);
	const fp b65 = __float2half( 4.0 / 20.0);
	const fp b71 = __float2half( -25.0 / 108.0);
	const fp b74 = __float2half(  125.0 / 108.0);
	const fp b75 = __float2half( -260.0 / 108.0);
	const fp b76 = __float2half(  250.0 / 108.0);
	const fp b81 = __float2half( 31.0/300.0);
	const fp b85 = __float2half( 61.0/225.0);
	const fp b86 = __float2half( -2.0/9.0);
	const fp b87 = __float2half( 13.0/900.0);
	const fp b91 = __float2half( 2.0);
	const fp b94 = __float2half( -53.0/6.0);
	const fp b95 = __float2half( 704.0 / 45.0);
	const fp b96 = __float2half( -107.0 / 9.0);
	const fp b97 = __float2half( 67.0 / 90.0);
	const fp b98 = __float2half( 3.0);
	const fp b10_1 = __float2half( -91.0 / 108.0);
	const fp b10_4 = __float2half( 23.0 / 108.0);
	const fp b10_5 = __float2half( -976.0 / 135.0);
	const fp b10_6 = __float2half( 311.0 / 54.0);
	const fp b10_7 = __float2half( -19.0 / 60.0);
	const fp b10_8 = __float2half( 17.0 / 6.0);
	const fp b10_9 = __float2half( -1.0 / 12.0);
	const fp b11_1 = __float2half( 2383.0 / 4100.0);
	const fp b11_4 = __float2half( -341.0 / 164.0);
	const fp b11_5 = __float2half( 4496.0 / 1025.0);
	const fp b11_6 = __float2half( -301.0 / 82.0);
	const fp b11_7 = __float2half( 2133.0 / 4100.0);
	const fp b11_8 = __float2half( 45.0 / 82.0);
	const fp b11_9 = __float2half( 45.0 / 164.0);
	const fp b11_10 = __float2half( 18.0 / 41.0);
	const fp b12_1 = __float2half( 3.0 / 205.0);
	const fp b12_6 = __float2half( - 6.0 / 41.0);
	const fp b12_7 = __float2half( - 3.0 / 205.0);
	const fp b12_8 = __float2half( - 3.0 / 41.0);
	const fp b12_9 = __float2half( 3.0 / 41.0);
	const fp b12_10 = __float2half( 6.0 / 41.0);
	const fp b13_1 = __float2half( -1777.0 / 4100.0);
	const fp b13_4 = __float2half( -341.0 / 164.0);
	const fp b13_5 = __float2half( 4496.0 / 1025.0);
	const fp b13_6 = __float2half( -289.0 / 82.0);
	const fp b13_7 = __float2half( 2193.0 / 4100.0);
	const fp b13_8 = __float2half( 51.0 / 82.0);
	const fp b13_9 = __float2half( 33.0 / 164.0);
	const fp b13_10 = __float2half( 12.0 / 41.0);

	const fp err_factor  = __float2half( -41.0 / 840.0);

	fp h2_7 = a2 * h;

	fp timeinst_temp;

	int i,j;

	//======================================================================================================================================================
	//		EVALUATIONS
	//======================================================================================================================================================

	for(j=0; j<13; j++){

		//===================================================================================================
		//		0
		//===================================================================================================

		if(j==0){

			timeinst_temp = timeinst;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] ;
			}

		}

		//===================================================================================================
		//		1
		//===================================================================================================

		else if(j==1){

			timeinst_temp = timeinst+h2_7;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h2_7 * (finavalu_temp[0*EQUATIONS+i]);
			}

		}

		//===================================================================================================
		//		2
		//===================================================================================================

		else if(j==2){

			timeinst_temp = timeinst+a3*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b31*finavalu_temp[0*EQUATIONS+i] + b32*finavalu_temp[1*EQUATIONS+i]);
			}

		}

		//===================================================================================================
		//		3
		//===================================================================================================

		else if(j==3){

			timeinst_temp = timeinst+a4*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b41*finavalu_temp[0*EQUATIONS+i] + b43*finavalu_temp[2*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		4
		//===================================================================================================

		else if(j==4){

			timeinst_temp = timeinst+a5*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b51*finavalu_temp[0*EQUATIONS+i] + b53*finavalu_temp[2*EQUATIONS+i] + b54*finavalu_temp[3*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		5
		//===================================================================================================

		else if(j==5){

			timeinst_temp = timeinst+a6*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b61*finavalu_temp[0*EQUATIONS+i] + b64*finavalu_temp[3*EQUATIONS+i] + b65*finavalu_temp[4*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		6
		//===================================================================================================

		else if(j==6){

			timeinst_temp = timeinst+a7*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b71*finavalu_temp[0*EQUATIONS+i] + b74*finavalu_temp[3*EQUATIONS+i] + b75*finavalu_temp[4*EQUATIONS+i] + b76*finavalu_temp[5*EQUATIONS+i]);
			}

		}

		//===================================================================================================
		//		7
		//===================================================================================================

		else if(j==7){

			timeinst_temp = timeinst+a8*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b81*finavalu_temp[0*EQUATIONS+i] + b85*finavalu_temp[4*EQUATIONS+i] + b86*finavalu_temp[5*EQUATIONS+i] + b87*finavalu_temp[6*EQUATIONS+i]);
			}

		}

		//===================================================================================================
		//		8
		//===================================================================================================

		else if(j==8){

			timeinst_temp = timeinst+a9*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b91*finavalu_temp[0*EQUATIONS+i] + b94*finavalu_temp[3*EQUATIONS+i] + b95*finavalu_temp[4*EQUATIONS+i] + b96*finavalu_temp[5*EQUATIONS+i] + b97*finavalu_temp[6*EQUATIONS+i]+ b98*finavalu_temp[7*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		9
		//===================================================================================================

		else if(j==9){

			timeinst_temp = timeinst+a10*h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b10_1*finavalu_temp[0*EQUATIONS+i] + b10_4*finavalu_temp[3*EQUATIONS+i] + b10_5*finavalu_temp[4*EQUATIONS+i] + b10_6*finavalu_temp[5*EQUATIONS+i] + b10_7*finavalu_temp[6*EQUATIONS+i] + b10_8*finavalu_temp[7*EQUATIONS+i] + b10_9*finavalu_temp[8*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		10
		//===================================================================================================

		else if(j==10){

			timeinst_temp = timeinst+h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b11_1*finavalu_temp[0*EQUATIONS+i] + b11_4*finavalu_temp[3*EQUATIONS+i] + b11_5*finavalu_temp[4*EQUATIONS+i] + b11_6*finavalu_temp[5*EQUATIONS+i] + b11_7*finavalu_temp[6*EQUATIONS+i] + b11_8*finavalu_temp[7*EQUATIONS+i] + b11_9*finavalu_temp[8*EQUATIONS+i]+ b11_10 * finavalu_temp[9*EQUATIONS+i]);
			}

		}

		//===================================================================================================
		//		11
		//===================================================================================================

		else if(j==11){

			timeinst_temp = timeinst;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b12_1*finavalu_temp[0*EQUATIONS+i] + b12_6*finavalu_temp[5*EQUATIONS+i] + b12_7*finavalu_temp[6*EQUATIONS+i] + b12_8*finavalu_temp[7*EQUATIONS+i] + b12_9*finavalu_temp[8*EQUATIONS+i] + b12_10 * finavalu_temp[9*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		12
		//===================================================================================================

		else if(j==12){

			timeinst_temp = timeinst+h;
			for(i=0; i<EQUATIONS; i++){
				initvalu_temp[i] = initvalu[i] + h * ( b13_1*finavalu_temp[0*EQUATIONS+i] + b13_4*finavalu_temp[3*EQUATIONS+i] + b13_5*finavalu_temp[4*EQUATIONS+i] + b13_6*finavalu_temp[5*EQUATIONS+i] + b13_7*finavalu_temp[6*EQUATIONS+i] + b13_8*finavalu_temp[7*EQUATIONS+i] + b13_9*finavalu_temp[8*EQUATIONS+i] + b13_10*finavalu_temp[9*EQUATIONS+i] + finavalu_temp[11*EQUATIONS+i]) ;
			}

		}

		//===================================================================================================
		//		EVALUATION
		//===================================================================================================

		kernel_2(	(int)(__half2float(timeinst_temp)),
							initvalu_temp,
							parameter,
							&finavalu_temp[j*EQUATIONS],
							com);

	}

	//======================================================================================================================================================
	//		FINAL VALUE
	//======================================================================================================================================================

	for(i=0; i<EQUATIONS; i++){
		finavalu[i]= initvalu[i] +  h * (c_1_11 * (finavalu_temp[0*EQUATIONS+i] + finavalu_temp[10*EQUATIONS+i])  + c6 * finavalu_temp[5*EQUATIONS+i] + c_7_8 * (finavalu_temp[6*EQUATIONS+i] + finavalu_temp[7*EQUATIONS+i]) + c_9_10 * (finavalu_temp[8*EQUATIONS+i] + finavalu_temp[9*EQUATIONS+i]) );
		// printf("finavalu_temp[0][%d] = %f\n", i, finavalu_temp[0][i]);
		// printf("finavalu_temp[10][%d] = %f\n", i, finavalu_temp[10][i]);
		// printf("finavalu_temp[5][%d] = %f\n", i, finavalu_temp[5][i]);
		// printf("finavalu_temp[6][%d] = %f\n", i, finavalu_temp[6][i]);
		// printf("finavalu_temp[7][%d] = %f\n", i, finavalu_temp[7][i]);
		// printf("finavalu_temp[8][%d] = %f\n", i, finavalu_temp[8][i]);
		// printf("finavalu_temp[9][%d] = %f\n", i, finavalu_temp[9][i]);
		// printf("finavalu[%d] = %f\n", i, finavalu[i]);
	}

	//======================================================================================================================================================
	//		RETURN
	//======================================================================================================================================================

	for(i=0; i<EQUATIONS; i++){
		error[i] = fabs(err_factor * (finavalu_temp[0*EQUATIONS+i] + finavalu_temp[10*EQUATIONS+i] - finavalu_temp[11*EQUATIONS+i] - finavalu_temp[12*EQUATIONS+i]));
		// printf("Error[%d] = %f\n", i, error[i]);
	}

}
//======================================================================================================================================================
//======================================================================================================================================================
//		INCLUDE
//======================================================================================================================================================
//======================================================================================================================================================


//======================================================================================================================================================
//======================================================================================================================================================
//		SOLVER FUNCTION
//======================================================================================================================================================
//======================================================================================================================================================

__global__ void solver_2(	int workload,
											int xmax,

											fp* x,
											fp* y,
											fp* params,

											fp* com,
											fp* err,
											fp* scale,
											fp* yy,
											fp* initvalu_temp,
											fp* finavalu_temp){

	//========================================================================================================================
	//	VARIABLES
	//========================================================================================================================

	// CUDA indexes
	int bx;																					// get current horizontal block index (0-n)
	int tx;																					// get current horizontal thread index (0-n)
	int tid;																					// thread identifier

	// pointers
	long y_pointer_initial;
	long y_pointer_current;
	long x_pointer_current;
	int err_pointer;
	int scale_pointer;
	int yy_pointer;
	int params_pointer;
	int initvalu_temp_pointer;
	int finavalu_temp_pointer;
	int com_pointer;

	// solver parameters
	fp err_exponent ;
	fp h_init;
	fp h;
	fp tolerance;
	int xmin;

	// temporary solver variables
	int error;
	int outside;
	fp scale_min;
	fp scale_fina;

	// counters
	int i, j, k;

	//========================================================================================================================
	//		INITIAL SETUP
	//========================================================================================================================

	// CUDA indexes
	bx = blockIdx.x;																// get current horizontal block index (0-n)
	tx = threadIdx.x;																// get current horizontal thread index (0-n)
	tid = bx*NUMBER_THREADS+tx;

	// save pointers, these pointers are one per workload, independent of time step
	err_pointer = tid*EQUATIONS;
	scale_pointer = tid*EQUATIONS;
	yy_pointer = tid*EQUATIONS;
	params_pointer = tid*PARAMETERS;
	initvalu_temp_pointer = tid*EQUATIONS;
	finavalu_temp_pointer = tid*13*EQUATIONS;
	com_pointer = tid*3;

	// solver parameters
	err_exponent = __float2half(1.0 / 7.0);
	h_init = __float2half(1);
	h = h_init;
	xmin = 0;
	tolerance = __float2half(10 / (float)(xmax - xmin));

	//========================================================================================================================
	//		RANGE AND STEP CHECKING
	//========================================================================================================================

	// Verify that the step size is positive and that the upper endpoint of integration is greater than the initial enpoint.               //
	if (xmax < xmin || h <= 0.0){
		return;
	}

	// If the upper endpoint of the independent variable agrees with the initial value of the independent variable.  Set the value of the dependent variable and return success. //
	if (xmax == xmin){
		return;
	}

	// Insure that the step size h is not larger than the length of the integration interval.                                            //
	if (h > (xmax - xmin) ) {
		h = __float2half((float)xmax - (float)xmin);
	}

	//========================================================================================================================
	//		SOLVING IF THERE ARE MANY WORKLOADS
	//========================================================================================================================

	// limit to useful threads
	if(tid<workload){

		for(k=1; k<(xmax+1); k++) {											// start after initial value

			y_pointer_initial = tid*((xmax+1)*EQUATIONS)+(k-1)*EQUATIONS;
			y_pointer_current = tid*((xmax+1)*EQUATIONS)+k*EQUATIONS;
			x_pointer_current = tid*(xmax+1)+k;

			x[x_pointer_current] = __float2half(k);															// set this to k if you want time incremente with respect to previous to be k+h, set this to k-1 if you want the increment to be h
			h = h_init;

			//==========================================================================================
			//		REINITIALIZE VARIABLES
			//==========================================================================================

			scale_fina = __float2half(1.0);

			//==========================================================================================
			//		MAKE ATTEMPTS TO MINIMIZE ERROR
			//==========================================================================================

			// make attempts to minimize error
			for (j = 0; j < ATTEMPTS; j++) {

				//============================================================
				//		REINITIALIZE VARIABLES
				//============================================================

				error = 0;
				outside = 0;
				scale_min = __float2half(MAX_SCALE_FACTOR);

				//============================================================
				//		EVALUATE ALL EQUATIONS
				//============================================================

				embedded_fehlberg_7_8_2(	h,																												// single value

																x[x_pointer_current],																				// single value
																&y[y_pointer_initial],																					// 91 array
																&y[y_pointer_current],																				// 91 array
																&params[params_pointer],																		// 18 array

																&err[err_pointer],																						// 91 array
																&initvalu_temp[initvalu_temp_pointer],													// 91 array
																&finavalu_temp[finavalu_temp_pointer],													// 13*91 array
																&com[com_pointer]);																				// 3 array

				//============================================================
				//		IF THERE WAS NO ERROR FOR ANY OF EQUATIONS, SET SCALE AND LEAVE THE LOOP
				//============================================================

				for(i=0; i<EQUATIONS; i++){
					if(err[err_pointer+i] > 0){
						error = 1;
					}
				}
				if (error != 1) {
					scale_fina = __float2half(MAX_SCALE_FACTOR);
					break;
				}

				//============================================================
				//		FIGURE OUT SCALE AS THE MINIMUM OF COMPONENT SCALES
				//============================================================

				for(i=0; i<EQUATIONS; i++){
					if(y[y_pointer_initial+i] == 0.0){
						yy[yy_pointer+i] = tolerance;
					}
					else{
						yy[yy_pointer+i] = fabs(y[y_pointer_initial+i]);
					}
					scale[scale_pointer+i] = 0.8 * pow( tolerance * yy[yy_pointer+i] / err[err_pointer+i] , err_exponent );
					if(scale[scale_pointer+i]<scale_min){
						scale_min = scale[scale_pointer+i];
					}
				}
				//~ scale_fina = min( max(scale_min, __float2half(MIN_SCALE_FACTOR)), __float2half(MAX_SCALE_FACTOR));
				fp temp_max = scale_min;
				if (scale_min < MIN_SCALE_FACTOR)
					temp_max = __float2half(MIN_SCALE_FACTOR);
				
				scale_fina = temp_max ;
				if(temp_max > MAX_SCALE_FACTOR)
					scale_fina = __float2half(MAX_SCALE_FACTOR);
				

				//============================================================
				//		IF WITHIN TOLERANCE, FINISH ATTEMPTS...
				//============================================================

				for(i=0; i<EQUATIONS; i++){
					if ( err[err_pointer+i] > ( tolerance * yy[yy_pointer+i] ) ){
						outside = 1;
					}
				}
				if (outside == 0){
					break;
				}

				//============================================================
				//		...OTHERWISE, ADJUST STEP FOR NEXT ATTEMPT
				//============================================================

				// scale next step in a default way
				h = h * scale_fina;

				// limit step to 0.9, because when it gets close to 1, it no longer makes sense, as 1 is already the next time instance (added to original algorithm)
				if (h >= 0.9) {
					h = __float2half(0.9);
				}

				// if instance+step exceeds range limit, limit to that range
				if ( x[x_pointer_current] + h > (float)xmax ){
					h = (float)xmax - x[x_pointer_current];
				}

				// if getting closer to range limit, decrease step
				else if ( x[x_pointer_current] + h + 0.5 * h > (float)xmax ){
					h = 0.5 * h;
				}

			}

			//==========================================================================================
			//		SAVE TIME INSTANCE THAT SOLVER ENDED UP USING
			//==========================================================================================

			x[x_pointer_current] = x[x_pointer_current] + h;

			//==========================================================================================
			//		IF MAXIMUM NUMBER OF ATTEMPTS REACHED AND CANNOT GIVE SOLUTION, EXIT PROGRAM WITH ERROR
			//==========================================================================================

			if ( j >= ATTEMPTS ) {
				return;
			}

		}

	}

	//========================================================================================================================
	//		FINAL RETURN
	//========================================================================================================================

	return;

//======================================================================================================================================================
//======================================================================================================================================================
//		END OF SOLVER FUNCTION
//======================================================================================================================================================
//======================================================================================================================================================

}
