#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include "half_operator_overload.cuh"

__global__ void
bpnn_layerforward_CUDA(__half *input_cuda,
	                   __half *output_hidden_cuda,
					   __half *input_hidden_cuda,
					   __half *hidden_partial_sum,
					   int in,
					   int hid)
{
   int by = blockIdx.y;
   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;

   int index_in = HEIGHT * by + ty + 1;

   __shared__ __half input_node[HEIGHT];
   __shared__ __half weight_matrix[HEIGHT][WIDTH];


   if ( tx == 0 )
   input_node[ty] = input_cuda[index_in] ;

   __syncthreads();

   weight_matrix[ty][tx] = input_hidden_cuda[index];

   __syncthreads();

   weight_matrix[ty][tx] = weight_matrix[ty][tx] * input_node[ty];

   __syncthreads();

   for ( int i = 1 ; i <= __log2f(HEIGHT) ; i++){

	   int power_two = __powf(2, i);

	   if( ty % power_two == 0 )
	   weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];

	   __syncthreads();

   }

   //__syncthreads();

   input_hidden_cuda[index] = weight_matrix[ty][tx];

/*
   for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){

	   unsigned int power_two = i - 1;

	   if( (ty & power_two) == 0 ) {
		weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
	   }

   }
   */

   __syncthreads();

   if ( tx == 0 ) {
	   hidden_partial_sum[by * hid + ty] = weight_matrix[tx][ty];
   }

}


__global__ void bpnn_adjust_weights_cuda(__half * delta,
										 int hid,
										 __half * ly,
										 int in,
										 __half * w,
										 __half * oldw)
{


   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
   oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }


}
#endif
