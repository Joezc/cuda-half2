#include "hip/hip_runtime.h"
long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}
#include "newhalf.hpp"
#include <hip/hip_fp16.h>
#include "half_operator_overload.cuh"
__global__ void hotspotOpt1(__half *p, __half* tIn, __half *tOut, __half sdc,
        int nx, int ny, int nz,
        __half ce_t, __half cw_t,
        __half cn_t, __half cs_t,
        __half ct_t, __half cb_t,
        __half cc_t)
{

	__half ce = ce_t; __half cw = cw_t;
	__half cn = cn_t; __half cs = cs_t;
	__half ct = ct_t; __half cb = cb_t;
	__half cc = cc_t;
    __half amb_temp = __float2half(80.0);

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
		//add in for performance measurement
#pragma unroll 1
	for(int run = 0; run <100; run++){
    int c = i + j * nx;
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    __half temp1, temp2, temp3;

		__half tinw, tine, tins, tinn, pc;

		tinw = tIn[W];
		tine = tIn[E];
		tins =  tIn[S];
		tinn = tIn[N] ;
		pc = p[c];

    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tinw + ce * tine + cs * tins
        + cn * tinn + cb * temp1 + ct * temp3 + sdc * pc + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;
		#pragma unroll 1
    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];

				tinw = tIn[W];
				tine = tIn[E];
				tins =  tIn[S];
				tinn = tIn[N] ;
				pc = p[c];
				tOut[c] = cc * temp2 + cw * tinw + ce * tine + cs * tins
		        + cn * tinn + cb * temp1 + ct * temp3 + sdc * pc + ct * amb_temp;
    //    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
    //        + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;

		tinw = tIn[W];
		tine = tIn[E];
		tins =  tIn[S];
		tinn = tIn[N] ;
		pc = p[c];
		tOut[c] = cc * temp2 + cw * tinw + ce * tine + cs * tins
				+ cn * tinn + cb * temp1 + ct * temp3 + sdc * pc + ct * amb_temp;
  //  tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
  //      + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;

} //endof loop

    return;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap,
        float Rx, float Ry, float Rz,
        float dt, int numiter)
{
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;

		half_float::half* p_half;
		half_float::half* tIn_half;
		half_float::half* tOut_half;

		size_t half_size = sizeof(half)* nx * ny * nz;
		p_half = (half_float::half*)malloc(half_size);
		tIn_half =(half_float::half*)malloc(half_size);
		tOut_half = (half_float::half*)malloc(half_size);

    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(half) * nx * ny * nz;

		for(int i = 0; i < nx * ny * nz; i++ ){

			p_half[i] = p[i];
			tIn_half[i] = tIn[i];
		}



    __half  *tIn_d, *tOut_d, *p_d;

		printf ("val: %f %f %f %f %f %f %f %f \n",stepDivCap,ce,cw,cn,ct,cb,cc);
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpy(tIn_d, tIn_half, s, hipMemcpyHostToDevice);
    hipMemcpy(p_d, p_half, s, hipMemcpyHostToDevice);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx / 64, ny / 4, 1);

		half_float::half stepDivCap_half = half_float::half(stepDivCap);
		__half stdc = *(__half*)&(stepDivCap_half);

		half_float::half ce_half = half_float::half(ce);
		__half ce_dev = *(__half*)&ce_half;

		half_float::half cw_half = half_float::half(cw);
		__half cw_dev = *(__half*)&cw_half;

		half_float::half cn_half = half_float::half(cn);
		__half cn_dev = *(__half*)&cn_half;

		half_float::half cs_half = half_float::half(cs);
		__half cs_dev = *(__half*)&cs_half;

		half_float::half ct_half = half_float::half(ct);
		__half ct_dev = *(__half*)&ct_half;

		half_float::half cb_half = half_float::half(cb);
		__half cb_dev = *(__half*)&cb_half;

		half_float::half cc_half = half_float::half(cc);
		__half cc_dev = *(__half*)&cc_half;

    long long start = get_time();
    for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stdc, nx, ny, nz, ce_dev, cw_dev, cn_dev, cs_dev, ct_dev, cb_dev, cc_dev);
        __half *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }
    hipDeviceSynchronize();
    long long stop = get_time();
    float time = (float)((stop - start)/(1000.0 * 1000.0));
    printf("Time: %.3f (s)\n",time);
    hipMemcpy(tOut_half, tOut_d, s, hipMemcpyDeviceToHost);

		for(int i = 0; i < nx * ny * nz; i++ ){
			tOut [i] = float(tOut_half[i]);

}

    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}
