#include "hip/hip_runtime.h"
// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <iostream>
#include <fstream>
#include <hip/hip_fp16.h>
#include "half_operator_overload.cuh"
#include "newhalf.hpp"

 
/*
 * Options 
 * 
 */ 
#define GAMMA 1.4f
#define iterations 2000
// #ifndef block_length
// 	#define block_length 192
// #endif



#define NDIM 3
//~ #define NNB 4 //rounding error accumulated with halfprecision , use NNB = 2 for correctness checking
#define NNB 2

#define RK 3	// 3rd order RK
#define ff_mach 1.2f
#define deg_angle_of_attack 0.0f

/*
 * not options
 */

#ifdef RD_WG_SIZE_0_0
	#define BLOCK_SIZE_0 RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
	#define BLOCK_SIZE_0 RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_0 RD_WG_SIZE
#else
	#define BLOCK_SIZE_0 192
#endif

#ifdef RD_WG_SIZE_1_0
	#define BLOCK_SIZE_1 RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_1 RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_1 RD_WG_SIZE
#else
	#define BLOCK_SIZE_1 192
#endif

#ifdef RD_WG_SIZE_2_0
	#define BLOCK_SIZE_2 RD_WG_SIZE_2_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_2 RD_WG_SIZE_2
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_2 RD_WG_SIZE
#else
	#define BLOCK_SIZE_2 192
#endif

#ifdef RD_WG_SIZE_3_0
	#define BLOCK_SIZE_3 RD_WG_SIZE_3_0
#elif defined(RD_WG_SIZE_3)
	#define BLOCK_SIZE_3 RD_WG_SIZE_3
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_3 RD_WG_SIZE
#else
	#define BLOCK_SIZE_3 192
#endif

#ifdef RD_WG_SIZE_4_0
	#define BLOCK_SIZE_4 RD_WG_SIZE_4_0
#elif defined(RD_WG_SIZE_4)
	#define BLOCK_SIZE_4 RD_WG_SIZE_4
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_4 RD_WG_SIZE
#else
	#define BLOCK_SIZE_4 192
#endif



// #if block_length > 128
// #warning "the kernels may fail too launch on some systems if the block length is too large"
// #endif


#define VAR_DENSITY 0
#define VAR_MOMENTUM  1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)
#define NVAR (VAR_DENSITY_ENERGY+1)


/*
 * Generic functions
 */
template <typename T>
T* alloc(int N)
{
	T* t;
	checkCudaErrors(hipMalloc((void**)&t, sizeof(T)*N));
	return t;
}

template <typename T>
void dealloc(T* array)
{
	checkCudaErrors(hipFree((void*)array));
}

template <typename T>
void copy(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToDevice));
}

template <typename T>
void upload(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
}
template <typename T> //for half
void upload(T* dst, half_float::half* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
void download(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost));
}

template <typename T>
void download(half_float::half* dst, T* src, int N) //for half
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost));
}

void dump(half* variables, int nel, int nelr)
{
	float* h_variables = new float[nelr*NVAR];
	half_float::half* h_variables_half = new half_float::half[nelr*NVAR];
	
	download<half>(h_variables_half, variables, nelr*NVAR);
	for(int i = 0;i<nelr*NVAR;i++){
		h_variables[i] = float(h_variables_half[i]);
		
		}

	{
		std::ofstream file("density");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY*nelr] << std::endl;
	}


	{
		std::ofstream file("momentum");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++)
		{
			for(int j = 0; j != NDIM; j++)
				file << h_variables[i + (VAR_MOMENTUM+j)*nelr] << " ";
			file << std::endl;
		}
	}
	
	{
		std::ofstream file("density_energy");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY_ENERGY*nelr] << std::endl;
	}
	delete[] h_variables;
	delete[] h_variables_half;
}

/*
 * Element-based Cell-centered FVM solver functions
 */
__constant__ half ff_variable[NVAR];
__constant__ half3 ff_flux_contribution_momentum_x[1];
__constant__ half3 ff_flux_contribution_momentum_y[1];
__constant__ half3 ff_flux_contribution_momentum_z[1];
__constant__ half3 ff_flux_contribution_density_energy[1];

__global__ void cuda_initialize_variables(int nelr, half* variables)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	for(int j = 0; j < NVAR; j++)
		variables[i + j*nelr] = ff_variable[j];
}
void initialize_variables(int nelr, half* variables)
{
	dim3 Dg(nelr / BLOCK_SIZE_1), Db(BLOCK_SIZE_1);
	cuda_initialize_variables<<<Dg, Db>>>(nelr, variables);
	getLastCudaError("initialize_variables failed");
}

__device__ inline void compute_flux_contribution(half& density, half3& momentum, half& density_energy, half& pressure, half3& velocity, half3& fc_momentum_x, half3& fc_momentum_y, half3& fc_momentum_z, half3& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	half de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}
__host__ inline void compute_flux_contribution(half_float::half& density, half3_host& momentum, half_float::half& density_energy, half_float::half& pressure, half3_host& velocity, half3_host& fc_momentum_x, half3_host& fc_momentum_y, half3_host& fc_momentum_z, half3_host& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	half_float::half de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}
__device__ inline void compute_velocity(half& density, half3& momentum, half3& velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}
	
__device__ inline half compute_speed_sqd(half3& velocity)
{
	return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__device__ inline half compute_pressure(half& density, half& density_energy, half& speed_sqd)
{
	return (__float2half(GAMMA)-__float2half(1.0f))*(density_energy - __float2half(0.5f)*density*speed_sqd);
}

__device__ inline half compute_speed_of_sound(half& density, half& pressure)
{
	return sqrtf(__float2half(GAMMA)*pressure/density);
}

__global__ void cuda_compute_step_factor(int nelr, half* variables, half* areas, half* step_factors)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	half density = variables[i + VAR_DENSITY*nelr];
	half3 momentum;
	momentum.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum.z = variables[i + (VAR_MOMENTUM+2)*nelr];
	
	half density_energy = variables[i + VAR_DENSITY_ENERGY*nelr];
	
	half3 velocity;       compute_velocity(density, momentum, velocity);
	half speed_sqd      = compute_speed_sqd(velocity);
	half pressure       = compute_pressure(density, density_energy, speed_sqd);
	half speed_of_sound = compute_speed_of_sound(density, pressure);

	// dt = float(0.5f) * sqrtf(areas[i]) /  (||v|| + c).... but when we do time stepping, this later would need to be divided by the area, so we just do it all at once
	step_factors[i] = __float2half(0.5f) / (sqrtf(areas[i]) * (sqrtf(speed_sqd) + speed_of_sound));
}
void compute_step_factor(int nelr, half* variables, half* areas, half* step_factors)
{
	dim3 Dg(nelr / BLOCK_SIZE_2), Db(BLOCK_SIZE_2);
	cuda_compute_step_factor<<<Dg, Db>>>(nelr, variables, areas, step_factors);		
	getLastCudaError("compute_step_factor failed");
}

/*
 *
 *
*/
__global__ void cuda_compute_flux(int nelr, int* elements_surrounding_elements, half* normals, half* variables, half* fluxes)
{
	const half smoothing_coefficient = __float2half(0.2f);
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	
	int j, nb;
	half3 normal; half normal_len;
	half factor;
	
	half density_i = variables[i + VAR_DENSITY*nelr];
	half3 momentum_i;
	momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

	half density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

	half3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	half speed_sqd_i                          = compute_speed_sqd(velocity_i);
	half speed_i                              = sqrtf(speed_sqd_i);
	half pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	half speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	half3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	half3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
	half flux_i_density = __float2half(0.0f);
	half3 flux_i_momentum;
	flux_i_momentum.x = __float2half(0.0f);
	flux_i_momentum.y = __float2half(0.0f);
	flux_i_momentum.z = __float2half(0.0f);
	half flux_i_density_energy = __float2half(0.0f);
		
	half3 velocity_nb;
	half density_nb, density_energy_nb;
	half3 momentum_nb;
	half3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	half3 flux_contribution_nb_density_energy;	
	half speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
	
	#pragma unroll
	for(j = 0; j < NNB; j++)
	//~ for(j = 1; j < 4; j++)
	{
		nb = elements_surrounding_elements[i + j*nelr];
		normal.x = normals[i + (j + 0*NNB)*nelr];
		normal.y = normals[i + (j + 1*NNB)*nelr];
		normal.z = normals[i + (j + 2*NNB)*nelr];
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
		if(nb >= 0) 	// a legitimate neighbor
		{
			density_nb = variables[nb + VAR_DENSITY*nelr];
			momentum_nb.x = variables[nb + (VAR_MOMENTUM+0)*nelr];
			momentum_nb.y = variables[nb + (VAR_MOMENTUM+1)*nelr];
			momentum_nb.z = variables[nb + (VAR_MOMENTUM+2)*nelr];
			density_energy_nb = variables[nb + VAR_DENSITY_ENERGY*nelr];
			//compute_velocity(density_nb, momentum_nb, velocity_nb);
			//calling inline functions is bad for automatic parsing, use below instead
			velocity_nb.x = momentum_nb.x / density_nb;
			velocity_nb.y = momentum_nb.y / density_nb;
			velocity_nb.z = momentum_nb.z / density_nb;	
			
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
//			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
//compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
////calling inline functions is bad for automatic parsing, use below instead
//compute_flux_contribution(half2& density, half2_3& momentum, half2& density_energy, half2& pressure, half2_3& velocity, half2_3& fc_momentum_x, half2_3& fc_momentum_y, half2_3& fc_momentum_z, half2_3& fc_density_energy)
			flux_contribution_nb_momentum_x.x = velocity_nb.x*momentum_nb.x + pressure_nb;
			flux_contribution_nb_momentum_x.y = velocity_nb.x*momentum_nb.y;
			flux_contribution_nb_momentum_x.z = velocity_nb.x*momentum_nb.z;
	
	
			flux_contribution_nb_momentum_y.x = flux_contribution_nb_momentum_x.y;
			flux_contribution_nb_momentum_y.y = velocity_nb.y*momentum_nb.y + pressure_nb;
			flux_contribution_nb_momentum_y.z = velocity_nb.y*momentum_nb.z;

			flux_contribution_nb_momentum_z.x = flux_contribution_nb_momentum_x.z;
			flux_contribution_nb_momentum_z.y = flux_contribution_nb_momentum_y.z;
			flux_contribution_nb_momentum_z.z = velocity_nb.z*momentum_nb.z + pressure_nb;

			half de_p = density_energy_nb+pressure_nb;
			flux_contribution_nb_density_energy.x = velocity_nb.x*de_p;
			flux_contribution_nb_density_energy.y = velocity_nb.y*de_p;
			flux_contribution_nb_density_energy.z = velocity_nb.z*de_p;	
				
			//end compute_flux_contribution
			

			
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
		else if(nb == -1)	// a wing boundary
		{
			flux_i_momentum.x += normal.x*pressure_i;
			flux_i_momentum.y += normal.y*pressure_i;
			flux_i_momentum.z += normal.z*pressure_i;
		}
		else if(nb == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

		}
		
	}

	fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
	fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
	fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
	fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
	fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
}
void compute_flux(int nelr, int* elements_surrounding_elements, half* normals, half* variables, half* fluxes)
{
	dim3 Dg(nelr / BLOCK_SIZE_3), Db(BLOCK_SIZE_3);
	cuda_compute_flux<<<Dg,Db>>>(nelr, elements_surrounding_elements, normals, variables, fluxes);
	getLastCudaError("compute_flux failed");
}

__global__ void cuda_time_step(int j, int nelr, half* old_variables, half* variables, half* step_factors, half* fluxes)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	half factor = step_factors[i]/float(RK+1-j);

	variables[i + VAR_DENSITY*nelr] = old_variables[i + VAR_DENSITY*nelr] + factor*fluxes[i + VAR_DENSITY*nelr];
	variables[i + VAR_DENSITY_ENERGY*nelr] = old_variables[i + VAR_DENSITY_ENERGY*nelr] + factor*fluxes[i + VAR_DENSITY_ENERGY*nelr];
	variables[i + (VAR_MOMENTUM+0)*nelr] = old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
	variables[i + (VAR_MOMENTUM+1)*nelr] = old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];	
	variables[i + (VAR_MOMENTUM+2)*nelr] = old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];	
	//~ variables[i + (VAR_MOMENTUM+0)*nelr] = fluxes[i + (VAR_MOMENTUM+0)*nelr];// old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
	//~ variables[i + (VAR_MOMENTUM+1)*nelr] = fluxes[i + (VAR_MOMENTUM+1)*nelr];//old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];	
	//~ variables[i + (VAR_MOMENTUM+2)*nelr] = fluxes[i + (VAR_MOMENTUM+2)*nelr];//old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];	
}
void time_step(int j, int nelr, half* old_variables, half* variables, half* step_factors, half* fluxes)
{
	dim3 Dg(nelr / BLOCK_SIZE_4), Db(BLOCK_SIZE_4);
	cuda_time_step<<<Dg,Db>>>(j, nelr, old_variables, variables, step_factors, fluxes);
	getLastCudaError("update failed");
}

/*
 * Main function
 */
int main(int argc, char** argv)
{
  printf("WG size of kernel:initialize = %d, WG size of kernel:compute_step_factor = %d, WG size of kernel:compute_flux = %d, WG size of kernel:time_step = %d\n", BLOCK_SIZE_1, BLOCK_SIZE_2, BLOCK_SIZE_3, BLOCK_SIZE_4);

	if (argc < 2)
	{
		std::cout << "specify data file name" << std::endl;
		return 0;
	}
	const char* data_file_name = argv[1];
	
	hipDeviceProp_t prop;
	int dev;
	
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipGetDevice(&dev));
	checkCudaErrors(hipGetDeviceProperties(&prop, dev));
	
	printf("Name:                     %s\n", prop.name);

	// set far field conditions and load them into constant memory on the gpu
	{
		half_float::half h_ff_variable[NVAR];
		const float angle_of_attack = float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);
		
		h_ff_variable[VAR_DENSITY] = half_float::half(1.4);
		
		half_float::half ff_pressure = half_float::half(1.0f);
		half_float::half ff_speed_of_sound = half_float::half(sqrt(GAMMA*ff_pressure / h_ff_variable[VAR_DENSITY]));
		half_float::half ff_speed = half_float::half(ff_mach)*ff_speed_of_sound;
		
		half3_host ff_velocity;
		ff_velocity.x = ff_speed*float(cos((float)angle_of_attack));
		ff_velocity.y = ff_speed*float(sin((float)angle_of_attack));
		ff_velocity.z = 0.0f;
		
		h_ff_variable[VAR_MOMENTUM+0] = h_ff_variable[VAR_DENSITY] * ff_velocity.x;
		h_ff_variable[VAR_MOMENTUM+1] = h_ff_variable[VAR_DENSITY] * ff_velocity.y;
		h_ff_variable[VAR_MOMENTUM+2] = h_ff_variable[VAR_DENSITY] * ff_velocity.z;
				
		h_ff_variable[VAR_DENSITY_ENERGY] = h_ff_variable[VAR_DENSITY]*(float(0.5f)*(ff_speed*ff_speed)) + (ff_pressure / float(GAMMA-1.0f));

		half3_host h_ff_momentum;
		h_ff_momentum.x = *(h_ff_variable+VAR_MOMENTUM+0);
		h_ff_momentum.y = *(h_ff_variable+VAR_MOMENTUM+1);
		h_ff_momentum.z = *(h_ff_variable+VAR_MOMENTUM+2);
		half3_host h_ff_flux_contribution_momentum_x;
		half3_host h_ff_flux_contribution_momentum_y;
		half3_host h_ff_flux_contribution_momentum_z;
		half3_host h_ff_flux_contribution_density_energy;
		
		compute_flux_contribution(h_ff_variable[VAR_DENSITY], h_ff_momentum, h_ff_variable[VAR_DENSITY_ENERGY], ff_pressure, ff_velocity, h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy);

		// copy far field conditions to the gpu
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),          h_ff_variable,          NVAR*sizeof(half)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(half3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(half3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(half3)) );
		
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(half3)) );		
	}
	int nel;
	int nelr;
	
	// read in domain geometry
	half* areas;
	half* normals;
	
	
	int* elements_surrounding_elements;
	//device mem 
	
	{
		std::ifstream file(data_file_name);
	
		file >> nel;
		nelr = BLOCK_SIZE_0*((nel / BLOCK_SIZE_0 )+ std::min(1, nel % BLOCK_SIZE_0));

		float* h_areas = new float[nelr];
		int* h_elements_surrounding_elements = new int[nelr*NNB];
		float* h_normals = new float[nelr*NDIM*NNB];

				
		// read in data
		for(int i = 0; i < nel; i++)
		{
			file >> h_areas[i];
			for(int j = 0; j < NNB; j++)
			{
				file >> h_elements_surrounding_elements[i + j*nelr];
				if(h_elements_surrounding_elements[i+j*nelr] < 0) h_elements_surrounding_elements[i+j*nelr] = -1;
				h_elements_surrounding_elements[i + j*nelr]--; //it's coming in with Fortran numbering				
				
				for(int k = 0; k < NDIM; k++)
				{
					file >> h_normals[i + (j + k*NNB)*nelr];
					h_normals[i + (j + k*NNB)*nelr] = -h_normals[i + (j + k*NNB)*nelr];
				}
			}
		}
		
		// fill in remaining data
		int last = nel-1;
		for(int i = nel; i < nelr; i++)
		{
			h_areas[i] = h_areas[last];
			for(int j = 0; j < NNB; j++)
			{
				// duplicate the last element
				h_elements_surrounding_elements[i + j*nelr] = h_elements_surrounding_elements[last + j*nelr];	
				for(int k = 0; k < NDIM; k++) h_normals[last + (j + k*NNB)*nelr] = h_normals[last + (j + k*NNB)*nelr];
			}
		}
		
		half_float::half* h_areas_half = new half_float::half[nelr];
		half_float::half* h_normals_half = new half_float::half[nelr*NDIM*NNB];
		
		for(int i= 0;i<nelr; i++){
			h_areas_half[i] = half_float::half(h_areas[i]);
			}
		for(int i=0;i<nelr*NDIM*NNB; i++){
			h_normals_half[i] = half_float::half(h_normals[i]);
			
			}
		
		
		areas = alloc<half>(nelr);
		
		upload<half>(areas, h_areas_half, nelr);

		elements_surrounding_elements = alloc<int>(nelr*NNB);
		
		upload<int>(elements_surrounding_elements, h_elements_surrounding_elements, nelr*NNB);

		normals = alloc<half>(nelr*NDIM*NNB);
		upload<half>(normals, h_normals_half, nelr*NDIM*NNB);
				
		delete[] h_areas;
		delete[] h_elements_surrounding_elements;
		delete[] h_normals;
		delete[] h_areas_half;
		delete[] h_normals_half;
	}

	// Create arrays and set initial conditions
	half* variables = alloc<half>(nelr*NVAR);
	initialize_variables(nelr, variables);

	half* old_variables = alloc<half>(nelr*NVAR);   	
	half* fluxes = alloc<half>(nelr*NVAR);
	half* step_factors = alloc<half>(nelr); 

	// make sure all memory is floatly allocated before we start timing
	initialize_variables(nelr, old_variables);
	initialize_variables(nelr, fluxes);
	
	hipMemset( (void*) step_factors, 0, sizeof(half)*nelr );
	// make sure CUDA isn't still doing something before we start timing
	hipDeviceSynchronize();

	// these need to be computed the first time in order to compute time step
	std::cout << "Starting..." << std::endl;

	StopWatchInterface *timer = 0;
	  //	unsigned int timer = 0;

	// CUT_SAFE_CALL( cutCreateTimer( &timer));
	// CUT_SAFE_CALL( cutStartTimer( timer));
	sdkCreateTimer(&timer); 
	sdkStartTimer(&timer); 
	// Begin iterations
	for(int i = 0; i < iterations; i++)
	{
		copy<half>(old_variables, variables, nelr*NVAR);
		
		// for the first iteration we compute the time step
		compute_step_factor(nelr, variables, areas, step_factors);
		getLastCudaError("compute_step_factor failed");
	
		//~ int j =0;
		for(int j = 0; j < RK; j++)
		{
			compute_flux(nelr, elements_surrounding_elements, normals, variables, fluxes);
			getLastCudaError("compute_flux failed");			
			time_step(j, nelr, old_variables, variables, step_factors, fluxes);
			getLastCudaError("time_step failed");			
		}
	}

	hipDeviceSynchronize();
	//	CUT_SAFE_CALL( cutStopTimer(timer) );  
	sdkStopTimer(&timer); 

	std::cout  << (sdkGetAverageTimerValue(&timer)/1000.0)  / iterations << " seconds per iteration" << std::endl;

	std::cout << "Saving solution..." << std::endl;
	dump(variables, nel, nelr);
	std::cout << "Saved solution..." << std::endl;

	
	std::cout << "Cleaning up..." << std::endl;
	dealloc<half>(areas);
	dealloc<int>(elements_surrounding_elements);
	dealloc<half>(normals);
	
	dealloc<half>(variables);
	dealloc<half>(old_variables);
	dealloc<half>(fluxes);
	dealloc<half>(step_factors);

	std::cout << "Done..." << std::endl;

	return 0;
}
