#include "hip/hip_runtime.h"


#ifndef _BACKPROP_CUDA_KERNEL_H_
#define _BACKPROP_CUDA_KERNEL_H_

#include <stdio.h>
#include "backprop.h"
#include "math.h"
#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include "half2_operator_overload.cuh"

__global__ void
bpnn_layerforward_CUDA(__half2 *input_cuda,
	                   __half2 *output_hidden_cuda,
					   __half2 *input_hidden_cuda,
					   __half2 *hidden_partial_sum,
					   int in,
					   int hid)
{
   int by = blockIdx.y;
   int tx = threadIdx.x;//div-ed by 2 
   int ty = threadIdx.y;

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;

   int index_in = HEIGHT * by + ty + 1;

   __shared__ __half2 input_node[HEIGHT];
   __shared__ __half2 weight_matrix[HEIGHT][WIDTH/2];


   if ( tx == 0 )
   input_node[ty] = input_cuda[index_in/2] ;

   __syncthreads();

   weight_matrix[ty][tx] = input_hidden_cuda[index/2];

   __syncthreads();

   weight_matrix[ty][tx] = weight_matrix[ty][tx] * input_node[ty];

   __syncthreads();

   for ( int i = 1 ; i <= __log2f(HEIGHT) ; i++){ //reduction in y dimension, must group them in SIMD X-DIM

	   int power_two = __powf(2, i);

	   if( ty % power_two == 0 )
	   weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];

	   __syncthreads();

   }

   //__syncthreads();

   input_hidden_cuda[index/2] = weight_matrix[ty][tx];

/*
   for ( unsigned int i = 2 ; i <= HEIGHT ; i *= 2){

	   unsigned int power_two = i - 1;

	   if( (ty & power_two) == 0 ) {
		weight_matrix[ty][tx] = weight_matrix[ty][tx] + weight_matrix[ty + power_two/2][tx];
	   }

   }
   */

   __syncthreads();

   if ( tx == 0 ) {
	   hidden_partial_sum[by * hid/2 + ty] = weight_matrix[tx][ty];
   }

}


__global__ void bpnn_adjust_weights_cuda(__half2 * delta,
										 int hid,
										 __half2 * ly,
										 int in,
										 __half2 * w,
										 __half2 * oldw)
{


   int by = blockIdx.y;

   int tx = threadIdx.x;
   int ty = threadIdx.y;

   int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 ) ;
   int index_y = HEIGHT * by + ty + 1;
   int index_x = tx + 1;
   //eta = 0.3;
   //momentum = 0.3;

   w[index] += ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));
   oldw[index] = ((ETA * delta[index_x] * ly[index_y]) + (MOMENTUM * oldw[index]));


   __syncthreads();

   if (ty == 0 && by ==0){
   w[index_x] += ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   oldw[index_x] = ((ETA * delta[index_x]) + (MOMENTUM * oldw[index_x]));
   }


}
#endif
