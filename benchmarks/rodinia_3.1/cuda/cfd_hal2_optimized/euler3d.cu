#include "hip/hip_runtime.h"
// Copyright 2009, Andrew Corrigan, acorriga@gmu.edu
// This code is from the AIAA-2009-4001 paper

//#include <cutil.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include <iostream>
#include <fstream>
#include <hip/hip_fp16.h>
#include "half_operator_overload.cuh"
#include "half2_operator_overload.cuh"
#include "newhalf.hpp"

 
/*
 * Options 
 * 
 */ 
#define GAMMA 1.4f
#define iterations 2000
// #ifndef block_length
// 	#define block_length 192
// #endif



#define NDIM 3
//~ #define NNB 4 //rounding error accumulated with halfprecision , use NNB = 2 for correctness checking
#define NNB 2

#define RK 3	// 3rd order RK
#define ff_mach 1.2f
#define deg_angle_of_attack 0.0f

/*
 * not options
 */

#ifdef RD_WG_SIZE_0_0
	#define BLOCK_SIZE_0 RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
	#define BLOCK_SIZE_0 RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_0 RD_WG_SIZE
#else
	#define BLOCK_SIZE_0 192
#endif

#ifdef RD_WG_SIZE_1_0
	#define BLOCK_SIZE_1 RD_WG_SIZE_1_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_1 RD_WG_SIZE_1
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_1 RD_WG_SIZE
#else
	#define BLOCK_SIZE_1 192
#endif

#ifdef RD_WG_SIZE_2_0
	#define BLOCK_SIZE_2 RD_WG_SIZE_2_0
#elif defined(RD_WG_SIZE_1)
	#define BLOCK_SIZE_2 RD_WG_SIZE_2
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_2 RD_WG_SIZE
#else
	#define BLOCK_SIZE_2 192
#endif

#ifdef RD_WG_SIZE_3_0
	#define BLOCK_SIZE_3 RD_WG_SIZE_3_0
#elif defined(RD_WG_SIZE_3)
	#define BLOCK_SIZE_3 RD_WG_SIZE_3
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_3 RD_WG_SIZE
#else
	#define BLOCK_SIZE_3 192
#endif

#ifdef RD_WG_SIZE_4_0
	#define BLOCK_SIZE_4 RD_WG_SIZE_4_0
#elif defined(RD_WG_SIZE_4)
	#define BLOCK_SIZE_4 RD_WG_SIZE_4
#elif defined(RD_WG_SIZE)
	#define BLOCK_SIZE_4 RD_WG_SIZE
#else
	#define BLOCK_SIZE_4 192
#endif



// #if block_length > 128
// #warning "the kernels may fail too launch on some systems if the block length is too large"
// #endif


#define VAR_DENSITY 0
#define VAR_MOMENTUM  1
#define VAR_DENSITY_ENERGY (VAR_MOMENTUM+NDIM)
#define NVAR (VAR_DENSITY_ENERGY+1)


/*
 * Generic functions
 */
template <typename T>
T* alloc(int N)
{
	T* t;
	checkCudaErrors(hipMalloc((void**)&t, sizeof(T)*N));
	return t;
}

template <typename T>
void dealloc(T* array)
{
	checkCudaErrors(hipFree((void*)array));
}

template <typename T>
void copy(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToDevice));
}

template <typename T>
void upload(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
}
template <typename T> //for half
void upload(T* dst, half_float::half* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
void download(T* dst, T* src, int N)
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost));
}

template <typename T>
void download(half_float::half* dst, T* src, int N) //for half
{
	checkCudaErrors(hipMemcpy((void*)dst, (void*)src, N*sizeof(T), hipMemcpyDeviceToHost));
}

void dump(half* variables, int nel, int nelr)
{
	float* h_variables = new float[nelr*NVAR];
	half_float::half* h_variables_half = new half_float::half[nelr*NVAR];
	
	download<half>(h_variables_half, variables, nelr*NVAR);
	for(int i = 0;i<nelr*NVAR;i++){
		h_variables[i] = float(h_variables_half[i]);
		
		}

	{
		std::ofstream file("density");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY*nelr] << std::endl;
	}


	{
		std::ofstream file("momentum");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++)
		{
			for(int j = 0; j != NDIM; j++)
				file << h_variables[i + (VAR_MOMENTUM+j)*nelr] << " ";
			file << std::endl;
		}
	}
	
	{
		std::ofstream file("density_energy");
		file << nel << " " << nelr << std::endl;
		for(int i = 0; i < nel; i++) file << h_variables[i + VAR_DENSITY_ENERGY*nelr] << std::endl;
	}
	delete[] h_variables;
	delete[] h_variables_half;
}

/*
 * Element-based Cell-centered FVM solver functions
 */
__constant__ half ff_variable[NVAR];
__constant__ half3 ff_flux_contribution_momentum_x[1];
__constant__ half3 ff_flux_contribution_momentum_y[1];
__constant__ half3 ff_flux_contribution_momentum_z[1];
__constant__ half3 ff_flux_contribution_density_energy[1];

__global__ void cuda_initialize_variables(int nelr, half2* variables)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);
	for(int j = 0; j < NVAR; j++)
		variables[i + j*nelr] = __half2half2(ff_variable[j]);
}
void initialize_variables(int nelr, half2* variables)
{
	dim3 Dg(nelr / BLOCK_SIZE_1), Db(BLOCK_SIZE_1);
	cuda_initialize_variables<<<Dg, Db>>>(nelr, variables);
	getLastCudaError("initialize_variables failed");
}

__device__ inline void compute_flux_contribution(half2& density, half2_3& momentum, half2& density_energy, half2& pressure, half2_3& velocity, half2_3& fc_momentum_x, half2_3& fc_momentum_y, half2_3& fc_momentum_z, half2_3& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	half2 de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}
__host__ inline void compute_flux_contribution(half_float::half& density, half3_host& momentum, half_float::half& density_energy, half_float::half& pressure, half3_host& velocity, half3_host& fc_momentum_x, half3_host& fc_momentum_y, half3_host& fc_momentum_z, half3_host& fc_density_energy)
{
	fc_momentum_x.x = velocity.x*momentum.x + pressure;
	fc_momentum_x.y = velocity.x*momentum.y;
	fc_momentum_x.z = velocity.x*momentum.z;
	
	
	fc_momentum_y.x = fc_momentum_x.y;
	fc_momentum_y.y = velocity.y*momentum.y + pressure;
	fc_momentum_y.z = velocity.y*momentum.z;

	fc_momentum_z.x = fc_momentum_x.z;
	fc_momentum_z.y = fc_momentum_y.z;
	fc_momentum_z.z = velocity.z*momentum.z + pressure;

	half_float::half de_p = density_energy+pressure;
	fc_density_energy.x = velocity.x*de_p;
	fc_density_energy.y = velocity.y*de_p;
	fc_density_energy.z = velocity.z*de_p;
}
__device__ inline void compute_velocity(half2& density, half2_3& momentum, half2_3& velocity)
{
	velocity.x = momentum.x / density;
	velocity.y = momentum.y / density;
	velocity.z = momentum.z / density;
}
	
__device__ inline half2 compute_speed_sqd(half2_3& velocity)
{
	return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
}

__device__ inline half2 compute_pressure(half2& density, half2& density_energy, half2& speed_sqd)
{
	return (__float2half2_rn(GAMMA)-__float2half2_rn(1.0f))*(density_energy - __float2half2_rn(0.5f)*density*speed_sqd);
}

__device__ inline half2 compute_speed_of_sound(half2& density, half2& pressure)
{
	return sqrtf(__float2half2_rn(GAMMA)*pressure/density);
}

__global__ void cuda_compute_step_factor(int nelr, half2* variables, half2* areas, half2* step_factors)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	half2 density = variables[i + VAR_DENSITY*nelr];
	half2_3 momentum;
	momentum.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum.z = variables[i + (VAR_MOMENTUM+2)*nelr];
	
	half2 density_energy = variables[i + VAR_DENSITY_ENERGY*nelr];
	
	half2_3 velocity;       compute_velocity(density, momentum, velocity);
	half2 speed_sqd      = compute_speed_sqd(velocity);
	half2 pressure       = compute_pressure(density, density_energy, speed_sqd);
	half2 speed_of_sound = compute_speed_of_sound(density, pressure);

	// dt = float(0.5f) * sqrtf(areas[i]) /  (||v|| + c).... but when we do time stepping, this later would need to be divided by the area, so we just do it all at once
	step_factors[i] = __float2half2_rn(0.5f) / (sqrtf(areas[i]) * (sqrtf(speed_sqd) + speed_of_sound));
}
void compute_step_factor(int nelr, half2* variables, half2* areas, half2* step_factors)
{
	dim3 Dg(nelr / BLOCK_SIZE_2), Db(BLOCK_SIZE_2);
	cuda_compute_step_factor<<<Dg, Db>>>(nelr, variables, areas, step_factors);		
	getLastCudaError("compute_step_factor failed");
}

/*
 *
 *
*/
__global__ void cuda_compute_flux(int nelr, int* elements_surrounding_elements, half2* normals, half2* variables, half2* fluxes)
{
	const half2 smoothing_coefficient = __float2half2_rn(0.2f);
	const int i = (blockDim.x*blockIdx.x + threadIdx.x); //i div-ed by 2 , nelr div-ed by 2
	half* variables_half = (half*)(variables);
	int j, nb;
	int nb1,nb2;
	half2_3 normal; half2 normal_len;
	half2 factor;
	
	half2 density_i = variables[i + VAR_DENSITY*nelr];
	half2_3 momentum_i;
	momentum_i.x = variables[i + (VAR_MOMENTUM+0)*nelr];
	momentum_i.y = variables[i + (VAR_MOMENTUM+1)*nelr];
	momentum_i.z = variables[i + (VAR_MOMENTUM+2)*nelr];

	half2 density_energy_i = variables[i + VAR_DENSITY_ENERGY*nelr];

	half2_3 velocity_i;             				compute_velocity(density_i, momentum_i, velocity_i);
	half2 speed_sqd_i                          = compute_speed_sqd(velocity_i);
	half2 speed_i                              = sqrtf(speed_sqd_i);
	half2 pressure_i                           = compute_pressure(density_i, density_energy_i, speed_sqd_i);
	half2 speed_of_sound_i                     = compute_speed_of_sound(density_i, pressure_i);
	half2_3 flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z;
	half2_3 flux_contribution_i_density_energy;	
	compute_flux_contribution(density_i, momentum_i, density_energy_i, pressure_i, velocity_i, flux_contribution_i_momentum_x, flux_contribution_i_momentum_y, flux_contribution_i_momentum_z, flux_contribution_i_density_energy);
	
	half2 flux_i_density = __float2half2_rn(0.0f);
	half2_3 flux_i_momentum;
	flux_i_momentum.x = __float2half2_rn(0.0f);
	flux_i_momentum.y = __float2half2_rn(0.0f);
	flux_i_momentum.z = __float2half2_rn(0.0f);
	half2 flux_i_density_energy = __float2half2_rn(0.0f);
		
	half2_3 velocity_nb;
	half2 density_nb, density_energy_nb;
	half2_3 momentum_nb;
	half2_3 flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z;
	half2_3 flux_contribution_nb_density_energy;	
	half2 speed_sqd_nb, speed_of_sound_nb, pressure_nb;
	
	
	//~ #pragma unroll
	for(j = 0; j < NNB; j++)
	//~ for(j = 1; j < 4; j++)
	{
		nb1 = elements_surrounding_elements[(i + j*nelr)*2];
		nb2 = elements_surrounding_elements[(i + j*nelr)*2+1];
		
		normal.x = normals[i + (j + 0*NNB)*nelr];
		normal.y = normals[i + (j + 1*NNB)*nelr];
		normal.z = normals[i + (j + 2*NNB)*nelr];
		normal_len = sqrtf(normal.x*normal.x + normal.y*normal.y + normal.z*normal.z);
		
		
		//prologue
		
		half2 density_nb_masked_if_1;// = density_nb;
		half2 momentum_nb_x_masked_if_1;// = momentum_nb.x;
		half2 momentum_nb_y_masked_if_1;// = momentum_nb.y;
		half2 momentum_nb_z_masked_if_1;// = momentum_nb.z;
		half2 density_energy_nb_masked_if_1;// = density_energy_nb;
		half2 velocity_nb_x_masked_if_1;// = velocity_nb.x;
		half2 velocity_nb_y_masked_if_1;// = velocity_nb.y;
		half2 velocity_nb_z_masked_if_1;// = velocity_nb.z;
		half2 speed_sqd_nb_masked_if_1;// = speed_sqd_nb;
		half2 pressure_nb_masked_if_1;// = pressure_nb;
		half2 speed_of_sound_nb_masked_if_1;// = speed_of_sound_nb;	
		//~ half2 flux_contribution_nb_momentum_x_x_masked_if_1 = flux_contribution_nb_momentum_x.x;
		//~ half2 flux_contribution_nb_momentum_x_y_masked_if_1 = flux_contribution_nb_momentum_x.y;
		//~ half2 flux_contribution_nb_momentum_x_z_masked_if_1 = flux_contribution_nb_momentum_x.z;
		
		//~ half2 flux_contribution_nb_momentum_y_x_masked_if_1 = flux_contribution_nb_momentum_y.x;
		//~ half2 flux_contribution_nb_momentum_y_y_masked_if_1 = flux_contribution_nb_momentum_y.y;
		//~ half2 flux_contribution_nb_momentum_y_z_masked_if_1 = flux_contribution_nb_momentum_y.z;
		
		//~ half2 flux_contribution_nb_momentum_z_x_masked_if_1 = flux_contribution_nb_momentum_z.x;
		//~ half2 flux_contribution_nb_momentum_z_y_masked_if_1 = flux_contribution_nb_momentum_z.y;
		//~ half2 flux_contribution_nb_momentum_z_z_masked_if_1 = flux_contribution_nb_momentum_z.z;
		
		//~ half2 flux_contribution_nb_density_energy_x_masked_if_1 = flux_contribution_nb_density_energy.x;
		//~ half2 flux_contribution_nb_density_energy_y_masked_if_1 = flux_contribution_nb_density_energy.y;
		//~ half2 flux_contribution_nb_density_energy_z_masked_if_1 = flux_contribution_nb_density_energy.z;
		
		half2 factor_masked_if_1;// = factor;
		half2 flux_i_density_masked_if_1 = flux_i_density;
		half2 flux_i_density_energy_masked_if_1 = flux_i_density_energy;
		half2 flux_i_momentum_x_masked_if_1 = flux_i_momentum.x;
		half2 flux_i_momentum_y_masked_if_1 = flux_i_momentum.y;
		half2 flux_i_momentum_z_masked_if_1 = flux_i_momentum.z;
		
		bool flag1_if_1 = (nb1 >= 0);
		bool flag2_if_1 = (nb2 >= 0);
		
		//end prologue for if_1 : if(nb >= 0) 
		
		//~ if(nb >= 0) 	// a legitimate neighbor
		//~ {
			
			((half*)&density_nb_masked_if_1)[0] = variables_half[nb1 + VAR_DENSITY*nelr*2];
			((half*)&momentum_nb_x_masked_if_1)[0] = variables_half[nb1 + (VAR_MOMENTUM+0)*nelr*2];
			((half*)&momentum_nb_y_masked_if_1)[0] = variables_half[nb1 + (VAR_MOMENTUM+1)*nelr*2];
			((half*)&momentum_nb_z_masked_if_1)[0] = variables_half[nb1 + (VAR_MOMENTUM+2)*nelr*2];
			((half*)&density_energy_nb_masked_if_1)[0] = variables_half[nb1 + VAR_DENSITY_ENERGY*nelr*2];
			
			((half*)&density_nb_masked_if_1)[1] = variables_half[nb2 + VAR_DENSITY*nelr*2];
			((half*)&momentum_nb_x_masked_if_1)[1] = variables_half[nb2 + (VAR_MOMENTUM+0)*nelr*2];
			((half*)&momentum_nb_y_masked_if_1)[1] = variables_half[nb2 + (VAR_MOMENTUM+1)*nelr*2];
			((half*)&momentum_nb_z_masked_if_1)[1] = variables_half[nb2 + (VAR_MOMENTUM+2)*nelr*2];
			((half*)&density_energy_nb_masked_if_1)[1] = variables_half[nb2 + VAR_DENSITY_ENERGY*nelr*2];
			
			//compute_velocity(density_nb, momentum_nb, velocity_nb);
			//calling inline functions is bad for automatic parsing, use below instead
			velocity_nb_x_masked_if_1 = momentum_nb_x_masked_if_1 / density_nb_masked_if_1;
			velocity_nb_y_masked_if_1 = momentum_nb_y_masked_if_1 / density_nb_masked_if_1;
			velocity_nb_z_masked_if_1 = momentum_nb_z_masked_if_1 / density_nb_masked_if_1;	
			
											
			//~ speed_sqd_nb_masked_if_1                      = compute_speed_sqd(velocity_nb_masked_if_1);
			speed_sqd_nb_masked_if_1                      = velocity_nb_x_masked_if_1*velocity_nb_x_masked_if_1 + velocity_nb_y_masked_if_1*velocity_nb_y_masked_if_1 + velocity_nb_z_masked_if_1*velocity_nb_z_masked_if_1;
//__device__ inline half2 compute_speed_sqd(half2_3& velocity)
//~ {
	//~ return velocity.x*velocity.x + velocity.y*velocity.y + velocity.z*velocity.z;
//~ }
			pressure_nb_masked_if_1                      = compute_pressure(density_nb_masked_if_1, density_energy_nb_masked_if_1, speed_sqd_nb_masked_if_1);
			speed_of_sound_nb_masked_if_1                 = compute_speed_of_sound(density_nb_masked_if_1, pressure_nb_masked_if_1);

//compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
////calling inline functions is bad for automatic parsing, use below instead
//compute_flux_contribution(half2& density, half2_3& momentum, half2& density_energy, half2& pressure, half2_3& velocity, half2_3& fc_momentum_x, half2_3& fc_momentum_y, half2_3& fc_momentum_z, half2_3& fc_density_energy)
			//~ flux_contribution_nb_momentum_x_x_masked_if_1 = velocity_nb_x_masked_if_1*momentum_nb_x_masked_if_1 + pressure_nb_masked_if_1;
			//~ flux_contribution_nb_momentum_x_y_masked_if_1 = velocity_nb_x_masked_if_1*momentum_nb_y_masked_if_1;
			//~ flux_contribution_nb_momentum_x_z_masked_if_1 = velocity_nb_x_masked_if_1*momentum_nb_z_masked_if_1;
	
	
			//~ flux_contribution_nb_momentum_y_x_masked_if_1 = flux_contribution_nb_momentum_x_y_masked_if_1;
			//~ flux_contribution_nb_momentum_y_y_masked_if_1 = velocity_nb_y_masked_if_1*momentum_nb_y_masked_if_1 + pressure_nb_masked_if_1;
			//~ flux_contribution_nb_momentum_y_z_masked_if_1 = velocity_nb_y_masked_if_1*momentum_nb_z_masked_if_1;

			//~ flux_contribution_nb_momentum_z_x_masked_if_1 = flux_contribution_nb_momentum_x_z_masked_if_1;
			//~ flux_contribution_nb_momentum_z_y_masked_if_1 = flux_contribution_nb_momentum_y_z_masked_if_1;
			//~ flux_contribution_nb_momentum_z_z_masked_if_1 = velocity_nb_z_masked_if_1*momentum_nb_z_masked_if_1 + pressure_nb_masked_if_1;

			//~ half2 de_p = density_energy_nb_masked_if_1+pressure_nb_masked_if_1;
			//~ flux_contribution_nb_density_energy_x_masked_if_1 = velocity_nb_x_masked_if_1*de_p;
			//~ flux_contribution_nb_density_energy_y_masked_if_1 = velocity_nb_y_masked_if_1*de_p;
			//~ flux_contribution_nb_density_energy_z_masked_if_1 = velocity_nb_z_masked_if_1*de_p;	

			flux_contribution_nb_momentum_x.x = velocity_nb_x_masked_if_1*momentum_nb_x_masked_if_1 + pressure_nb_masked_if_1;
			flux_contribution_nb_momentum_x.y = velocity_nb_x_masked_if_1*momentum_nb_y_masked_if_1;
			flux_contribution_nb_momentum_x.z = velocity_nb_x_masked_if_1*momentum_nb_z_masked_if_1;
	
	
			flux_contribution_nb_momentum_y.x = flux_contribution_nb_momentum_x.y;
			flux_contribution_nb_momentum_y.y = velocity_nb_y_masked_if_1*momentum_nb.y + pressure_nb;
			flux_contribution_nb_momentum_y.z = velocity_nb_y_masked_if_1*momentum_nb.z;

			flux_contribution_nb_momentum_z.x = flux_contribution_nb_momentum_x.z;
			flux_contribution_nb_momentum_z.y = flux_contribution_nb_momentum_y.z;
			flux_contribution_nb_momentum_z.z = velocity_nb_z_masked_if_1*momentum_nb_z_masked_if_1 + pressure_nb_masked_if_1;

			half2 de_p = density_energy_nb_masked_if_1+pressure_nb_masked_if_1;
			flux_contribution_nb_density_energy.x = velocity_nb_x_masked_if_1*de_p;
			flux_contribution_nb_density_energy.y = velocity_nb_y_masked_if_1*de_p;
			flux_contribution_nb_density_energy.z = velocity_nb_z_masked_if_1*de_p;	


				
			//end compute_flux_contribution

			// artificial viscosity
			factor_masked_if_1 = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb_masked_if_1) + speed_of_sound_i + speed_of_sound_nb_masked_if_1);
			flux_i_density_masked_if_1 += factor_masked_if_1*(density_i-density_nb_masked_if_1);
			flux_i_density_energy_masked_if_1 += factor_masked_if_1*(density_energy_i-density_energy_nb_masked_if_1);
			flux_i_momentum_x_masked_if_1 += factor_masked_if_1*(momentum_i.x-momentum_nb_x_masked_if_1);
			flux_i_momentum_y_masked_if_1 += factor_masked_if_1*(momentum_i.y-momentum_nb_y_masked_if_1);
			flux_i_momentum_z_masked_if_1 += factor_masked_if_1*(momentum_i.z-momentum_nb_z_masked_if_1);

			// accumulate cell-centered fluxes
			factor_masked_if_1 = float(0.5f)*normal.x;
			flux_i_density_masked_if_1 += factor_masked_if_1*(momentum_nb_x_masked_if_1+momentum_i.x);
			flux_i_density_energy_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum_x_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum_y_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum_z_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor_masked_if_1 = float(0.5f)*normal.y;
			flux_i_density_masked_if_1 += factor_masked_if_1*(momentum_nb_y_masked_if_1+momentum_i.y);
			flux_i_density_energy_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum_x_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum_y_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum_z_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor_masked_if_1 = float(0.5f)*normal.z;
			flux_i_density_masked_if_1 += factor_masked_if_1*(momentum_nb_z_masked_if_1+momentum_i.z);
			flux_i_density_energy_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum_x_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum_y_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum_z_masked_if_1 += factor_masked_if_1*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);

		//~ }



		
		//prologue if_2 
			half2 flux_i_momentum_x_masked_if_2 = flux_i_momentum.x;
			half2 flux_i_momentum_y_masked_if_2 = flux_i_momentum.y;
			half2 flux_i_momentum_z_masked_if_2 = flux_i_momentum.z;
			
			bool flag1_if_2 = (nb1 == -1);
			bool flag2_if_2 = (nb2 == -1);
		//end prologue if 2
		
		//~ else if(nb == -1)	// a wing boundary
		//~ {
		
			flux_i_momentum_x_masked_if_2 += normal.x*pressure_i;
			flux_i_momentum_y_masked_if_2 += normal.y*pressure_i;
			flux_i_momentum_z_masked_if_2 += normal.z*pressure_i;
			
		//~ }
		
		//prologue if_3
			half2 factor_masked_if_3 = factor;
			half2 flux_i_density_masked_if_3 = flux_i_density;
			half2 flux_i_density_energy_masked_if_3 = flux_i_density_energy;
			half2 flux_i_momentum_x_masked_if_3 = flux_i_momentum.x;
			half2 flux_i_momentum_y_masked_if_3 = flux_i_momentum.y;
			half2 flux_i_momentum_z_masked_if_3 = flux_i_momentum.z;
			
			bool flag1_if_3 = (nb1 == -2);
			bool flag2_if_3 = (nb2 == -2);		
		
		//end prologue if_3
		
		//~ else if(nb == -2) // a far field boundary
		//~ {
			factor_masked_if_3 = float(0.5f)*normal.x;
			flux_i_density_masked_if_3 += factor_masked_if_3*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
			flux_i_density_energy_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
			flux_i_momentum_x_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
			flux_i_momentum_y_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
			flux_i_momentum_z_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
			
			factor_masked_if_3 = float(0.5f)*normal.y;
			flux_i_density += factor_masked_if_3*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
			flux_i_density_energy += factor_masked_if_3*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
			flux_i_momentum_x_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
			flux_i_momentum_y_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
			flux_i_momentum_z_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

			factor_masked_if_3 = float(0.5f)*normal.z;
			flux_i_density += factor_masked_if_3*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
			flux_i_density_energy += factor_masked_if_3*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
			flux_i_momentum_x_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
			flux_i_momentum_y_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
			flux_i_momentum_z_masked_if_3 += factor_masked_if_3*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

		//~ }
		//epilogue if_1
		if (flag1_if_1){
		//~ ((half*)&(density_nb))[0] = ((half*)&(density_nb_masked_if_1))[0];
		//~ ((half*)&(momentum_nb.x))[0] = ((half*)&( momentum_nb_x_masked_if_1))[0];
		//~ ((half*)&(momentum_nb.y))[0] = ((half*)&( momentum_nb_y_masked_if_1))[0];
		//~ ((half*)&(momentum_nb.z))[0] = ((half*)&( momentum_nb_z_masked_if_1))[0];
		 //~ ((half*)&(density_energy_nb))[0] = ((half*)&(density_energy_nb_masked_if_1 ))[0];
		 //~ ((half*)&(velocity_nb.x))[0] = ((half*)&( velocity_nb_x_masked_if_1))[0];
		 //~ ((half*)&(velocity_nb.y))[0] = ((half*)&( velocity_nb_y_masked_if_1))[0];
		 //~ ((half*)&( velocity_nb.z))[0] = ((half*)&( velocity_nb_z_masked_if_1))[0];
		   //~ ((half*)&( speed_sqd_nb))[0] = ((half*)&( speed_sqd_nb_masked_if_1))[0];
		   //~ ((half*)&( pressure_nb))[0] = ((half*)&( pressure_nb_masked_if_1))[0];
		  //~ ((half*)&(  speed_of_sound_nb))[0] = ((half*)&( speed_of_sound_nb_masked_if_1))[0];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.x))[0] = ((half*)&(flux_contribution_nb_momentum_x_x_masked_if_1 ))[0];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.y))[0] = ((half*)&( flux_contribution_nb_momentum_x_y_masked_if_1))[0];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.z))[0] = ((half*)&( flux_contribution_nb_momentum_x_z_masked_if_1))[0];
		
		 //~ ((half*)&(   flux_contribution_nb_momentum_y.x))[0] = ((half*)&( flux_contribution_nb_momentum_y_x_masked_if_1))[0];
		  //~ ((half*)&(  flux_contribution_nb_momentum_y.y))[0] = ((half*)&(flux_contribution_nb_momentum_y_y_masked_if_1 ))[0];
		 //~ ((half*)&(   flux_contribution_nb_momentum_y.z))[0] = ((half*)&(flux_contribution_nb_momentum_y_z_masked_if_1 ))[0];
		
		 //~ ((half*)&(   flux_contribution_nb_momentum_z.x))[0] = ((half*)&(flux_contribution_nb_momentum_z_x_masked_if_1 ))[0];
		  //~ ((half*)&(  flux_contribution_nb_momentum_z.y))[0] = ((half*)&( flux_contribution_nb_momentum_z_y_masked_if_1))[0];
		   //~ ((half*)&( flux_contribution_nb_momentum_z.z))[0] = ((half*)&(flux_contribution_nb_momentum_z_z_masked_if_1 ))[0];
		
		   //~ ((half*)&( flux_contribution_nb_density_energy.x))[0] = ((half*)&( flux_contribution_nb_density_energy_x_masked_if_1))[0];
		   //~ ((half*)&( flux_contribution_nb_density_energy.y))[0] = ((half*)&( flux_contribution_nb_density_energy_y_masked_if_1))[0];
		   //~ ((half*)&( flux_contribution_nb_density_energy.z))[0] = ((half*)&( flux_contribution_nb_density_energy_z_masked_if_1))[0];
		
		   //~ ((half*)&( factor))[0] = ((half*)&( factor_masked_if_1))[0];
		   ((half*)&( flux_i_density))[0] = ((half*)&( flux_i_density_masked_if_1))[0];
		   ((half*)&( flux_i_density_energy))[0] = ((half*)&( flux_i_density_energy_masked_if_1))[0];
		   ((half*)&( flux_i_momentum.x))[0] = ((half*)&( flux_i_momentum_x_masked_if_1))[0];
		   ((half*)&( flux_i_momentum.y))[0] = ((half*)&( flux_i_momentum_y_masked_if_1))[0];
		   ((half*)&( flux_i_momentum.z))[0] = ((half*)&( flux_i_momentum_z_masked_if_1))[0];		
		}
		if (flag2_if_1){
		//~ ((half*)&(density_nb))[1] = ((half*)&(density_nb_masked_if_1))[1];
		//~ ((half*)&(momentum_nb.x))[1] = ((half*)&( momentum_nb_x_masked_if_1))[1];
		//~ ((half*)&(momentum_nb.y))[1] = ((half*)&( momentum_nb_y_masked_if_1))[1];
		//~ ((half*)&(momentum_nb.z))[1] = ((half*)&( momentum_nb_z_masked_if_1))[1];
		 //~ ((half*)&(density_energy_nb))[1] = ((half*)&(density_energy_nb_masked_if_1 ))[1];
		 //~ ((half*)&(velocity_nb.x))[1] = ((half*)&( velocity_nb_x_masked_if_1))[1];
		 //~ ((half*)&(velocity_nb.y))[1] = ((half*)&( velocity_nb_y_masked_if_1))[1];
		 //~ ((half*)&( velocity_nb.z))[1] = ((half*)&( velocity_nb_z_masked_if_1))[1];
		   //~ ((half*)&( speed_sqd_nb))[1] = ((half*)&( speed_sqd_nb_masked_if_1))[1];
		   //~ ((half*)&( pressure_nb))[1] = ((half*)&( pressure_nb_masked_if_1))[1];
		  //~ ((half*)&(  speed_of_sound_nb))[1] = ((half*)&( speed_of_sound_nb_masked_if_1))[1];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.x))[1] = ((half*)&(flux_contribution_nb_momentum_x_x_masked_if_1 ))[1];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.y))[1] = ((half*)&( flux_contribution_nb_momentum_x_y_masked_if_1))[1];
		  //~ ((half*)&(  flux_contribution_nb_momentum_x.z))[1] = ((half*)&( flux_contribution_nb_momentum_x_z_masked_if_1))[1];
		
		 //~ ((half*)&(   flux_contribution_nb_momentum_y.x))[1] = ((half*)&( flux_contribution_nb_momentum_y_x_masked_if_1))[1];
		  //~ ((half*)&(  flux_contribution_nb_momentum_y.y))[1] = ((half*)&(flux_contribution_nb_momentum_y_y_masked_if_1 ))[1];
		 //~ ((half*)&(   flux_contribution_nb_momentum_y.z))[1] = ((half*)&(flux_contribution_nb_momentum_y_z_masked_if_1 ))[1];
		
		 //~ ((half*)&(   flux_contribution_nb_momentum_z.x))[1] = ((half*)&(flux_contribution_nb_momentum_z_x_masked_if_1 ))[1];
		  //~ ((half*)&(  flux_contribution_nb_momentum_z.y))[1] = ((half*)&( flux_contribution_nb_momentum_z_y_masked_if_1))[1];
		   //~ ((half*)&( flux_contribution_nb_momentum_z.z))[1] = ((half*)&(flux_contribution_nb_momentum_z_z_masked_if_1 ))[1];
		
		   //~ ((half*)&( flux_contribution_nb_density_energy.x))[1] = ((half*)&( flux_contribution_nb_density_energy_x_masked_if_1))[1];
		   //~ ((half*)&( flux_contribution_nb_density_energy.y))[1] = ((half*)&( flux_contribution_nb_density_energy_y_masked_if_1))[1];
		   //~ ((half*)&( flux_contribution_nb_density_energy.z))[1] = ((half*)&( flux_contribution_nb_density_energy_z_masked_if_1))[1];
		
		   //~ ((half*)&( factor))[1] = ((half*)&( factor_masked_if_1))[1];
		   ((half*)&( flux_i_density))[1] = ((half*)&( flux_i_density_masked_if_1))[1];
		   ((half*)&( flux_i_density_energy))[1] = ((half*)&( flux_i_density_energy_masked_if_1))[1];
		   ((half*)&( flux_i_momentum.x))[1] = ((half*)&( flux_i_momentum_x_masked_if_1))[1];
		   ((half*)&( flux_i_momentum.y))[1] = ((half*)&( flux_i_momentum_y_masked_if_1))[1];
		   ((half*)&( flux_i_momentum.z))[1] = ((half*)&( flux_i_momentum_z_masked_if_1))[1];		
		}
		//epilogue if_2
		if(flag1_if_2){
				((half*)&(flux_i_momentum.x))[0] = ((half*)&(flux_i_momentum_x_masked_if_2))[0];
				((half*)&(flux_i_momentum.y))[0] = ((half*)&(flux_i_momentum_y_masked_if_2))[0];
			((half*)&(flux_i_momentum.z))[0] = ((half*)&(flux_i_momentum_z_masked_if_2))[0];
			}
			if(flag2_if_2){
				((half*)&(flux_i_momentum.x))[1] = ((half*)&(flux_i_momentum_x_masked_if_2))[1];
				((half*)&(flux_i_momentum.y))[1] = ((half*)&(flux_i_momentum_y_masked_if_2))[1];
				((half*)&(flux_i_momentum.z))[1] = ((half*)&(flux_i_momentum_z_masked_if_2))[1];
			}
		//epilogue if_3
		if(flag1_if_3){
			//~ ((half*)&(factor))[0] = ((half*)&(factor_masked_if_3))[0];
			((half*)&(flux_i_density))[0] = ((half*)&(flux_i_density_masked_if_3))[0] ;
			((half*)&(flux_i_density_energy))[0] = ((half*)&(flux_i_density_energy_masked_if_3))[0] ;
			((half*)&(flux_i_momentum.x))[0]  = ((half*)&(flux_i_momentum_x_masked_if_3))[0];
			((half*)&(flux_i_momentum.y))[0]  = ((half*)&(flux_i_momentum_y_masked_if_3))[0] ;
			((half*)&(flux_i_momentum.z))[0]  = ((half*)&(flux_i_momentum_z_masked_if_3))[0];
			
			}
		if(flag2_if_3){
			//~ ((half*)&(factor))[1] = ((half*)&(factor_masked_if_3))[1];
			((half*)&(flux_i_density))[1] = ((half*)&(flux_i_density_masked_if_3))[1] ;
			((half*)&(flux_i_density_energy))[1] = ((half*)&(flux_i_density_energy_masked_if_3))[1] ;
			((half*)&(flux_i_momentum.x))[1]  = ((half*)&(flux_i_momentum_x_masked_if_3))[1];
			((half*)&(flux_i_momentum.y))[1]  = ((half*)&(flux_i_momentum_y_masked_if_3))[1] ;
			((half*)&(flux_i_momentum.z))[1]  = ((half*)&(flux_i_momentum_z_masked_if_3))[1];
			
			}
			


/*		if(nb1 >= 0) 	// a legitimate neighbor
		{
			density_nb = __half2half2(variables_half[nb1 + VAR_DENSITY*nelr*2]);
			momentum_nb.x = __half2half2(variables_half[nb1 + (VAR_MOMENTUM+0)*nelr*2]);
			momentum_nb.y = __half2half2(variables_half[nb1 + (VAR_MOMENTUM+1)*nelr*2]);
			momentum_nb.z = __half2half2(variables_half[nb1 + (VAR_MOMENTUM+2)*nelr*2]);
			density_energy_nb = __half2half2(variables_half[nb1 + VAR_DENSITY_ENERGY*nelr*2]);
			
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			((half*)&flux_i_density)[0] += __low2half(factor*(density_i-density_nb));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(density_energy_i-density_energy_nb));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(momentum_i.x-momentum_nb.x));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(momentum_i.y-momentum_nb.y));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(momentum_i.z-momentum_nb.z));

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			((half*)&flux_i_density)[0] += __low2half(factor*(momentum_nb.x+momentum_i.x));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x));
			
			factor = float(0.5f)*normal.y;
			((half*)&flux_i_density)[0] += __low2half(factor*(momentum_nb.y+momentum_i.y));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y));
			
			factor = float(0.5f)*normal.z;
			((half*)&flux_i_density)[0] += __low2half(factor*(momentum_nb.z+momentum_i.z));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z));
		}
		else if(nb1 == -1)	// a wing boundary
		{
			((half*)&flux_i_momentum.x)[0] += __low2half(normal.x*pressure_i);
			((half*)&flux_i_momentum.y)[0] += __low2half(normal.y*pressure_i);
			((half*)&flux_i_momentum.z)[0] += __low2half(normal.z*pressure_i);
		}
		else if(nb1 == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			((half*)&flux_i_density)[0] += __low2half(factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x));
			
			factor = float(0.5f)*normal.y;
			((half*)&flux_i_density)[0] += __low2half(factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y));

			factor = float(0.5f)*normal.z;
			((half*)&flux_i_density)[0] += __low2half(factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z));
			((half*)&flux_i_density_energy)[0] += __low2half(factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z));
			((half*)&flux_i_momentum.x)[0] += __low2half(factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z));
			((half*)&flux_i_momentum.y)[0] += __low2half(factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z));
			((half*)&flux_i_momentum.z)[0] += __low2half(factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z));

		}
		
		if(nb2 >= 0) 	// a legitimate neighbor
		{
			density_nb = __half2half2(variables_half[nb2 + VAR_DENSITY*nelr*2]);
			momentum_nb.x = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+0)*nelr*2]);
			momentum_nb.y = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+1)*nelr*2]);
			momentum_nb.z = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+2)*nelr*2]);
			density_energy_nb = __half2half2(variables_half[nb2 + VAR_DENSITY_ENERGY*nelr*2]);
			
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			((half*)&flux_i_density)[1] += __high2half(factor*(density_i-density_nb));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(density_energy_i-density_energy_nb));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(momentum_i.x-momentum_nb.x));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(momentum_i.y-momentum_nb.y));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(momentum_i.z-momentum_nb.z));

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			((half*)&flux_i_density)[1] += __high2half(factor*(momentum_nb.x+momentum_i.x));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x));
			
			factor = float(0.5f)*normal.y;
			((half*)&flux_i_density)[1] += __high2half(factor*(momentum_nb.y+momentum_i.y));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y));
			
			factor = float(0.5f)*normal.z;
			((half*)&flux_i_density)[1] += __high2half(factor*(momentum_nb.z+momentum_i.z));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z));
		}
		else if(nb2 == -1)	// a wing boundary
		{
			((half*)&flux_i_momentum.x)[1] += __high2half(normal.x*pressure_i);
			((half*)&flux_i_momentum.y)[1] += __high2half(normal.y*pressure_i);
			((half*)&flux_i_momentum.z)[1] += __high2half(normal.z*pressure_i);
		}
		else if(nb2 == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			((half*)&flux_i_density)[1] += __high2half(factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(ff_flux_contribution_density_energy[1].x+flux_contribution_i_density_energy.x));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(ff_flux_contribution_momentum_x[1].x + flux_contribution_i_momentum_x.x));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(ff_flux_contribution_momentum_y[1].x + flux_contribution_i_momentum_y.x));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(ff_flux_contribution_momentum_z[1].x + flux_contribution_i_momentum_z.x));
			
			factor = float(0.5f)*normal.y;
			((half*)&flux_i_density)[1] += __high2half(factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(ff_flux_contribution_density_energy[1].y+flux_contribution_i_density_energy.y));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(ff_flux_contribution_momentum_x[1].y + flux_contribution_i_momentum_x.y));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(ff_flux_contribution_momentum_y[1].y + flux_contribution_i_momentum_y.y));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(ff_flux_contribution_momentum_z[1].y + flux_contribution_i_momentum_z.y));

			factor = float(0.5f)*normal.z;
			((half*)&flux_i_density)[1] += __high2half(factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z));
			((half*)&flux_i_density_energy)[1] += __high2half(factor*(ff_flux_contribution_density_energy[1].z+flux_contribution_i_density_energy.z));
			((half*)&flux_i_momentum.x)[1] += __high2half(factor*(ff_flux_contribution_momentum_x[1].z + flux_contribution_i_momentum_x.z));
			((half*)&flux_i_momentum.y)[1] += __high2half(factor*(ff_flux_contribution_momentum_y[1].z + flux_contribution_i_momentum_y.z));
			((half*)&flux_i_momentum.z)[1] += __high2half(factor*(ff_flux_contribution_momentum_z[1].z + flux_contribution_i_momentum_z.z));

		}
		* 
		*/
		
	/*		
		if(nb2 >= 0) 	// a legitimate neighbor
		{
			density_nb = __half2half2(variables_half[nb2 + VAR_DENSITY*nelr*2]);
			momentum_nb.x = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+0)*nelr*2]);
			momentum_nb.y = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+1)*nelr*2]);
			momentum_nb.z = __half2half2(variables_half[nb2 + (VAR_MOMENTUM+2)*nelr*2]);
			density_energy_nb = __half2half2(variables_half[nb2 + VAR_DENSITY_ENERGY*nelr*2]);
			
												compute_velocity(density_nb, momentum_nb, velocity_nb);
			speed_sqd_nb                      = compute_speed_sqd(velocity_nb);
			pressure_nb                       = compute_pressure(density_nb, density_energy_nb, speed_sqd_nb);
			speed_of_sound_nb                 = compute_speed_of_sound(density_nb, pressure_nb);
			                                    compute_flux_contribution(density_nb, momentum_nb, density_energy_nb, pressure_nb, velocity_nb, flux_contribution_nb_momentum_x, flux_contribution_nb_momentum_y, flux_contribution_nb_momentum_z, flux_contribution_nb_density_energy);
			
			// artificial viscosity
			factor = -normal_len*smoothing_coefficient*float(0.5f)*(speed_i + sqrtf(speed_sqd_nb) + speed_of_sound_i + speed_of_sound_nb);
			flux_i_density += factor*(density_i-density_nb);
			flux_i_density_energy += factor*(density_energy_i-density_energy_nb);
			flux_i_momentum.x += factor*(momentum_i.x-momentum_nb.x);
			flux_i_momentum.y += factor*(momentum_i.y-momentum_nb.y);
			flux_i_momentum.z += factor*(momentum_i.z-momentum_nb.z);

			// accumulate cell-centered fluxes
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(momentum_nb.x+momentum_i.x);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.x+flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.x+flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.x+flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(momentum_nb.y+momentum_i.y);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.y+flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.y+flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.y+flux_contribution_i_momentum_z.y);
			
			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(momentum_nb.z+momentum_i.z);
			flux_i_density_energy += factor*(flux_contribution_nb_density_energy.z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(flux_contribution_nb_momentum_x.z+flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(flux_contribution_nb_momentum_y.z+flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(flux_contribution_nb_momentum_z.z+flux_contribution_i_momentum_z.z);
		}
		else if(nb2 == -1)	// a wing boundary
		{
			flux_i_momentum.x += normal.x*pressure_i;
			flux_i_momentum.y += normal.y*pressure_i;
			flux_i_momentum.z += normal.z*pressure_i;
		}
		else if(nb2 == -2) // a far field boundary
		{
			factor = float(0.5f)*normal.x;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+0]+momentum_i.x);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].x+flux_contribution_i_density_energy.x);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].x + flux_contribution_i_momentum_x.x);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].x + flux_contribution_i_momentum_y.x);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].x + flux_contribution_i_momentum_z.x);
			
			factor = float(0.5f)*normal.y;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+1]+momentum_i.y);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].y+flux_contribution_i_density_energy.y);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].y + flux_contribution_i_momentum_x.y);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].y + flux_contribution_i_momentum_y.y);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].y + flux_contribution_i_momentum_z.y);

			factor = float(0.5f)*normal.z;
			flux_i_density += factor*(ff_variable[VAR_MOMENTUM+2]+momentum_i.z);
			flux_i_density_energy += factor*(ff_flux_contribution_density_energy[0].z+flux_contribution_i_density_energy.z);
			flux_i_momentum.x += factor*(ff_flux_contribution_momentum_x[0].z + flux_contribution_i_momentum_x.z);
			flux_i_momentum.y += factor*(ff_flux_contribution_momentum_y[0].z + flux_contribution_i_momentum_y.z);
			flux_i_momentum.z += factor*(ff_flux_contribution_momentum_z[0].z + flux_contribution_i_momentum_z.z);

		}
		
	*/	
		
	}
	//~ ((half*)&(fluxes[i + VAR_DENSITY*nelr]))[0] = ((half*)&flux_i_density)[0];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+0)*nelr]))[0] = ((half*)&flux_i_momentum.x)[0];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+1)*nelr]))[0] = ((half*)&flux_i_momentum.y)[0];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+2)*nelr]))[0] = ((half*)&flux_i_momentum.z)[0];
	//~ ((half*)&(fluxes[i + VAR_DENSITY_ENERGY*nelr]))[0] = ((half*)&flux_i_density_energy)[0];

	//~ ((half*)&(fluxes[i + VAR_DENSITY*nelr]))[1] = ((half*)&flux_i_density)[1];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+0)*nelr]))[1] = ((half*)&flux_i_momentum.x)[1];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+1)*nelr]))[1] = ((half*)&flux_i_momentum.y)[1];
	//~ ((half*)&(fluxes[i + (VAR_MOMENTUM+2)*nelr]))[1] = ((half*)&flux_i_momentum.z)[1];
	//~ ((half*)&(fluxes[i + VAR_DENSITY_ENERGY*nelr]))[1] = ((half*)&flux_i_density_energy)[1];
			
	fluxes[i + VAR_DENSITY*nelr] = flux_i_density;
	fluxes[i + (VAR_MOMENTUM+0)*nelr] = flux_i_momentum.x;
	fluxes[i + (VAR_MOMENTUM+1)*nelr] = flux_i_momentum.y;
	fluxes[i + (VAR_MOMENTUM+2)*nelr] = flux_i_momentum.z;
	fluxes[i + VAR_DENSITY_ENERGY*nelr] = flux_i_density_energy;
}
void compute_flux(int nelr, int* elements_surrounding_elements, half2* normals, half2* variables, half2* fluxes)
{
	//~ dim3 Dg(nelr / BLOCK_SIZE_3), Db(BLOCK_SIZE_3);
	dim3 Dg(nelr / BLOCK_SIZE_3), Db(BLOCK_SIZE_3);
	cuda_compute_flux<<<Dg,Db>>>(nelr, elements_surrounding_elements, normals, variables, fluxes);
	getLastCudaError("compute_flux failed");
}

__global__ void cuda_time_step(int j, int nelr, half2* old_variables, half2* variables, half2* step_factors, half2* fluxes)
{
	const int i = (blockDim.x*blockIdx.x + threadIdx.x);

	half2 factor = step_factors[i]/float(RK+1-j);

	variables[i + VAR_DENSITY*nelr] = old_variables[i + VAR_DENSITY*nelr] + factor*fluxes[i + VAR_DENSITY*nelr];
	variables[i + VAR_DENSITY_ENERGY*nelr] = old_variables[i + VAR_DENSITY_ENERGY*nelr] + factor*fluxes[i + VAR_DENSITY_ENERGY*nelr];
	variables[i + (VAR_MOMENTUM+0)*nelr] = old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
	variables[i + (VAR_MOMENTUM+1)*nelr] = old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];	
	variables[i + (VAR_MOMENTUM+2)*nelr] = old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];	
	//~ variables[i + (VAR_MOMENTUM+0)*nelr] = fluxes[i + (VAR_MOMENTUM+0)*nelr];// old_variables[i + (VAR_MOMENTUM+0)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+0)*nelr];
	//~ variables[i + (VAR_MOMENTUM+1)*nelr] = fluxes[i + (VAR_MOMENTUM+1)*nelr];//old_variables[i + (VAR_MOMENTUM+1)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+1)*nelr];	
	//~ variables[i + (VAR_MOMENTUM+2)*nelr] = fluxes[i + (VAR_MOMENTUM+2)*nelr];//old_variables[i + (VAR_MOMENTUM+2)*nelr] + factor*fluxes[i + (VAR_MOMENTUM+2)*nelr];	
}
void time_step(int j, int nelr, half2* old_variables, half2* variables, half2* step_factors, half2* fluxes)
{
	//~ dim3 Dg(nelr / BLOCK_SIZE_4), Db(BLOCK_SIZE_4);
	dim3 Dg(nelr / BLOCK_SIZE_4), Db(BLOCK_SIZE_4);
	cuda_time_step<<<Dg,Db>>>(j, nelr, old_variables, variables, step_factors, fluxes);
	getLastCudaError("update failed");
}

/*
 * Main function
 */
int main(int argc, char** argv)
{
  printf("WG size of kernel:initialize = %d, WG size of kernel:compute_step_factor = %d, WG size of kernel:compute_flux = %d, WG size of kernel:time_step = %d\n", BLOCK_SIZE_1, BLOCK_SIZE_2, BLOCK_SIZE_3, BLOCK_SIZE_4);

	if (argc < 2)
	{
		std::cout << "specify data file name" << std::endl;
		return 0;
	}
	const char* data_file_name = argv[1];
	
	hipDeviceProp_t prop;
	int dev;
	
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipGetDevice(&dev));
	checkCudaErrors(hipGetDeviceProperties(&prop, dev));
	
	printf("Name:                     %s\n", prop.name);

	// set far field conditions and load them into constant memory on the gpu
	{
		half_float::half h_ff_variable[NVAR];
		const float angle_of_attack = float(3.1415926535897931 / 180.0f) * float(deg_angle_of_attack);
		
		h_ff_variable[VAR_DENSITY] = half_float::half(1.4);
		
		half_float::half ff_pressure = half_float::half(1.0f);
		half_float::half ff_speed_of_sound = half_float::half(sqrt(GAMMA*ff_pressure / h_ff_variable[VAR_DENSITY]));
		half_float::half ff_speed = half_float::half(ff_mach)*ff_speed_of_sound;
		
		half3_host ff_velocity;
		ff_velocity.x = ff_speed*float(cos((float)angle_of_attack));
		ff_velocity.y = ff_speed*float(sin((float)angle_of_attack));
		ff_velocity.z = 0.0f;
		
		h_ff_variable[VAR_MOMENTUM+0] = h_ff_variable[VAR_DENSITY] * ff_velocity.x;
		h_ff_variable[VAR_MOMENTUM+1] = h_ff_variable[VAR_DENSITY] * ff_velocity.y;
		h_ff_variable[VAR_MOMENTUM+2] = h_ff_variable[VAR_DENSITY] * ff_velocity.z;
				
		h_ff_variable[VAR_DENSITY_ENERGY] = h_ff_variable[VAR_DENSITY]*(float(0.5f)*(ff_speed*ff_speed)) + (ff_pressure / float(GAMMA-1.0f));

		half3_host h_ff_momentum;
		h_ff_momentum.x = *(h_ff_variable+VAR_MOMENTUM+0);
		h_ff_momentum.y = *(h_ff_variable+VAR_MOMENTUM+1);
		h_ff_momentum.z = *(h_ff_variable+VAR_MOMENTUM+2);
		half3_host h_ff_flux_contribution_momentum_x;
		half3_host h_ff_flux_contribution_momentum_y;
		half3_host h_ff_flux_contribution_momentum_z;
		half3_host h_ff_flux_contribution_density_energy;
		
		compute_flux_contribution(h_ff_variable[VAR_DENSITY], h_ff_momentum, h_ff_variable[VAR_DENSITY_ENERGY], ff_pressure, ff_velocity, h_ff_flux_contribution_momentum_x, h_ff_flux_contribution_momentum_y, h_ff_flux_contribution_momentum_z, h_ff_flux_contribution_density_energy);

		// copy far field conditions to the gpu
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_variable),          h_ff_variable,          NVAR*sizeof(half)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_x), &h_ff_flux_contribution_momentum_x, sizeof(half3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_y), &h_ff_flux_contribution_momentum_y, sizeof(half3)) );
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_momentum_z), &h_ff_flux_contribution_momentum_z, sizeof(half3)) );
		
		checkCudaErrors( hipMemcpyToSymbol(HIP_SYMBOL(ff_flux_contribution_density_energy), &h_ff_flux_contribution_density_energy, sizeof(half3)) );		
	}
	int nel;
	int nelr;
	
	// read in domain geometry
	half2* areas;
	half2* normals;
	
	
	int* elements_surrounding_elements;
	//device mem 
	
	{
		std::ifstream file(data_file_name);
	
		file >> nel;
		nelr = BLOCK_SIZE_0*((nel / BLOCK_SIZE_0 )+ std::min(1, nel % BLOCK_SIZE_0));

		float* h_areas = new float[nelr];
		int* h_elements_surrounding_elements = new int[nelr*NNB];
		float* h_normals = new float[nelr*NDIM*NNB];

				
		// read in data
		for(int i = 0; i < nel; i++)
		{
			file >> h_areas[i];
			for(int j = 0; j < NNB; j++)
			{
				file >> h_elements_surrounding_elements[i + j*nelr];
				if(h_elements_surrounding_elements[i+j*nelr] < 0) h_elements_surrounding_elements[i+j*nelr] = -1;
				h_elements_surrounding_elements[i + j*nelr]--; //it's coming in with Fortran numbering				
				
				for(int k = 0; k < NDIM; k++)
				{
					file >> h_normals[i + (j + k*NNB)*nelr];
					h_normals[i + (j + k*NNB)*nelr] = -h_normals[i + (j + k*NNB)*nelr];
				}
			}
		}
		
		// fill in remaining data
		int last = nel-1;
		for(int i = nel; i < nelr; i++)
		{
			h_areas[i] = h_areas[last];
			for(int j = 0; j < NNB; j++)
			{
				// duplicate the last element
				h_elements_surrounding_elements[i + j*nelr] = h_elements_surrounding_elements[last + j*nelr];	
				for(int k = 0; k < NDIM; k++) h_normals[last + (j + k*NNB)*nelr] = h_normals[last + (j + k*NNB)*nelr];
			}
		}
		
		half_float::half* h_areas_half = new half_float::half[nelr];
		half_float::half* h_normals_half = new half_float::half[nelr*NDIM*NNB];
		
		for(int i= 0;i<nelr; i++){
			h_areas_half[i] = half_float::half(h_areas[i]);
			}
		for(int i=0;i<nelr*NDIM*NNB; i++){
			h_normals_half[i] = half_float::half(h_normals[i]);
			
			}
		
		
		//~ areas = alloc<half>(nelr);
		areas = alloc<half2>(nelr/2);
		
		//~ upload<half>(areas, h_areas_half, nelr);
		upload<half2>(areas, h_areas_half, nelr/2);

		elements_surrounding_elements = alloc<int>(nelr*NNB);
		
		upload<int>(elements_surrounding_elements, h_elements_surrounding_elements, nelr*NNB);

		//~ normals = alloc<half>(nelr*NDIM*NNB);
		//~ upload<half>(normals, h_normals_half, nelr*NDIM*NNB);
		normals = alloc<half2>(nelr*NDIM*NNB/2);
		upload<half2>(normals, h_normals_half, nelr*NDIM*NNB/2);
				
		delete[] h_areas;
		delete[] h_elements_surrounding_elements;
		delete[] h_normals;
		delete[] h_areas_half;
		delete[] h_normals_half;
	}

	// Create arrays and set initial conditions
	half2* variables = alloc<half2>(nelr*NVAR/2);
	initialize_variables(nelr/2, variables);

	//~ half* old_variables = alloc<half>(nelr*NVAR);   	
	//~ half* fluxes = alloc<half>(nelr*NVAR);
	//~ half* step_factors = alloc<half>(nelr); 
	half2* old_variables = alloc<half2>(nelr*NVAR/2);   	
	half2* fluxes = alloc<half2>(nelr*NVAR/2);
	half2* step_factors = alloc<half2>(nelr/2); 

	// make sure all memory is floatly allocated before we start timing
	initialize_variables(nelr/2, old_variables);
	initialize_variables(nelr/2, fluxes);
	
	hipMemset( (void*) step_factors, 0, sizeof(half)*nelr );
	// make sure CUDA isn't still doing something before we start timing
	hipDeviceSynchronize();

	// these need to be computed the first time in order to compute time step
	std::cout << "Starting..." << std::endl;

	StopWatchInterface *timer = 0;
	  //	unsigned int timer = 0;

	// CUT_SAFE_CALL( cutCreateTimer( &timer));
	// CUT_SAFE_CALL( cutStartTimer( timer));
	sdkCreateTimer(&timer); 
	sdkStartTimer(&timer); 
	// Begin iterations
	//~ for(int i = 0; i < iterations; i++)
	{
		//~ copy<half2>(old_variables, variables, nelr*NVAR/2);
		copy<half2>(old_variables, variables, nelr*NVAR/2);
		
		// for the first iteration we compute the time step
		//~ compute_step_factor(nelr, variables, areas, step_factors);
		compute_step_factor(nelr/2, variables, areas, step_factors);
		getLastCudaError("compute_step_factor failed");
	
		int j =0;
		//~ for(int j = 0; j < RK; j++)
		{
			//~ compute_flux(nelr, elements_surrounding_elements, normals, variables, fluxes);
			compute_flux(nelr/2, elements_surrounding_elements, normals, variables, fluxes);
			getLastCudaError("compute_flux failed");			
			//~ time_step(j, nelr, old_variables, variables, step_factors, fluxes);
			time_step(j, nelr/2, old_variables, variables, step_factors, fluxes);
			getLastCudaError("time_step failed");			
		}
	}

	hipDeviceSynchronize();
	//	CUT_SAFE_CALL( cutStopTimer(timer) );  
	sdkStopTimer(&timer); 

	std::cout  << (sdkGetAverageTimerValue(&timer)/1000.0)  / iterations << " seconds per iteration" << std::endl;

	std::cout << "Saving solution..." << std::endl;
	dump((half*)variables, nel, nelr);
	std::cout << "Saved solution..." << std::endl;

	
	std::cout << "Cleaning up..." << std::endl;
	dealloc<half2>(areas);
	dealloc<int>(elements_surrounding_elements);
	dealloc<half2>(normals);
	
	dealloc<half2>(variables);
	dealloc<half2>(old_variables);
	dealloc<half2>(fluxes);
	dealloc<half2>(step_factors);

	std::cout << "Done..." << std::endl;

	return 0;
}
