#include "hip/hip_runtime.h"
#ifndef _KMEANS_CUDA_KERNEL_H_
#define _KMEANS_CUDA_KERNEL_H_

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "kmeans.h"

// FIXME: Make this a runtime selectable variable!
#define ASSUMED_NR_CLUSTERS 32

#define SDATA( index)      CUT_BANK_CHECKER(sdata, index)

// t_features has the layout dim0[points 0-m-1]dim1[ points 0-m-1]...
texture<short, 1, hipReadModeElementType> t_features;
// t_features_flipped has the layout point0[dim 0-n-1]point1[dim 0-n-1]
texture<short, 1, hipReadModeElementType> t_features_flipped;
texture<short, 1, hipReadModeElementType> t_clusters;


__constant__ half c_clusters[ASSUMED_NR_CLUSTERS*34];		/* constant memory for cluster centers */

/* ----------------- invert_mapping() --------------------- */
/* inverts data array from row-major to column-major.

   [p0,dim0][p0,dim1][p0,dim2] ...
   [p1,dim0][p1,dim1][p1,dim2] ...
   [p2,dim0][p2,dim1][p2,dim2] ...
										to
   [dim0,p0][dim0,p1][dim0,p2] ...
   [dim1,p0][dim1,p1][dim1,p2] ...
   [dim2,p0][dim2,p1][dim2,p2] ...
*/
__global__ void invert_mapping(half *input,			/* original */
							   half *output,			/* inverted */
							   int npoints,				/* npoints */
							   int nfeatures)			/* nfeatures */
{
	int point_id = threadIdx.x + blockDim.x*blockIdx.x;	/* id of thread */
	int i;

	if(point_id < npoints){
		for(i=0;i<nfeatures;i++)
			output[point_id + npoints*i] = input[point_id*nfeatures + i];
	}
	return;
}
/* ----------------- invert_mapping() end --------------------- */

/* to turn on the GPU delta and center reduction */
//#define GPU_DELTA_REDUCTION
//#define GPU_NEW_CENTER_REDUCTION


/* ----------------- kmeansPoint() --------------------- */
/* find the index of nearest cluster centers and change membership*/
__global__ void
kmeansPoint(half  *features,			/* in: [npoints*nfeatures] */
            int     nfeatures,
            int     npoints,
            int     nclusters,
            int    *membership,
			half  *clusters,
			half  *block_clusters,
			int    *block_deltas)
{

	// block ID
	const unsigned int block_id = gridDim.x*blockIdx.y+blockIdx.x;
	// point/thread ID
	const unsigned int point_id = block_id*blockDim.x*blockDim.y + threadIdx.x;

	int  index = -1;

	if (point_id < npoints)
	{
		int i, j;
		half min_dist = __float2half(65535);
		half dist;													/* distance square between a point to cluster center */

		/* find the cluster center id with min distance to pt */
		for (i=0; i<nclusters; i++) {
			int cluster_base_index = i*nfeatures;					/* base index of cluster centers for inverted array */
			half ans=__float2half(0.0);												/* Euclidean distance sqaure */

			for (j=0; j < nfeatures; j++)
			{
				int addr = point_id + j*npoints;					/* appropriate index of data point */
				short temp = tex1Dfetch(t_features,addr);
				half diff = (*((half*)(&temp)) -
							  c_clusters[cluster_base_index + j]);	/* distance between a data point to cluster centers */

				ans += diff*diff;									/* sum of squares */
			}
			dist = ans;

			/* see if distance is smaller than previous ones:
			if so, change minimum distance and save index of cluster center */
			if (dist < min_dist) {
				min_dist = dist;
				index    = i;
			}
		}

		membership[point_id] = index;
	}

}
#endif // #ifndef _KMEANS_CUDA_KERNEL_H_
