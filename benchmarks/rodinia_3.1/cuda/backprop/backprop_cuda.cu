#include "hip/hip_runtime.h"


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

// includes, kernels
#include "backprop_cuda_kernel.cu"
#include "backprop.h"

////////////////////////////////////////////////////////////////////////////////

extern "C"
void bpnn_layerforward(float *l1, float *l2, float **conn, int n1, int n2);

extern "C"
void bpnn_output_error(float *delta, float *target, float *output, int nj, float *err);

extern "C"
void bpnn_hidden_error(float *delta_h, int nh, float *delta_o, int no, float **who, float *hidden, float *err);

extern "C"
void bpnn_adjust_weights(float *delta, int ndelta, float *ly, int nly, float **w, float **oldw);


extern "C"
int setup(int argc, char** argv);

extern "C"
float **alloc_2d_dbl(int m, int n);

extern "C"
float squash(float x);

double gettime() {
  struct timeval t;
  gettimeofday(&t,NULL);
  return t.tv_sec+t.tv_usec*1e-6;
}

unsigned int num_threads = 0;
unsigned int num_blocks = 0;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{
	setup(argc, argv);
}


extern "C"
void bpnn_train_cuda(BPNN *net, float *eo, float *eh)
{
  int in, hid, out;
  //~ float out_err, hid_err;
  float *out_err = eo;
  float *hid_err = eh;

  in = net->input_n;
  hid = net->hidden_n;
  out = net->output_n;


  int m = 0;
  float *input_hidden_cuda;
  float *input_cuda;
  float *output_hidden_cuda;
  float *partial_sum;
  float *hidden_partial_sum;
  float *hidden_delta_cuda;
  float *input_prev_weights_cuda;
  float sum;
  float *input_weights_one_dim;
  float *input_weights_prev_one_dim;

  num_blocks = in / BLOCK_SIZE;
  dim3  grid( 1 , num_blocks);
  dim3  threads(BLOCK_SIZE , BLOCK_SIZE);

  input_weights_one_dim = (float *) malloc((in + 1)* (hid + 1) * sizeof(float));
  input_weights_prev_one_dim = (float *) malloc((in + 1)* (hid + 1) * sizeof(float));
  partial_sum = (float *) malloc(num_blocks * WIDTH * sizeof(float));

  // this preprocessing stage is added to correct the bugs of wrong memcopy using two-dimensional net->inputweights
  for (int k = 0; k <= in; k++) {
   for (int j = 0; j <= hid; j++) {
	  input_weights_one_dim[m] = net->input_weights[k][j];
	  input_weights_prev_one_dim[m] = net-> input_prev_weights[k][j];
	  m++;
    }
  }

  hipMalloc((void**) &input_cuda, (in + 1) * sizeof(float));
  hipMalloc((void**) &output_hidden_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float));
  hipMalloc((void**) &hidden_partial_sum, num_blocks * WIDTH * sizeof(float));


  printf("Performing GPU computation\n");

  printf("in= %d, hid = %d, numblocks = %d\n", in, hid, num_blocks);

  hipMemcpy(input_cuda, net->input_units, (in + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);

	float time_kernel = 0.0;
    float tmp_t;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

  bpnn_layerforward_CUDA<<< grid, threads >>>(input_cuda,
	                                          output_hidden_cuda,
											  input_hidden_cuda,
											  hidden_partial_sum,
											  in,
											  hid);

  hipDeviceSynchronize();
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	time_kernel += tmp_t;


  hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("bpnn kernel error: %s\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}

  hipMemcpy(partial_sum, hidden_partial_sum, num_blocks * WIDTH * sizeof(float), hipMemcpyDeviceToHost);

  for (int j = 1; j <= hid; j++) {
    sum = 0.0;
    for (int k = 0; k < num_blocks; k++) {
      sum += partial_sum[k * hid + j-1] ;
    }
	sum += net->input_weights[0][j];
	net-> hidden_units[j] = float(1.0 / (1.0 + exp(-sum)));
  }


 bpnn_layerforward(net->hidden_units, net->output_units, net->hidden_weights, hid, out);
  bpnn_output_error(net->output_delta, net->target, net->output_units, out, out_err);
  bpnn_hidden_error(net->hidden_delta, hid, net->output_delta, out, net->hidden_weights, net->hidden_units, hid_err);
  bpnn_adjust_weights(net->output_delta, out, net->hidden_units, hid, net->hidden_weights, net->hidden_prev_weights);



  hipMalloc((void**) &hidden_delta_cuda, (hid + 1) * sizeof(float));
  hipMalloc((void**) &input_prev_weights_cuda, (in + 1) * (hid + 1) * sizeof(float));

  hipMemcpy(hidden_delta_cuda, net->hidden_delta, (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_prev_weights_cuda, input_weights_prev_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(input_hidden_cuda, input_weights_one_dim, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyHostToDevice);


	hipEventRecord(start,0);

  bpnn_adjust_weights_cuda<<< grid, threads >>>(hidden_delta_cuda,
												hid,
												input_cuda,
												in,
												input_hidden_cuda,
												input_prev_weights_cuda
												);



	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&tmp_t, start, stop);
	time_kernel += tmp_t;

  hipMemcpy(net->input_units, input_cuda, (in + 1) * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(input_weights_one_dim, input_hidden_cuda, (in + 1) * (hid + 1) * sizeof(float), hipMemcpyDeviceToHost);


/*
  float max_rel_err_input = 0.0;
  float max_rel_err_weights = 0.0;
  int max_err_index;
  float rel_err= 0.0;
  FILE *pFile;
  pFile = fopen("gold_output.txt", "r");
  if (pFile == NULL) {
    printf ("err open file gold_output\n");
  }
  //fprintf(pFile, "net->input_units\n");
  float gold_input_units_val;
  for (int k = 0; k < in + 1; k++) {
    //fprintf(pFile, "%f\n", net->input_units[k]);
    fscanf(pFile, "%f\n", &gold_input_units_val);
    if(net->input_units[k] != 0){
      rel_err = fabs((gold_input_units_val - net->input_units[k])/net->input_units[k]);
    }
    else
      rel_err = fabs((gold_input_units_val));
    if (rel_err > max_rel_err_input){
      max_rel_err_input = rel_err;
      max_err_index = k;
    }
  }

  float gold_weights_one_dim_val;
  //fprintf(pFile, "input_weights_one_dim\n");
  for (int k = 0; k < (in + 1) * (hid + 1); k++) {
    //fprintf(pFile, "%f\n", input_weights_one_dim[k]);
    fscanf(pFile, "%f\n", &gold_weights_one_dim_val);
    if (input_weights_one_dim[k]!=0)
        rel_err = fabs((gold_weights_one_dim_val - input_weights_one_dim[k])/input_weights_one_dim[k]);
    else
        rel_err = fabs(gold_weights_one_dim_val);
    if(rel_err > max_rel_err_weights)
      max_rel_err_weights = rel_err;

  }

  printf ("%f, %f \n",max_rel_err_input,max_rel_err_weights);

*/

 FILE *pFile;
 pFile = fopen("gold_output.txt", "w");
 if (pFile == NULL) {
 fputs("fopen example", pFile);
   return;
 }
 //fprintf(pFile, "net->input_units\n");
 /*for (int k = 0; k < in + 1; k++)
 fprintf(pFile, "%f\n", net->input_units[k]);
 */
 //fprintf(pFile, "input_weights_one_dim\n");
 for (int k = 0; k < (in + 1) * (hid + 1); k++)
 fprintf(pFile, "%f, ", input_weights_one_dim[k]);

printf ("time %f \n ", time_kernel);

  hipFree(input_cuda);
  hipFree(output_hidden_cuda);
  hipFree(input_hidden_cuda);
  hipFree(hidden_partial_sum);
  hipFree(input_prev_weights_cuda);
  hipFree(hidden_delta_cuda);

  free(partial_sum);
  free(input_weights_one_dim);
  free(input_weights_prev_one_dim);



}
