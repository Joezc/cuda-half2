#include "hip/hip_runtime.h"
long long get_time() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return (tv.tv_sec * 1000000) + tv.tv_usec;
}
#include "newhalf.hpp"
#include <hip/hip_fp16.h>
#include "half2_operator_overload.cuh"
__global__ void hotspotOpt1(__half2 *p, __half2* tIn, __half2 *tOut, __half2 sdc,
        int nx, int ny, int nz,
        __half2 ce, __half2 cw,
        __half2 cn, __half2 cs,
        __half2 ct, __half2 cb,
        __half2 cc)
{
    __half2 amb_temp = __float2half2_rn(80.0);

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int c = (i*2 + j * nx);
    int xy = nx * ny;

    int W = (i == 0)        ? c : c - 1;
    int E = (i == nx/2-1)     ? c : c + 1;
    int N = (j == 0)        ? c : c - nx;
    int S = (j == ny-1)     ? c : c + nx;

    __half2 temp1, temp2, temp3;
    temp1 = temp2 = tIn[c/2];
    temp3 = tIn[(c+xy)/2];
    tOut[c/2] = cc * temp2 + cw * tIn[W/2] + ce * tIn[E/2] + cs * tIn[S/2]
        + cn * tIn[N/2] + cb * temp1 + ct * temp3 + sdc * p[c/2] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[(c+xy)/2];
        tOut[c/2] = cc * temp2 + cw * tIn[W/2] + ce * tIn[E/2] + cs * tIn[S/2]
            + cn * tIn[N/2] + cb * temp1 + ct * temp3 + sdc * p[c/2] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c/2] = cc * temp2 + cw * tIn[W/2] + ce * tIn[E/2] + cs * tIn[S/2]
        + cn * tIn[N/2] + cb * temp1 + ct * temp3 + sdc * p[c/2] + ct * amb_temp;
    return;
}

void hotspot_opt1(float *p, float *tIn, float *tOut,
        int nx, int ny, int nz,
        float Cap,
        float Rx, float Ry, float Rz,
        float dt, int numiter)
{
    float ce, cw, cn, cs, ct, cb, cc;
    float stepDivCap = dt / Cap;

		half_float::half* p_half;
		half_float::half* tIn_half;
		half_float::half* tOut_half;

		size_t half_size = sizeof(half)* nx * ny * nz;
		p_half = (half_float::half*)malloc(half_size);
		tIn_half =(half_float::half*)malloc(half_size);
		tOut_half = (half_float::half*)malloc(half_size);

    ce = cw =stepDivCap/ Rx;
    cn = cs =stepDivCap/ Ry;
    ct = cb =stepDivCap/ Rz;

    cc = 1.0 - (2.0*ce + 2.0*cn + 3.0*ct);

    size_t s = sizeof(half) * nx * ny * nz;

		for(int i = 0; i < nx * ny * nz; i++ ){

			p_half[i] = p[i];
			tIn_half[i] = tIn[i];
		}



    __half2  *tIn_d, *tOut_d, *p_d;

		printf ("val: %f %f %f %f %f %f %f %f \n",stepDivCap,ce,cw,cn,ct,cb,cc);
    hipMalloc((void**)&p_d,s);
    hipMalloc((void**)&tIn_d,s);
    hipMalloc((void**)&tOut_d,s);
    hipMemcpy(tIn_d, tIn_half, s, hipMemcpyHostToDevice);
    hipMemcpy(p_d, p_half, s, hipMemcpyHostToDevice);

    hipFuncSetCacheConfig(reinterpret_cast<const void*>(hotspotOpt1), hipFuncCachePreferL1);

    dim3 block_dim(64, 4, 1);
    dim3 grid_dim(nx /2 /64, ny / 4, 1);

		uint32_t stepDivCap_half = floats2half2 (stepDivCap,stepDivCap );
		__half2 stdc = *(__half2*)&(stepDivCap_half);

		uint32_t ce_half = floats2half2 (ce ,ce);
		__half2 ce_dev = *(__half2*)&ce_half;

		uint32_t cw_half = floats2half2 (cw, cw);
		__half2 cw_dev = *(__half2*)&cw_half;

		uint32_t cn_half = floats2half2 (cn, cn);
		__half2 cn_dev = *(__half2*)&cn_half;

		uint32_t cs_half = floats2half2 (cs, cs);
		__half2 cs_dev = *(__half2*)&cs_half;

		uint32_t ct_half = floats2half2 (ct, ct);
		__half2 ct_dev = *(__half2*)&ct_half;

		uint32_t cb_half = floats2half2 (cb, cb);
		__half2 cb_dev = *(__half2*)&cb_half;

		uint32_t cc_half = floats2half2 (cc,cc);
		__half2 cc_dev = *(__half2*)&cc_half;
float time_kernel = 0.0;
float tmp_t;
    long long start = get_time();
    hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);
	hipEventRecord(start_event,0);
	
    for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stdc, nx, ny, nz, ce_dev, cw_dev, cn_dev, cs_dev, ct_dev, cb_dev, cc_dev);
        __half2 *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }
/*
		for (int i = 0; i < numiter; ++i) {
        hotspotOpt1<<<grid_dim, block_dim>>>
            (p_d, tIn_d, tOut_d, stepDivCap, nx, ny, nz, ce, cw, cn, cs, ct, cb, cc);
        __half2 *t = tIn_d;
        tIn_d = tOut_d;
        tOut_d = t;
    }
*/
    hipDeviceSynchronize();
    
     hipDeviceSynchronize();
	hipEventRecord(stop_event,0);
	hipEventSynchronize(stop_event);
	hipEventElapsedTime(&tmp_t, start_event, stop_event);
	time_kernel += tmp_t;

    long long stop = get_time();
    float time = (float)((stop - start)/(1000.0 * 1000.0));
    printf("Time: %.3f (s)\n",time);
    printf("Time kernel: %.3f (s)\n",time_kernel);
    hipMemcpy(tOut_half, tOut_d, s, hipMemcpyDeviceToHost);

		for(int i = 0; i < nx * ny * nz; i++ ){
			tOut [i] = float(tOut_half[i]);

}

    hipFree(p_d);
    hipFree(tIn_d);
    hipFree(tOut_d);
    return;
}
