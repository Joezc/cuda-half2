
#include <hip/hip_runtime.h>
//~ #include <half.hpp>
__device__ void Vec_add(float *x, float *y , float* z, float gaaa[], int n) {
   /* blockDim.x = threads_per_block                            */
   /* First block gets first threads_per_block components.      */
   /* Second block gets next threads_per_block components, etc. */
   int i = blockDim.x * blockIdx.x + threadIdx.x;
	float test_val, test_val2;
   /* block_count*threads_per_block may be >= n */
   if (i < n) {


	z[i] = x[i] + y[i+2]  + gaaa[(2*3+i)+456];

		return;
   }
}  /* Vec_add */
